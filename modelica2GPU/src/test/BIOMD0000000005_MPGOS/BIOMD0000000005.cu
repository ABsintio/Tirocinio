#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD0000000005_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45
#define PRECISION double
const int NT   = 1000;
const int SD   = 28;
const int NCP  = 1;
const int NSP  = 10;
const int NISP = 0;
const int NE   = 0;
const int NA   = 20;
const int NIA  = 0;
const int NDO  = 1000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        // Settaggio dei valori iniziali degli ActualState
        Solver.SetHost(ProblemNumber, ActualState, 0, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 1, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 2, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 3, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 4, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 5, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 6, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 7, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 8, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 9, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 10, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 11, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 12, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 13, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 14, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 15, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 16, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 17, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 18, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 19, 1.0);
        Solver.SetHost(ProblemNumber, ActualState, 26, 1.0); 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)
        Solver.SetHost(ProblemNumber, Accessories, 1, 0.0);
        Solver.SetHost(ProblemNumber, Accessories, 4, 0.0);
        Solver.SetHost(ProblemNumber, Accessories, 5, 0.0);
        Solver.SetHost(ProblemNumber, Accessories, 15, 0.0);
        Solver.SetHost(ProblemNumber, Accessories, 18, 0.0);

        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)
    Solver.SetHost(SharedParameters, 0, 1.0);
    Solver.SetHost(SharedParameters, 1, 1000000.0);
    Solver.SetHost(SharedParameters, 2, 1000.0);
    Solver.SetHost(SharedParameters, 3, 200.0);
    Solver.SetHost(SharedParameters, 4, 0.0);
    Solver.SetHost(SharedParameters, 5, 0.015);
    Solver.SetHost(SharedParameters, 6, 0.0);
    Solver.SetHost(SharedParameters, 7, 0.6);
    Solver.SetHost(SharedParameters, 8, 180.0);
    Solver.SetHost(SharedParameters, 9, 0.018);

    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product1" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product11" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product13" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product15" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product17" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product19" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product2" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product4" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product6" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product9" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant0" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant10" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant12" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant14" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant16" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant18" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant3" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant5" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant7" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant8" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_C2_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_CP_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_EmptySet_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_M_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_YP_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_Y_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_cell" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_pM_amount" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction1_elmt_k6" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction2_elmt_k8notP" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction3_elmt_k9" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction4_elmt_k3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction5_elmt_k5notP" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction6_elmt_k1aa" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction7_elmt_k2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction8_elmt_k7" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction9_elmt_k4" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction9_elmt_k4prime" << ',';
        DataFile.width(Width); DataFile << "ACC_$cse1" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_C2_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_CP_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_CT_amount" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_EmptySet_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_M_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction1" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction2" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction3" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction4" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction5" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction6" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction7" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction8" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction9" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_YP_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_YT_amount" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_YT_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Y_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_pM_conc" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 19) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = 2000; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 64; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 1.0);
    Solver.SolverOption(TimeStepShrinkLimit, 0.2);

    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD0000000005.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

