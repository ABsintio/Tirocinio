#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD042_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 15;
const int NCP  = 1;
const int NSP  = 25;
const int NISP = 0;
const int NE   = 0;
const int NA   = 0;
const int NIA  = 0;
const int NDO  = 10000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_ACA" << ',';
        DataFile.width(Width); DataFile << "X_ADP" << ',';
        DataFile.width(Width); DataFile << "X_AMP" << ',';
        DataFile.width(Width); DataFile << "X_ATP" << ',';
        DataFile.width(Width); DataFile << "X_DPG" << ',';
        DataFile.width(Width); DataFile << "X_EtOH" << ',';
        DataFile.width(Width); DataFile << "X_F6P" << ',';
        DataFile.width(Width); DataFile << "X_FBP" << ',';
        DataFile.width(Width); DataFile << "X_GAP" << ',';
        DataFile.width(Width); DataFile << "X_GLC" << ',';
        DataFile.width(Width); DataFile << "X_NAD" << ',';
        DataFile.width(Width); DataFile << "X_NADH" << ',';
        DataFile.width(Width); DataFile << "X_P" << ',';
        DataFile.width(Width); DataFile << "X_PEP" << ',';
        DataFile.width(Width); DataFile << "X_PYR" << ',';
        DataFile.width(Width); DataFile << "sPAR_K1ATP" << ',';
        DataFile.width(Width); DataFile << "sPAR_K1GLC" << ',';
        DataFile.width(Width); DataFile << "sPAR_K2" << ',';
        DataFile.width(Width); DataFile << "sPAR_K2ATP" << ',';
        DataFile.width(Width); DataFile << "sPAR_K4GAP" << ',';
        DataFile.width(Width); DataFile << "sPAR_K4NAD" << ',';
        DataFile.width(Width); DataFile << "sPAR_K6ADP" << ',';
        DataFile.width(Width); DataFile << "sPAR_K6PEP" << ',';
        DataFile.width(Width); DataFile << "sPAR_K7PYR" << ',';
        DataFile.width(Width); DataFile << "sPAR_V1" << ',';
        DataFile.width(Width); DataFile << "sPAR_V2" << ',';
        DataFile.width(Width); DataFile << "sPAR_V4" << ',';
        DataFile.width(Width); DataFile << "sPAR_V6" << ',';
        DataFile.width(Width); DataFile << "sPAR_V7" << ',';
        DataFile.width(Width); DataFile << "sPAR_fl" << ',';
        DataFile.width(Width); DataFile << "sPAR_k10" << ',';
        DataFile.width(Width); DataFile << "sPAR_k2" << ',';
        DataFile.width(Width); DataFile << "sPAR_k3b" << ',';
        DataFile.width(Width); DataFile << "sPAR_k3f" << ',';
        DataFile.width(Width); DataFile << "sPAR_k5b" << ',';
        DataFile.width(Width); DataFile << "sPAR_k5f" << ',';
        DataFile.width(Width); DataFile << "sPAR_k8b" << ',';
        DataFile.width(Width); DataFile << "sPAR_k8f" << ',';
        DataFile.width(Width); DataFile << "sPAR_k9b" << ',';
        DataFile.width(Width); DataFile << "sPAR_k9f" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 64; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 1.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD042.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

