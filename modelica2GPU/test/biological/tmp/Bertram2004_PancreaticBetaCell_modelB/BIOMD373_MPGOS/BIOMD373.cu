#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD373_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 7;
const int NCP  = 1;
const int NSP  = 43;
const int NISP = 0;
const int NE   = 0;
const int NA   = 71;
const int NIA  = 0;
const int NDO  = 100000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_V" << ',';
        DataFile.width(Width); DataFile << "X_adp" << ',';
        DataFile.width(Width); DataFile << "X_c" << ',';
        DataFile.width(Width); DataFile << "X_cer" << ',';
        DataFile.width(Width); DataFile << "X_fbp" << ',';
        DataFile.width(Width); DataFile << "X_g6p" << ',';
        DataFile.width(Width); DataFile << "X_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_COMpartment" << ',';
        DataFile.width(Width); DataFile << "sPAR_Cm" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kserca" << ',';
        DataFile.width(Width); DataFile << "sPAR_Rgk" << ',';
        DataFile.width(Width); DataFile << "sPAR_VCa" << ',';
        DataFile.width(Width); DataFile << "sPAR_VK" << ',';
        DataFile.width(Width); DataFile << "sPAR_alpha" << ',';
        DataFile.width(Width); DataFile << "sPAR_atot" << ',';
        DataFile.width(Width); DataFile << "sPAR_bottom1" << ',';
        DataFile.width(Width); DataFile << "sPAR_cat" << ',';
        DataFile.width(Width); DataFile << "sPAR_epser" << ',';
        DataFile.width(Width); DataFile << "sPAR_famp" << ',';
        DataFile.width(Width); DataFile << "sPAR_fatp" << ',';
        DataFile.width(Width); DataFile << "sPAR_fbt" << ',';
        DataFile.width(Width); DataFile << "sPAR_fcyt" << ',';
        DataFile.width(Width); DataFile << "sPAR_fer" << ',';
        DataFile.width(Width); DataFile << "sPAR_ffbp" << ',';
        DataFile.width(Width); DataFile << "sPAR_fmt" << ',';
        DataFile.width(Width); DataFile << "sPAR_gCa" << ',';
        DataFile.width(Width); DataFile << "sPAR_gK" << ',';
        DataFile.width(Width); DataFile << "sPAR_gKCa" << ',';
        DataFile.width(Width); DataFile << "sPAR_gkatpbar" << ',';
        DataFile.width(Width); DataFile << "sPAR_k1" << ',';
        DataFile.width(Width); DataFile << "sPAR_k2" << ',';
        DataFile.width(Width); DataFile << "sPAR_k3" << ',';
        DataFile.width(Width); DataFile << "sPAR_k4" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdd" << ',';
        DataFile.width(Width); DataFile << "sPAR_kg" << ',';
        DataFile.width(Width); DataFile << "sPAR_kpmca" << ',';
        DataFile.width(Width); DataFile << "sPAR_ktd" << ',';
        DataFile.width(Width); DataFile << "sPAR_ktt" << ',';
        DataFile.width(Width); DataFile << "sPAR_lambda" << ',';
        DataFile.width(Width); DataFile << "sPAR_lambdaer" << ',';
        DataFile.width(Width); DataFile << "sPAR_pfkbas" << ',';
        DataFile.width(Width); DataFile << "sPAR_pleak" << ',';
        DataFile.width(Width); DataFile << "sPAR_r" << ',';
        DataFile.width(Width); DataFile << "sPAR_r1" << ',';
        DataFile.width(Width); DataFile << "sPAR_sigmaV" << ',';
        DataFile.width(Width); DataFile << "sPAR_taua" << ',';
        DataFile.width(Width); DataFile << "sPAR_taun" << ',';
        DataFile.width(Width); DataFile << "sPAR_topa1" << ',';
        DataFile.width(Width); DataFile << "sPAR_vg" << ',';
        DataFile.width(Width); DataFile << "ACC_ICa" << ',';
        DataFile.width(Width); DataFile << "ACC_IK" << ',';
        DataFile.width(Width); DataFile << "ACC_IKATP" << ',';
        DataFile.width(Width); DataFile << "ACC_IKCa" << ',';
        DataFile.width(Width); DataFile << "ACC_Jer" << ',';
        DataFile.width(Width); DataFile << "ACC_Jleak" << ',';
        DataFile.width(Width); DataFile << "ACC_Jmem" << ',';
        DataFile.width(Width); DataFile << "ACC_Jserca" << ',';
        DataFile.width(Width); DataFile << "ACC_adp3m" << ',';
        DataFile.width(Width); DataFile << "ACC_amp" << ',';
        DataFile.width(Width); DataFile << "ACC_atp" << ',';
        DataFile.width(Width); DataFile << "ACC_atp4m" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom10" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom11" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom12" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom13" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom14" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom15" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom16" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom2" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom3" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom4" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom5" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom6" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom7" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom8" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom9" << ',';
        DataFile.width(Width); DataFile << "ACC_bottomo" << ',';
        DataFile.width(Width); DataFile << "ACC_f6p" << ',';
        DataFile.width(Width); DataFile << "ACC_fback" << ',';
        DataFile.width(Width); DataFile << "ACC_katpo" << ',';
        DataFile.width(Width); DataFile << "ACC_mgadp" << ',';
        DataFile.width(Width); DataFile << "ACC_minf" << ',';
        DataFile.width(Width); DataFile << "ACC_ninf" << ',';
        DataFile.width(Width); DataFile << "ACC_pfk" << ',';
        DataFile.width(Width); DataFile << "ACC_rad" << ',';
        DataFile.width(Width); DataFile << "ACC_ratio" << ',';
        DataFile.width(Width); DataFile << "ACC_rgpdh" << ',';
        DataFile.width(Width); DataFile << "ACC_topa10" << ',';
        DataFile.width(Width); DataFile << "ACC_topa11" << ',';
        DataFile.width(Width); DataFile << "ACC_topa12" << ',';
        DataFile.width(Width); DataFile << "ACC_topa13" << ',';
        DataFile.width(Width); DataFile << "ACC_topa14" << ',';
        DataFile.width(Width); DataFile << "ACC_topa15" << ',';
        DataFile.width(Width); DataFile << "ACC_topa16" << ',';
        DataFile.width(Width); DataFile << "ACC_topa2" << ',';
        DataFile.width(Width); DataFile << "ACC_topa3" << ',';
        DataFile.width(Width); DataFile << "ACC_topa4" << ',';
        DataFile.width(Width); DataFile << "ACC_topa5" << ',';
        DataFile.width(Width); DataFile << "ACC_topa6" << ',';
        DataFile.width(Width); DataFile << "ACC_topa7" << ',';
        DataFile.width(Width); DataFile << "ACC_topa8" << ',';
        DataFile.width(Width); DataFile << "ACC_topa9" << ',';
        DataFile.width(Width); DataFile << "ACC_topb" << ',';
        DataFile.width(Width); DataFile << "ACC_topo" << ',';
        DataFile.width(Width); DataFile << "ACC_weight10" << ',';
        DataFile.width(Width); DataFile << "ACC_weight11" << ',';
        DataFile.width(Width); DataFile << "ACC_weight12" << ',';
        DataFile.width(Width); DataFile << "ACC_weight13" << ',';
        DataFile.width(Width); DataFile << "ACC_weight14" << ',';
        DataFile.width(Width); DataFile << "ACC_weight15" << ',';
        DataFile.width(Width); DataFile << "ACC_weight16" << ',';
        DataFile.width(Width); DataFile << "ACC_weight2" << ',';
        DataFile.width(Width); DataFile << "ACC_weight3" << ',';
        DataFile.width(Width); DataFile << "ACC_weight4" << ',';
        DataFile.width(Width); DataFile << "ACC_weight5" << ',';
        DataFile.width(Width); DataFile << "ACC_weight6" << ',';
        DataFile.width(Width); DataFile << "ACC_weight7" << ',';
        DataFile.width(Width); DataFile << "ACC_weight8" << ',';
        DataFile.width(Width); DataFile << "ACC_weight9" << ',';
        DataFile.width(Width); DataFile << "ACC_y" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 70) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD373.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

