#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD303_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 42;
const int NCP  = 1;
const int NSP  = 90;
const int NISP = 0;
const int NE   = 0;
const int NA   = 1;
const int NIA  = 0;
const int NDO  = 100000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_C1" << ',';
        DataFile.width(Width); DataFile << "X_C2" << ',';
        DataFile.width(Width); DataFile << "X_C2a" << ',';
        DataFile.width(Width); DataFile << "X_C2b" << ',';
        DataFile.width(Width); DataFile << "X_C3" << ',';
        DataFile.width(Width); DataFile << "X_C3a" << ',';
        DataFile.width(Width); DataFile << "X_C3b" << ',';
        DataFile.width(Width); DataFile << "X_C4" << ',';
        DataFile.width(Width); DataFile << "X_C4BP" << ',';
        DataFile.width(Width); DataFile << "X_C4BP_C4b" << ',';
        DataFile.width(Width); DataFile << "X_C4BP_GlcNac_LF_CRP" << ',';
        DataFile.width(Width); DataFile << "X_C4BP_PC_CRP" << ',';
        DataFile.width(Width); DataFile << "X_C4BP_PC_CRP_LF" << ',';
        DataFile.width(Width); DataFile << "X_C4a" << ',';
        DataFile.width(Width); DataFile << "X_C4b" << ',';
        DataFile.width(Width); DataFile << "X_C4b_C2a" << ',';
        DataFile.width(Width); DataFile << "X_C4b_C2a_C4BP" << ',';
        DataFile.width(Width); DataFile << "X_CRP" << ',';
        DataFile.width(Width); DataFile << "X_GlcNac" << ',';
        DataFile.width(Width); DataFile << "X_GlcNac_HF" << ',';
        DataFile.width(Width); DataFile << "X_GlcNac_HF_MASP" << ',';
        DataFile.width(Width); DataFile << "X_GlcNac_LF" << ',';
        DataFile.width(Width); DataFile << "X_GlcNac_LF_C1_MASP" << ',';
        DataFile.width(Width); DataFile << "X_GlcNac_LF_CRP" << ',';
        DataFile.width(Width); DataFile << "X_GlcNac_LF_CRP_C1" << ',';
        DataFile.width(Width); DataFile << "X_GlcNac_LF_CRP_MASP" << ',';
        DataFile.width(Width); DataFile << "X_GlcNac_LF_MASP" << ',';
        DataFile.width(Width); DataFile << "X_HF" << ',';
        DataFile.width(Width); DataFile << "X_LF" << ',';
        DataFile.width(Width); DataFile << "X_MASP" << ',';
        DataFile.width(Width); DataFile << "X_PC" << ',';
        DataFile.width(Width); DataFile << "X_PC_CRP" << ',';
        DataFile.width(Width); DataFile << "X_PC_CRP_C1" << ',';
        DataFile.width(Width); DataFile << "X_PC_CRP_LF" << ',';
        DataFile.width(Width); DataFile << "X_PC_CRP_LF_C1" << ',';
        DataFile.width(Width); DataFile << "X_PC_CRP_LF_C1_MASP" << ',';
        DataFile.width(Width); DataFile << "X_PC_CRP_LF_MASP" << ',';
        DataFile.width(Width); DataFile << "X_X" << ',';
        DataFile.width(Width); DataFile << "X_dC3b" << ',';
        DataFile.width(Width); DataFile << "X_dC4b_C2a" << ',';
        DataFile.width(Width); DataFile << "X_dC4b_C2a_C4BP" << ',';
        DataFile.width(Width); DataFile << "X_iC4b_C2a" << ',';
        DataFile.width(Width); DataFile << "sPAR_compartment" << ',';
        DataFile.width(Width); DataFile << "sPAR_k1_4_27" << ',';
        DataFile.width(Width); DataFile << "sPAR_k1_4_28" << ',';
        DataFile.width(Width); DataFile << "sPAR_k1_4_29" << ',';
        DataFile.width(Width); DataFile << "sPAR_k1_4_33" << ',';
        DataFile.width(Width); DataFile << "sPAR_k2_33" << ',';
        DataFile.width(Width); DataFile << "sPAR_ka01_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ka01_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_ka02_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ka02_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_ka03_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ka03_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_ka04_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ka04_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kb01_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kb01_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kb02_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kb02_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kb03_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kb03_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kb04_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kb04_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kc01_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kc01_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kc02" << ',';
        DataFile.width(Width); DataFile << "sPAR_kc03_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kc03_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kc04_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kc04_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd01_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd01_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd02_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd02_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd03_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd03_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd04_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd04_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd05_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd05_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd06_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd06_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd07_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd07_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd08_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd08_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd09_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd09_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd10_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd10_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd11_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd11_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_ke01_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ke01_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_ke02_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ke02_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_ke03_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ke03_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_ke04_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ke04_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_ke05_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ke05_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_ke06_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ke06_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_ke07_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ke07_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf01_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf01_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf02_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf02_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf03" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf04_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf04_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf05" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf06_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf06_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf07_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf07_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kg01_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kg01_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kg02_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kg02_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kg03_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kg03_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kg04_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kg04_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kt01" << ',';
        DataFile.width(Width); DataFile << "sPAR_kt02" << ',';
        DataFile.width(Width); DataFile << "sPAR_kt03" << ',';
        DataFile.width(Width); DataFile << "sPAR_kt04_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kt04_2" << ',';
        DataFile.width(Width); DataFile << "ACC_mC3" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 83) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 84) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 85) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 86) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 87) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 88) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 89) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD303.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

