#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD293_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 137;
const int NCP  = 1;
const int NSP  = 83;
const int NISP = 0;
const int NE   = 0;
const int NA   = 9;
const int NIA  = 0;
const int NDO  = 100000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_ADP" << ',';
        DataFile.width(Width); DataFile << "X_AMP" << ',';
        DataFile.width(Width); DataFile << "X_ATP" << ',';
        DataFile.width(Width); DataFile << "X_AggA1" << ',';
        DataFile.width(Width); DataFile << "X_AggA2" << ',';
        DataFile.width(Width); DataFile << "X_AggA3" << ',';
        DataFile.width(Width); DataFile << "X_AggA4" << ',';
        DataFile.width(Width); DataFile << "X_AggA5" << ',';
        DataFile.width(Width); DataFile << "X_AggD1" << ',';
        DataFile.width(Width); DataFile << "X_AggD2" << ',';
        DataFile.width(Width); DataFile << "X_AggD3" << ',';
        DataFile.width(Width); DataFile << "X_AggD4" << ',';
        DataFile.width(Width); DataFile << "X_AggD5" << ',';
        DataFile.width(Width); DataFile << "X_AggP1" << ',';
        DataFile.width(Width); DataFile << "X_AggP2" << ',';
        DataFile.width(Width); DataFile << "X_AggP3" << ',';
        DataFile.width(Width); DataFile << "X_AggP4" << ',';
        DataFile.width(Width); DataFile << "X_AggP5" << ',';
        DataFile.width(Width); DataFile << "X_AggP_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_AggS1" << ',';
        DataFile.width(Width); DataFile << "X_AggS2" << ',';
        DataFile.width(Width); DataFile << "X_AggS3" << ',';
        DataFile.width(Width); DataFile << "X_AggS4" << ',';
        DataFile.width(Width); DataFile << "X_AggS5" << ',';
        DataFile.width(Width); DataFile << "X_AggU1" << ',';
        DataFile.width(Width); DataFile << "X_AggU2" << ',';
        DataFile.width(Width); DataFile << "X_AggU3" << ',';
        DataFile.width(Width); DataFile << "X_AggU4" << ',';
        DataFile.width(Width); DataFile << "X_AggU5" << ',';
        DataFile.width(Width); DataFile << "X_DUB" << ',';
        DataFile.width(Width); DataFile << "X_E1" << ',';
        DataFile.width(Width); DataFile << "X_E1_Ub" << ',';
        DataFile.width(Width); DataFile << "X_E2" << ',';
        DataFile.width(Width); DataFile << "X_E2_Ub" << ',';
        DataFile.width(Width); DataFile << "X_E3" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded_Ub" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded_Ub2" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded_Ub2_UCHL1" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded_Ub3" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded_Ub3_UCHL1" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded_Ub4" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded_Ub4_UCHL1" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded_Ub5" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded_Ub5_UCHL1" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded_Ub6" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded_Ub6_UCHL1" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded_Ub7" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded_Ub7_UCHL1" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded_Ub8" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded_Ub8_UCHL1" << ',';
        DataFile.width(Width); DataFile << "X_E3SUB_SUB_misfolded_Ub_UCHL1" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP_Ub" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP_Ub2" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP_Ub2_DUB" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP_Ub3" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP_Ub3_DUB" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP_Ub4" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP_Ub4_DUB" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP_Ub5" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP_Ub5_DUB" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP_Ub6" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP_Ub6_DUB" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP_Ub7" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP_Ub7_DUB" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP_Ub8" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP_Ub8_DUB" << ',';
        DataFile.width(Width); DataFile << "X_E3_MisP_Ub_DUB" << ',';
        DataFile.width(Width); DataFile << "X_Lamp2a" << ',';
        DataFile.width(Width); DataFile << "X_Lamp2a_UCHL1_damaged" << ',';
        DataFile.width(Width); DataFile << "X_Lysosome" << ',';
        DataFile.width(Width); DataFile << "X_MisP" << ',';
        DataFile.width(Width); DataFile << "X_MisP_Ub4_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_MisP_Ub5_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_MisP_Ub6_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_MisP_Ub7_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_MisP_Ub8_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_NatP" << ',';
        DataFile.width(Width); DataFile << "X_Parkin" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam_Ub" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam_Ub2" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam_Ub2_DUB" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam_Ub3" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam_Ub3_DUB" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam_Ub4" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam_Ub4_DUB" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam_Ub5" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam_Ub5_DUB" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam_Ub6" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam_Ub6_DUB" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam_Ub7" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam_Ub7_DUB" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam_Ub8" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam_Ub8_DUB" << ',';
        DataFile.width(Width); DataFile << "X_Parkin_asyn_dam_Ub_DUB" << ',';
        DataFile.width(Width); DataFile << "X_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_ROS" << ',';
        DataFile.width(Width); DataFile << "X_SUB" << ',';
        DataFile.width(Width); DataFile << "X_SUB_misfolded" << ',';
        DataFile.width(Width); DataFile << "X_SUB_misfolded_Ub4_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_SUB_misfolded_Ub5_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_SUB_misfolded_Ub6_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_SUB_misfolded_Ub7_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_SUB_misfolded_Ub8_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_SeqAggP" << ',';
        DataFile.width(Width); DataFile << "X_Sink" << ',';
        DataFile.width(Width); DataFile << "X_Source" << ',';
        DataFile.width(Width); DataFile << "X_UCHL1" << ',';
        DataFile.width(Width); DataFile << "X_UCHL1_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_UCHL1_damaged" << ',';
        DataFile.width(Width); DataFile << "X_UCHL1_damaged_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_Ub" << ',';
        DataFile.width(Width); DataFile << "X_Ub_UCHL1" << ',';
        DataFile.width(Width); DataFile << "X_aggDUB" << ',';
        DataFile.width(Width); DataFile << "X_aggE3" << ',';
        DataFile.width(Width); DataFile << "X_aggMisP" << ',';
        DataFile.width(Width); DataFile << "X_aggParkin" << ',';
        DataFile.width(Width); DataFile << "X_aggSUB" << ',';
        DataFile.width(Width); DataFile << "X_aggUb" << ',';
        DataFile.width(Width); DataFile << "X_aggUchl1" << ',';
        DataFile.width(Width); DataFile << "X_aggUchl1dam" << ',';
        DataFile.width(Width); DataFile << "X_aggasyn" << ',';
        DataFile.width(Width); DataFile << "X_aggasyndam" << ',';
        DataFile.width(Width); DataFile << "X_asyn" << ',';
        DataFile.width(Width); DataFile << "X_asyn_Lamp2a" << ',';
        DataFile.width(Width); DataFile << "X_asyn_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_asyn_dam" << ',';
        DataFile.width(Width); DataFile << "X_asyn_dam_Ub4_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_asyn_dam_Ub5_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_asyn_dam_Ub6_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_asyn_dam_Ub7_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_asyn_dam_Ub8_Proteasome" << ',';
        DataFile.width(Width); DataFile << "X_kproteff" << ',';
        DataFile.width(Width); DataFile << "X_upregUb" << ',';
        DataFile.width(Width); DataFile << "sPAR_cytosol" << ',';
        DataFile.width(Width); DataFile << "sPAR_kCMAasyn" << ',';
        DataFile.width(Width); DataFile << "sPAR_kactDUB" << ',';
        DataFile.width(Width); DataFile << "sPAR_kactDUBProt" << ',';
        DataFile.width(Width); DataFile << "sPAR_kactProt" << ',';
        DataFile.width(Width); DataFile << "sPAR_kactUchl1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kagg1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kagg1dam" << ',';
        DataFile.width(Width); DataFile << "sPAR_kagg2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kagg2dam" << ',';
        DataFile.width(Width); DataFile << "sPAR_kaggSUB1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kaggSUB2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kaggasyn1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kaggasyn2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbinAggProt" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbinE1Ub" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbinE2Ub" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbinE3SUB" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbinLamp2aUCHL1dam" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbinMisPDUB" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbinMisPE3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbinProt" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbinSUBUCHL1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbinUCHL1Prot" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbinUbUCHL1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbinasynDUB" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbinasynLamp2a" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbinasynParkin" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbinasynProt" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdamUCHL1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdamasyn" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdegLysUCHL1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdegLysUCHL1dam" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdegProtUCHL1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdegasynProt" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisagg1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisagg2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisagg3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisagg4" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisagg5" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisaggSUB1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisaggSUB2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisaggSUB3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisaggSUB4" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisaggSUB5" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisaggasyn1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisaggasyn2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisaggasyn3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisaggasyn4" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisaggasyn5" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisaggasyndam1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisaggasyndam2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisaggasyndam3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisaggasyndam4" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisaggasyndam5" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisagguchl1dam1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisagguchl1dam2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisagguchl1dam3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisagguchl1dam4" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdisagguchl1dam5" << ',';
        DataFile.width(Width); DataFile << "sPAR_kgenROS" << ',';
        DataFile.width(Width); DataFile << "sPAR_kgenROSAggP" << ',';
        DataFile.width(Width); DataFile << "sPAR_kigrowth1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kigrowth2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kmisfold" << ',';
        DataFile.width(Width); DataFile << "sPAR_kmisfoldSUB" << ',';
        DataFile.width(Width); DataFile << "sPAR_kmonoUb" << ',';
        DataFile.width(Width); DataFile << "sPAR_kpolyUb" << ',';
        DataFile.width(Width); DataFile << "sPAR_krefold" << ',';
        DataFile.width(Width); DataFile << "sPAR_krefoldSUB" << ',';
        DataFile.width(Width); DataFile << "sPAR_krelE3SUB" << ',';
        DataFile.width(Width); DataFile << "sPAR_krelLamp2aUCHL1dam" << ',';
        DataFile.width(Width); DataFile << "sPAR_krelMisPE3" << ',';
        DataFile.width(Width); DataFile << "sPAR_krelUbUCHL1" << ',';
        DataFile.width(Width); DataFile << "sPAR_krelasynParkin" << ',';
        DataFile.width(Width); DataFile << "sPAR_kremROS" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksynNatP" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksynSUB" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksynUCHL1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksynasyn" << ',';
        DataFile.width(Width); DataFile << "sPAR_kubd" << ',';
        DataFile.width(Width); DataFile << "sPAR_kubs" << ',';
        DataFile.width(Width); DataFile << "sPAR_kubss" << ',';
        DataFile.width(Width); DataFile << "ACC_AggP" << ',';
        DataFile.width(Width); DataFile << "ACC_Tot_MisP" << ',';
        DataFile.width(Width); DataFile << "ACC_Tot_Protein" << ',';
        DataFile.width(Width); DataFile << "ACC_Tot_UCHL1" << ',';
        DataFile.width(Width); DataFile << "ACC_Tot_Ub" << ',';
        DataFile.width(Width); DataFile << "ACC_Tot_asyn" << ',';
        DataFile.width(Width); DataFile << "ACC_Tot_asyn_dam" << ',';
        DataFile.width(Width); DataFile << "ACC_UCHL1_substrate" << ',';
        DataFile.width(Width); DataFile << "ACC_Ub_Conjugates" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 83) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 84) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 85) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 86) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 87) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 88) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 89) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 90) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 91) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 92) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 93) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 94) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 95) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 96) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 97) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 98) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 99) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 100) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 101) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 102) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 103) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 104) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 105) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 106) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 107) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 108) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 109) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 110) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 111) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 112) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 113) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 114) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 115) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 116) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 117) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 118) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 119) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 120) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 121) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 122) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 123) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 124) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 125) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 126) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 127) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 128) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 129) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 130) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 131) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 132) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 133) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 134) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 135) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 136) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 42, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 43, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 44, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 45, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 46, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 47, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 48, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 49, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 50, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 51, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 52, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 53, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 54, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 55, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 56, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 57, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 58, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 59, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 60, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 61, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 62, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 63, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 64, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 65, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 66, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 67, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 68, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 69, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 70, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 71, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 72, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 73, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 74, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 75, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 76, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 77, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 78, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 79, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 80, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 81, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 82, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 83, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 84, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 85, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 86, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 87, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 88, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 89, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 90, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 91, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 92, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 93, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 94, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 95, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 96, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 97, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 98, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 99, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 100, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 101, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 102, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 103, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 104, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 105, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 106, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 107, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 108, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 109, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 110, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 111, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 112, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 113, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 114, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 115, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 116, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 117, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 118, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 119, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 120, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 121, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 122, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 123, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 124, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 125, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 126, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 127, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 128, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 129, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 130, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 131, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 132, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 133, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 134, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 135, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 136, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
    Solver.SolverOption(RelativeTolerance, 42, 1e-06);
    Solver.SolverOption(RelativeTolerance, 43, 1e-06);
    Solver.SolverOption(RelativeTolerance, 44, 1e-06);
    Solver.SolverOption(RelativeTolerance, 45, 1e-06);
    Solver.SolverOption(RelativeTolerance, 46, 1e-06);
    Solver.SolverOption(RelativeTolerance, 47, 1e-06);
    Solver.SolverOption(RelativeTolerance, 48, 1e-06);
    Solver.SolverOption(RelativeTolerance, 49, 1e-06);
    Solver.SolverOption(RelativeTolerance, 50, 1e-06);
    Solver.SolverOption(RelativeTolerance, 51, 1e-06);
    Solver.SolverOption(RelativeTolerance, 52, 1e-06);
    Solver.SolverOption(RelativeTolerance, 53, 1e-06);
    Solver.SolverOption(RelativeTolerance, 54, 1e-06);
    Solver.SolverOption(RelativeTolerance, 55, 1e-06);
    Solver.SolverOption(RelativeTolerance, 56, 1e-06);
    Solver.SolverOption(RelativeTolerance, 57, 1e-06);
    Solver.SolverOption(RelativeTolerance, 58, 1e-06);
    Solver.SolverOption(RelativeTolerance, 59, 1e-06);
    Solver.SolverOption(RelativeTolerance, 60, 1e-06);
    Solver.SolverOption(RelativeTolerance, 61, 1e-06);
    Solver.SolverOption(RelativeTolerance, 62, 1e-06);
    Solver.SolverOption(RelativeTolerance, 63, 1e-06);
    Solver.SolverOption(RelativeTolerance, 64, 1e-06);
    Solver.SolverOption(RelativeTolerance, 65, 1e-06);
    Solver.SolverOption(RelativeTolerance, 66, 1e-06);
    Solver.SolverOption(RelativeTolerance, 67, 1e-06);
    Solver.SolverOption(RelativeTolerance, 68, 1e-06);
    Solver.SolverOption(RelativeTolerance, 69, 1e-06);
    Solver.SolverOption(RelativeTolerance, 70, 1e-06);
    Solver.SolverOption(RelativeTolerance, 71, 1e-06);
    Solver.SolverOption(RelativeTolerance, 72, 1e-06);
    Solver.SolverOption(RelativeTolerance, 73, 1e-06);
    Solver.SolverOption(RelativeTolerance, 74, 1e-06);
    Solver.SolverOption(RelativeTolerance, 75, 1e-06);
    Solver.SolverOption(RelativeTolerance, 76, 1e-06);
    Solver.SolverOption(RelativeTolerance, 77, 1e-06);
    Solver.SolverOption(RelativeTolerance, 78, 1e-06);
    Solver.SolverOption(RelativeTolerance, 79, 1e-06);
    Solver.SolverOption(RelativeTolerance, 80, 1e-06);
    Solver.SolverOption(RelativeTolerance, 81, 1e-06);
    Solver.SolverOption(RelativeTolerance, 82, 1e-06);
    Solver.SolverOption(RelativeTolerance, 83, 1e-06);
    Solver.SolverOption(RelativeTolerance, 84, 1e-06);
    Solver.SolverOption(RelativeTolerance, 85, 1e-06);
    Solver.SolverOption(RelativeTolerance, 86, 1e-06);
    Solver.SolverOption(RelativeTolerance, 87, 1e-06);
    Solver.SolverOption(RelativeTolerance, 88, 1e-06);
    Solver.SolverOption(RelativeTolerance, 89, 1e-06);
    Solver.SolverOption(RelativeTolerance, 90, 1e-06);
    Solver.SolverOption(RelativeTolerance, 91, 1e-06);
    Solver.SolverOption(RelativeTolerance, 92, 1e-06);
    Solver.SolverOption(RelativeTolerance, 93, 1e-06);
    Solver.SolverOption(RelativeTolerance, 94, 1e-06);
    Solver.SolverOption(RelativeTolerance, 95, 1e-06);
    Solver.SolverOption(RelativeTolerance, 96, 1e-06);
    Solver.SolverOption(RelativeTolerance, 97, 1e-06);
    Solver.SolverOption(RelativeTolerance, 98, 1e-06);
    Solver.SolverOption(RelativeTolerance, 99, 1e-06);
    Solver.SolverOption(RelativeTolerance, 100, 1e-06);
    Solver.SolverOption(RelativeTolerance, 101, 1e-06);
    Solver.SolverOption(RelativeTolerance, 102, 1e-06);
    Solver.SolverOption(RelativeTolerance, 103, 1e-06);
    Solver.SolverOption(RelativeTolerance, 104, 1e-06);
    Solver.SolverOption(RelativeTolerance, 105, 1e-06);
    Solver.SolverOption(RelativeTolerance, 106, 1e-06);
    Solver.SolverOption(RelativeTolerance, 107, 1e-06);
    Solver.SolverOption(RelativeTolerance, 108, 1e-06);
    Solver.SolverOption(RelativeTolerance, 109, 1e-06);
    Solver.SolverOption(RelativeTolerance, 110, 1e-06);
    Solver.SolverOption(RelativeTolerance, 111, 1e-06);
    Solver.SolverOption(RelativeTolerance, 112, 1e-06);
    Solver.SolverOption(RelativeTolerance, 113, 1e-06);
    Solver.SolverOption(RelativeTolerance, 114, 1e-06);
    Solver.SolverOption(RelativeTolerance, 115, 1e-06);
    Solver.SolverOption(RelativeTolerance, 116, 1e-06);
    Solver.SolverOption(RelativeTolerance, 117, 1e-06);
    Solver.SolverOption(RelativeTolerance, 118, 1e-06);
    Solver.SolverOption(RelativeTolerance, 119, 1e-06);
    Solver.SolverOption(RelativeTolerance, 120, 1e-06);
    Solver.SolverOption(RelativeTolerance, 121, 1e-06);
    Solver.SolverOption(RelativeTolerance, 122, 1e-06);
    Solver.SolverOption(RelativeTolerance, 123, 1e-06);
    Solver.SolverOption(RelativeTolerance, 124, 1e-06);
    Solver.SolverOption(RelativeTolerance, 125, 1e-06);
    Solver.SolverOption(RelativeTolerance, 126, 1e-06);
    Solver.SolverOption(RelativeTolerance, 127, 1e-06);
    Solver.SolverOption(RelativeTolerance, 128, 1e-06);
    Solver.SolverOption(RelativeTolerance, 129, 1e-06);
    Solver.SolverOption(RelativeTolerance, 130, 1e-06);
    Solver.SolverOption(RelativeTolerance, 131, 1e-06);
    Solver.SolverOption(RelativeTolerance, 132, 1e-06);
    Solver.SolverOption(RelativeTolerance, 133, 1e-06);
    Solver.SolverOption(RelativeTolerance, 134, 1e-06);
    Solver.SolverOption(RelativeTolerance, 135, 1e-06);
    Solver.SolverOption(RelativeTolerance, 136, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD293.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

