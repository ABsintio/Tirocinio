#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD376_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 11;
const int NCP  = 1;
const int NSP  = 69;
const int NISP = 0;
const int NE   = 0;
const int NA   = 83;
const int NIA  = 0;
const int NDO  = 40000000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 40000.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_ADPm" << ',';
        DataFile.width(Width); DataFile << "X_Caer" << ',';
        DataFile.width(Width); DataFile << "X_Cam" << ',';
        DataFile.width(Width); DataFile << "X_FBP" << ',';
        DataFile.width(Width); DataFile << "X_G6P" << ',';
        DataFile.width(Width); DataFile << "X_NADHm" << ',';
        DataFile.width(Width); DataFile << "X_Vm" << ',';
        DataFile.width(Width); DataFile << "X_adp" << ',';
        DataFile.width(Width); DataFile << "X_c" << ',';
        DataFile.width(Width); DataFile << "X_delta_psi" << ',';
        DataFile.width(Width); DataFile << "X_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_AMP" << ',';
        DataFile.width(Width); DataFile << "sPAR_Amtot" << ',';
        DataFile.width(Width); DataFile << "sPAR_Cbas" << ',';
        DataFile.width(Width); DataFile << "sPAR_Cmito" << ',';
        DataFile.width(Width); DataFile << "sPAR_Compartment" << ',';
        DataFile.width(Width); DataFile << "sPAR_FRT" << ',';
        DataFile.width(Width); DataFile << "sPAR_JGK" << ',';
        DataFile.width(Width); DataFile << "sPAR_JGPDHbas" << ',';
        DataFile.width(Width); DataFile << "sPAR_NADmtot" << ',';
        DataFile.width(Width); DataFile << "sPAR_VCa" << ',';
        DataFile.width(Width); DataFile << "sPAR_VK" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vc_Ver" << ',';
        DataFile.width(Width); DataFile << "sPAR_VmaxPFK" << ',';
        DataFile.width(Width); DataFile << "sPAR_alpha" << ',';
        DataFile.width(Width); DataFile << "sPAR_atot" << ',';
        DataFile.width(Width); DataFile << "sPAR_bottom1" << ',';
        DataFile.width(Width); DataFile << "sPAR_cm" << ',';
        DataFile.width(Width); DataFile << "sPAR_f13" << ',';
        DataFile.width(Width); DataFile << "sPAR_f23" << ',';
        DataFile.width(Width); DataFile << "sPAR_f41" << ',';
        DataFile.width(Width); DataFile << "sPAR_f42" << ',';
        DataFile.width(Width); DataFile << "sPAR_f43" << ',';
        DataFile.width(Width); DataFile << "sPAR_fcyt" << ',';
        DataFile.width(Width); DataFile << "sPAR_fer" << ',';
        DataFile.width(Width); DataFile << "sPAR_fmito" << ',';
        DataFile.width(Width); DataFile << "sPAR_gCa" << ',';
        DataFile.width(Width); DataFile << "sPAR_gK" << ',';
        DataFile.width(Width); DataFile << "sPAR_gamma" << ',';
        DataFile.width(Width); DataFile << "sPAR_gkATP_" << ',';
        DataFile.width(Width); DataFile << "sPAR_gkCa" << ',';
        DataFile.width(Width); DataFile << "sPAR_k1" << ',';
        DataFile.width(Width); DataFile << "sPAR_k2" << ',';
        DataFile.width(Width); DataFile << "sPAR_k3" << ',';
        DataFile.width(Width); DataFile << "sPAR_k4" << ',';
        DataFile.width(Width); DataFile << "sPAR_kGPDH" << ',';
        DataFile.width(Width); DataFile << "sPAR_kPMCA" << ',';
        DataFile.width(Width); DataFile << "sPAR_kSERCA" << ',';
        DataFile.width(Width); DataFile << "sPAR_kappa" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd" << ',';
        DataFile.width(Width); DataFile << "sPAR_khyd" << ',';
        DataFile.width(Width); DataFile << "sPAR_khydbas" << ',';
        DataFile.width(Width); DataFile << "sPAR_lambda" << ',';
        DataFile.width(Width); DataFile << "sPAR_p1" << ',';
        DataFile.width(Width); DataFile << "sPAR_p10" << ',';
        DataFile.width(Width); DataFile << "sPAR_p11" << ',';
        DataFile.width(Width); DataFile << "sPAR_p13" << ',';
        DataFile.width(Width); DataFile << "sPAR_p14" << ',';
        DataFile.width(Width); DataFile << "sPAR_p15" << ',';
        DataFile.width(Width); DataFile << "sPAR_p16" << ',';
        DataFile.width(Width); DataFile << "sPAR_p17" << ',';
        DataFile.width(Width); DataFile << "sPAR_p18" << ',';
        DataFile.width(Width); DataFile << "sPAR_p19" << ',';
        DataFile.width(Width); DataFile << "sPAR_p2" << ',';
        DataFile.width(Width); DataFile << "sPAR_p20" << ',';
        DataFile.width(Width); DataFile << "sPAR_p21" << ',';
        DataFile.width(Width); DataFile << "sPAR_p22" << ',';
        DataFile.width(Width); DataFile << "sPAR_p23" << ',';
        DataFile.width(Width); DataFile << "sPAR_p24" << ',';
        DataFile.width(Width); DataFile << "sPAR_p3" << ',';
        DataFile.width(Width); DataFile << "sPAR_p4" << ',';
        DataFile.width(Width); DataFile << "sPAR_p5" << ',';
        DataFile.width(Width); DataFile << "sPAR_p6" << ',';
        DataFile.width(Width); DataFile << "sPAR_p7" << ',';
        DataFile.width(Width); DataFile << "sPAR_p8" << ',';
        DataFile.width(Width); DataFile << "sPAR_p9" << ',';
        DataFile.width(Width); DataFile << "sPAR_pleak" << ',';
        DataFile.width(Width); DataFile << "sPAR_tau_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_topa1" << ',';
        DataFile.width(Width); DataFile << "sPAR_weight1" << ',';
        DataFile.width(Width); DataFile << "ACC_ATPm" << ',';
        DataFile.width(Width); DataFile << "ACC_F6P" << ',';
        DataFile.width(Width); DataFile << "ACC_Ica" << ',';
        DataFile.width(Width); DataFile << "ACC_Ik" << ',';
        DataFile.width(Width); DataFile << "ACC_Ikatp" << ',';
        DataFile.width(Width); DataFile << "ACC_Ikca" << ',';
        DataFile.width(Width); DataFile << "ACC_JANT" << ',';
        DataFile.width(Width); DataFile << "ACC_JF1F0" << ',';
        DataFile.width(Width); DataFile << "ACC_JGK_ms" << ',';
        DataFile.width(Width); DataFile << "ACC_JGPDH" << ',';
        DataFile.width(Width); DataFile << "ACC_JHatp" << ',';
        DataFile.width(Width); DataFile << "ACC_JHleak" << ',';
        DataFile.width(Width); DataFile << "ACC_JHres" << ',';
        DataFile.width(Width); DataFile << "ACC_JNaCa" << ',';
        DataFile.width(Width); DataFile << "ACC_JO" << ',';
        DataFile.width(Width); DataFile << "ACC_JPDH" << ',';
        DataFile.width(Width); DataFile << "ACC_JPFK" << ',';
        DataFile.width(Width); DataFile << "ACC_JPFK_ms" << ',';
        DataFile.width(Width); DataFile << "ACC_JSERCA" << ',';
        DataFile.width(Width); DataFile << "ACC_Jer" << ',';
        DataFile.width(Width); DataFile << "ACC_Jhyd" << ',';
        DataFile.width(Width); DataFile << "ACC_Jleak" << ',';
        DataFile.width(Width); DataFile << "ACC_Jmem" << ',';
        DataFile.width(Width); DataFile << "ACC_Jmito" << ',';
        DataFile.width(Width); DataFile << "ACC_Juni" << ',';
        DataFile.width(Width); DataFile << "ACC_NADm" << ',';
        DataFile.width(Width); DataFile << "ACC_RATm" << ',';
        DataFile.width(Width); DataFile << "ACC_adp3m" << ',';
        DataFile.width(Width); DataFile << "ACC_atp" << ',';
        DataFile.width(Width); DataFile << "ACC_atp4m" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom10" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom11" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom12" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom13" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom14" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom15" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom16" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom2" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom3" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom4" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom5" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom6" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom7" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom8" << ',';
        DataFile.width(Width); DataFile << "ACC_bottom9" << ',';
        DataFile.width(Width); DataFile << "ACC_bottomo" << ',';
        DataFile.width(Width); DataFile << "ACC_delta" << ',';
        DataFile.width(Width); DataFile << "ACC_katpo" << ',';
        DataFile.width(Width); DataFile << "ACC_m_infinity" << ',';
        DataFile.width(Width); DataFile << "ACC_mgadp" << ',';
        DataFile.width(Width); DataFile << "ACC_n_infinity" << ',';
        DataFile.width(Width); DataFile << "ACC_topa10" << ',';
        DataFile.width(Width); DataFile << "ACC_topa11" << ',';
        DataFile.width(Width); DataFile << "ACC_topa12" << ',';
        DataFile.width(Width); DataFile << "ACC_topa13" << ',';
        DataFile.width(Width); DataFile << "ACC_topa14" << ',';
        DataFile.width(Width); DataFile << "ACC_topa15" << ',';
        DataFile.width(Width); DataFile << "ACC_topa16" << ',';
        DataFile.width(Width); DataFile << "ACC_topa2" << ',';
        DataFile.width(Width); DataFile << "ACC_topa3" << ',';
        DataFile.width(Width); DataFile << "ACC_topa4" << ',';
        DataFile.width(Width); DataFile << "ACC_topa5" << ',';
        DataFile.width(Width); DataFile << "ACC_topa6" << ',';
        DataFile.width(Width); DataFile << "ACC_topa7" << ',';
        DataFile.width(Width); DataFile << "ACC_topa8" << ',';
        DataFile.width(Width); DataFile << "ACC_topa9" << ',';
        DataFile.width(Width); DataFile << "ACC_topb" << ',';
        DataFile.width(Width); DataFile << "ACC_topo" << ',';
        DataFile.width(Width); DataFile << "ACC_weight10" << ',';
        DataFile.width(Width); DataFile << "ACC_weight11" << ',';
        DataFile.width(Width); DataFile << "ACC_weight12" << ',';
        DataFile.width(Width); DataFile << "ACC_weight13" << ',';
        DataFile.width(Width); DataFile << "ACC_weight14" << ',';
        DataFile.width(Width); DataFile << "ACC_weight15" << ',';
        DataFile.width(Width); DataFile << "ACC_weight16" << ',';
        DataFile.width(Width); DataFile << "ACC_weight2" << ',';
        DataFile.width(Width); DataFile << "ACC_weight3" << ',';
        DataFile.width(Width); DataFile << "ACC_weight4" << ',';
        DataFile.width(Width); DataFile << "ACC_weight5" << ',';
        DataFile.width(Width); DataFile << "ACC_weight6" << ',';
        DataFile.width(Width); DataFile << "ACC_weight7" << ',';
        DataFile.width(Width); DataFile << "ACC_weight8" << ',';
        DataFile.width(Width); DataFile << "ACC_weight9" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 82) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD376.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

