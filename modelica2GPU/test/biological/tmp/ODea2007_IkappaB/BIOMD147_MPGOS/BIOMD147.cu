#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD147_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 24;
const int NCP  = 1;
const int NSP  = 72;
const int NISP = 0;
const int NE   = 0;
const int NA   = 3;
const int NIA  = 0;
const int NDO  = 100000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_IKK" << ',';
        DataFile.width(Width); DataFile << "X_IkBaIKK" << ',';
        DataFile.width(Width); DataFile << "X_IkBaIKKNFkB" << ',';
        DataFile.width(Width); DataFile << "X_IkBaNFkB_cytoplasm" << ',';
        DataFile.width(Width); DataFile << "X_IkBaNFkB_nucleus" << ',';
        DataFile.width(Width); DataFile << "X_IkBa_cytoplasm" << ',';
        DataFile.width(Width); DataFile << "X_IkBa_mRNA" << ',';
        DataFile.width(Width); DataFile << "X_IkBa_nucleus" << ',';
        DataFile.width(Width); DataFile << "X_IkBbIKK" << ',';
        DataFile.width(Width); DataFile << "X_IkBbIKKNFkB" << ',';
        DataFile.width(Width); DataFile << "X_IkBbNFkB_cytoplasm" << ',';
        DataFile.width(Width); DataFile << "X_IkBbNFkB_nucleus" << ',';
        DataFile.width(Width); DataFile << "X_IkBb_cytoplasm" << ',';
        DataFile.width(Width); DataFile << "X_IkBb_mRNA" << ',';
        DataFile.width(Width); DataFile << "X_IkBb_nucleus" << ',';
        DataFile.width(Width); DataFile << "X_IkBeIKK" << ',';
        DataFile.width(Width); DataFile << "X_IkBeIKKNFkB" << ',';
        DataFile.width(Width); DataFile << "X_IkBeNFkB_cytoplasm" << ',';
        DataFile.width(Width); DataFile << "X_IkBeNFkB_nucleus" << ',';
        DataFile.width(Width); DataFile << "X_IkBe_cytoplasm" << ',';
        DataFile.width(Width); DataFile << "X_IkBe_mRNA" << ',';
        DataFile.width(Width); DataFile << "X_IkBe_nucleus" << ',';
        DataFile.width(Width); DataFile << "X_NFkB_cytoplasm" << ',';
        DataFile.width(Width); DataFile << "X_NFkB_nucleus" << ',';
        DataFile.width(Width); DataFile << "sPAR_a1_int_ai" << ',';
        DataFile.width(Width); DataFile << "sPAR_a2_int_bi" << ',';
        DataFile.width(Width); DataFile << "sPAR_a3_int_ei" << ',';
        DataFile.width(Width); DataFile << "sPAR_a4_1_int_an" << ',';
        DataFile.width(Width); DataFile << "sPAR_a4_2_int_an_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_a4_3_int_2ain" << ',';
        DataFile.width(Width); DataFile << "sPAR_a5_1_int_bn" << ',';
        DataFile.width(Width); DataFile << "sPAR_a5_2_int_bn_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_a5_3_int_2bin" << ',';
        DataFile.width(Width); DataFile << "sPAR_a6_1_int_en" << ',';
        DataFile.width(Width); DataFile << "sPAR_a6_2_int_en_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_a6_3_int_2ein" << ',';
        DataFile.width(Width); DataFile << "sPAR_a7_int_2ani" << ',';
        DataFile.width(Width); DataFile << "sPAR_a8_int_2bni" << ',';
        DataFile.width(Width); DataFile << "sPAR_a9_int_2eni" << ',';
        DataFile.width(Width); DataFile << "sPAR_cytoplasm" << ',';
        DataFile.width(Width); DataFile << "sPAR_d1_1_int_ai" << ',';
        DataFile.width(Width); DataFile << "sPAR_d1_2_int_2ani" << ',';
        DataFile.width(Width); DataFile << "sPAR_d2_1_int_bi" << ',';
        DataFile.width(Width); DataFile << "sPAR_d2_2_int_2bni" << ',';
        DataFile.width(Width); DataFile << "sPAR_d3_1_int_ei" << ',';
        DataFile.width(Width); DataFile << "sPAR_d3_2_int_2eni" << ',';
        DataFile.width(Width); DataFile << "sPAR_d4_1_int_an" << ',';
        DataFile.width(Width); DataFile << "sPAR_d4_2_int_an_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_d4_3_int_2ain" << ',';
        DataFile.width(Width); DataFile << "sPAR_d5_1_int_bn" << ',';
        DataFile.width(Width); DataFile << "sPAR_d5_2_int_bn_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_d5_3_int_2bin" << ',';
        DataFile.width(Width); DataFile << "sPAR_d6_1_int_en" << ',';
        DataFile.width(Width); DataFile << "sPAR_d6_2_int_en_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_d6_3_int_2ein" << ',';
        DataFile.width(Width); DataFile << "sPAR_deg1_c_deg_a" << ',';
        DataFile.width(Width); DataFile << "sPAR_deg1_n_deg_a_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_deg2_c_deg_b" << ',';
        DataFile.width(Width); DataFile << "sPAR_deg2_n_deg_b_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_deg3_c_deg_e" << ',';
        DataFile.width(Width); DataFile << "sPAR_deg3_n_deg_e_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_deg4_c_deg_an" << ',';
        DataFile.width(Width); DataFile << "sPAR_deg4_n_deg_an_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_deg5_c_deg_bn" << ',';
        DataFile.width(Width); DataFile << "sPAR_deg5_n_deg_bn_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_deg6_c_deg_en" << ',';
        DataFile.width(Width); DataFile << "sPAR_deg6_n_deg_en_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_k1_1_loc_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_k1_2_loc_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_k2_a_loc_an" << ',';
        DataFile.width(Width); DataFile << "sPAR_k2_b_loc_bn" << ',';
        DataFile.width(Width); DataFile << "sPAR_k2_e_loc_en" << ',';
        DataFile.width(Width); DataFile << "sPAR_k_IKK_deg_IKK_deg" << ',';
        DataFile.width(Width); DataFile << "sPAR_nucleus" << ',';
        DataFile.width(Width); DataFile << "sPAR_r1_deg_ai" << ',';
        DataFile.width(Width); DataFile << "sPAR_r2_deg_bi" << ',';
        DataFile.width(Width); DataFile << "sPAR_r3_deg_ei" << ',';
        DataFile.width(Width); DataFile << "sPAR_r4_deg_ain" << ',';
        DataFile.width(Width); DataFile << "sPAR_r5_deg_bin" << ',';
        DataFile.width(Width); DataFile << "sPAR_r6_deg_ein" << ',';
        DataFile.width(Width); DataFile << "sPAR_tp1a_loc_a" << ',';
        DataFile.width(Width); DataFile << "sPAR_tp1b_loc_b" << ',';
        DataFile.width(Width); DataFile << "sPAR_tp1e_loc_e" << ',';
        DataFile.width(Width); DataFile << "sPAR_tp2a_loc_a" << ',';
        DataFile.width(Width); DataFile << "sPAR_tp2b_loc_b" << ',';
        DataFile.width(Width); DataFile << "sPAR_tp2e_loc_e" << ',';
        DataFile.width(Width); DataFile << "sPAR_tr1a_tsl_a" << ',';
        DataFile.width(Width); DataFile << "sPAR_tr1b_tsl_b" << ',';
        DataFile.width(Width); DataFile << "sPAR_tr1e_tsl_e" << ',';
        DataFile.width(Width); DataFile << "sPAR_tr2a_i_itxn_a" << ',';
        DataFile.width(Width); DataFile << "sPAR_tr2a_txn_a" << ',';
        DataFile.width(Width); DataFile << "sPAR_tr2b_txn_b" << ',';
        DataFile.width(Width); DataFile << "sPAR_tr2e_txn_e" << ',';
        DataFile.width(Width); DataFile << "sPAR_tr3a_mdeg_a" << ',';
        DataFile.width(Width); DataFile << "sPAR_tr3b_mdeg_b" << ',';
        DataFile.width(Width); DataFile << "sPAR_tr3e_mdeg_e" << ',';
        DataFile.width(Width); DataFile << "ACC_Total_IkBalpha" << ',';
        DataFile.width(Width); DataFile << "ACC_Total_IkBbeta" << ',';
        DataFile.width(Width); DataFile << "ACC_Total_IkBeps" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD147.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

