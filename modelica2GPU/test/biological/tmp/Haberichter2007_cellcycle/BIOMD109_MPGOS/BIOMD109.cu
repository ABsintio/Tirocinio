#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD109_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 10000;
const int SD   = 50;
const int NCP  = 1;
const int NSP  = 57;
const int NISP = 0;
const int NE   = 1;
const int NA   = 13;
const int NIA  = 0;
const int NDO  = 1000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 300.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_APCC" << ',';
        DataFile.width(Width); DataFile << "X_APCCYCdk1Y00YCdk1Y01YInt" << ',';
        DataFile.width(Width); DataFile << "X_APCCYCdk1Y10YCdk1Y11YInt" << ',';
        DataFile.width(Width); DataFile << "X_APCCYCdk2Y000YCdk2Y002YInt" << ',';
        DataFile.width(Width); DataFile << "X_APCCYCdk2Y010YCdk2Y012YInt" << ',';
        DataFile.width(Width); DataFile << "X_APCCYCdk2Y100YCdk2Y102YInt" << ',';
        DataFile.width(Width); DataFile << "X_APCCYCdk2Y110YCdk2Y112YInt" << ',';
        DataFile.width(Width); DataFile << "X_APCCYCyclinAYInt" << ',';
        DataFile.width(Width); DataFile << "X_APCCYEmi1" << ',';
        DataFile.width(Width); DataFile << "X_Cdk1Y00" << ',';
        DataFile.width(Width); DataFile << "X_Cdk1Y01" << ',';
        DataFile.width(Width); DataFile << "X_Cdk1Y10" << ',';
        DataFile.width(Width); DataFile << "X_Cdk1Y11" << ',';
        DataFile.width(Width); DataFile << "X_Cdk1Y11YpRbY10YpRbY20YInt" << ',';
        DataFile.width(Width); DataFile << "X_Cdk1Y11YpRbY11YpRbY21YInt" << ',';
        DataFile.width(Width); DataFile << "X_Cdk2Y000" << ',';
        DataFile.width(Width); DataFile << "X_Cdk2Y001" << ',';
        DataFile.width(Width); DataFile << "X_Cdk2Y002" << ',';
        DataFile.width(Width); DataFile << "X_Cdk2Y010" << ',';
        DataFile.width(Width); DataFile << "X_Cdk2Y011" << ',';
        DataFile.width(Width); DataFile << "X_Cdk2Y011YpRbY10YpRbY20YInt" << ',';
        DataFile.width(Width); DataFile << "X_Cdk2Y011YpRbY11YpRbY21YInt" << ',';
        DataFile.width(Width); DataFile << "X_Cdk2Y012" << ',';
        DataFile.width(Width); DataFile << "X_Cdk2Y012YpRbY10YpRbY20YInt" << ',';
        DataFile.width(Width); DataFile << "X_Cdk2Y012YpRbY11YpRbY21YInt" << ',';
        DataFile.width(Width); DataFile << "X_Cdk2Y100" << ',';
        DataFile.width(Width); DataFile << "X_Cdk2Y101" << ',';
        DataFile.width(Width); DataFile << "X_Cdk2Y102" << ',';
        DataFile.width(Width); DataFile << "X_Cdk2Y110" << ',';
        DataFile.width(Width); DataFile << "X_Cdk2Y111" << ',';
        DataFile.width(Width); DataFile << "X_Cdk2Y112" << ',';
        DataFile.width(Width); DataFile << "X_Cdk4Y00" << ',';
        DataFile.width(Width); DataFile << "X_Cdk4Y01" << ',';
        DataFile.width(Width); DataFile << "X_Cdk4Y01YpRbY00YpRbY10YInt" << ',';
        DataFile.width(Width); DataFile << "X_Cdk4Y01YpRbY01YpRbY11YInt" << ',';
        DataFile.width(Width); DataFile << "X_Cdk4Y10" << ',';
        DataFile.width(Width); DataFile << "X_Cdk4Y11" << ',';
        DataFile.width(Width); DataFile << "X_CyclinA" << ',';
        DataFile.width(Width); DataFile << "X_CyclinD" << ',';
        DataFile.width(Width); DataFile << "X_CyclinE" << ',';
        DataFile.width(Width); DataFile << "X_E2F" << ',';
        DataFile.width(Width); DataFile << "X_Emi1" << ',';
        DataFile.width(Width); DataFile << "X_kYact" << ',';
        DataFile.width(Width); DataFile << "X_p27" << ',';
        DataFile.width(Width); DataFile << "X_pRbY00" << ',';
        DataFile.width(Width); DataFile << "X_pRbY01" << ',';
        DataFile.width(Width); DataFile << "X_pRbY10" << ',';
        DataFile.width(Width); DataFile << "X_pRbY11" << ',';
        DataFile.width(Width); DataFile << "X_pRbY20" << ',';
        DataFile.width(Width); DataFile << "X_pRbY21" << ',';
        DataFile.width(Width); DataFile << "sPAR_X" << ',';
        DataFile.width(Width); DataFile << "sPAR_k1Yact" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbYA1YYpRb" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbYA2YYpRb" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbYAPCCYYCyclinA" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbYCyclinAYYCdk1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbYCyclinAYYCdk2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbYCyclinDYYCdk4" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbYCyclinEYYCdk2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbYD4YYpRb" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbYE2FYYpRb" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbYE2YYpRb" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbYEmi1YYAPCC" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbYp27YYCdk2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kbYp27YYCdk4" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd0YE2F" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd1Yp27" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdYCyclinA" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdYCyclinD" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdYCyclinE" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdYE2F" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdYEmi1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdYp27" << ',';
        DataFile.width(Width); DataFile << "sPAR_ks0YCyclinA" << ',';
        DataFile.width(Width); DataFile << "sPAR_ks0YCyclinE" << ',';
        DataFile.width(Width); DataFile << "sPAR_ks0YE2F" << ',';
        DataFile.width(Width); DataFile << "sPAR_ks0YEmi1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ks1YCyclinA" << ',';
        DataFile.width(Width); DataFile << "sPAR_ks1YCyclinE" << ',';
        DataFile.width(Width); DataFile << "sPAR_ks1YE2F" << ',';
        DataFile.width(Width); DataFile << "sPAR_ks1YEmi1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksMYCyclinA" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksMYCyclinE" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksMYE2F" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksMYEmi1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksYCyclinD" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksYp27" << ',';
        DataFile.width(Width); DataFile << "sPAR_ktYpRbYYDephos" << ',';
        DataFile.width(Width); DataFile << "sPAR_kuYA1YYpRb" << ',';
        DataFile.width(Width); DataFile << "sPAR_kuYA2YYpRb" << ',';
        DataFile.width(Width); DataFile << "sPAR_kuYAPCCYYCyclinA" << ',';
        DataFile.width(Width); DataFile << "sPAR_kuYCyclinAYYCdk1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kuYCyclinAYYCdk2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kuYCyclinDYYCdk4" << ',';
        DataFile.width(Width); DataFile << "sPAR_kuYCyclinEYYCdk2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kuYD4YYpRb" << ',';
        DataFile.width(Width); DataFile << "sPAR_kuYE2FYYpRb" << ',';
        DataFile.width(Width); DataFile << "sPAR_kuYE2YYpRb" << ',';
        DataFile.width(Width); DataFile << "sPAR_kuYEmi1YYAPCC" << ',';
        DataFile.width(Width); DataFile << "sPAR_kuYp27YYCdk2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kuYp27YYCdk4" << ',';
        DataFile.width(Width); DataFile << "sPAR_kudYAPCCYYCyclinA" << ',';
        DataFile.width(Width); DataFile << "sPAR_kupYA1YYpRb" << ',';
        DataFile.width(Width); DataFile << "sPAR_kupYA2YYpRb" << ',';
        DataFile.width(Width); DataFile << "sPAR_kupYD4YYpRb" << ',';
        DataFile.width(Width); DataFile << "sPAR_kupYE2YYpRb" << ',';
        DataFile.width(Width); DataFile << "sPAR_timeYModifier" << ',';
        DataFile.width(Width); DataFile << "ACC_activeYCdk2" << ',';
        DataFile.width(Width); DataFile << "ACC_hyperphosphorylatedYpRb" << ',';
        DataFile.width(Width); DataFile << "ACC_hypophosphorylatedYpRb" << ',';
        DataFile.width(Width); DataFile << "ACC_ksYCyclinA" << ',';
        DataFile.width(Width); DataFile << "ACC_ksYCyclinE" << ',';
        DataFile.width(Width); DataFile << "ACC_ksYE2F" << ',';
        DataFile.width(Width); DataFile << "ACC_ksYEmi1" << ',';
        DataFile.width(Width); DataFile << "ACC_totalYCyclinYA" << ',';
        DataFile.width(Width); DataFile << "ACC_totalYCyclinYD" << ',';
        DataFile.width(Width); DataFile << "ACC_totalYCyclinYE" << ',';
        DataFile.width(Width); DataFile << "ACC_totalYEmi1" << ',';
        DataFile.width(Width); DataFile << "ACC_totalYp27" << ',';
        DataFile.width(Width); DataFile << "ACC_$whenCondition1" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);


    Solver.SolverOption(EventDirection, 0, 0);

    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 42, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 43, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 44, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 45, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 46, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 47, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 48, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 49, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
    Solver.SolverOption(RelativeTolerance, 42, 1e-06);
    Solver.SolverOption(RelativeTolerance, 43, 1e-06);
    Solver.SolverOption(RelativeTolerance, 44, 1e-06);
    Solver.SolverOption(RelativeTolerance, 45, 1e-06);
    Solver.SolverOption(RelativeTolerance, 46, 1e-06);
    Solver.SolverOption(RelativeTolerance, 47, 1e-06);
    Solver.SolverOption(RelativeTolerance, 48, 1e-06);
    Solver.SolverOption(RelativeTolerance, 49, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD109.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

