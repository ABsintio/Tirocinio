#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#include "BIOMD406_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 50;
const int NCP  = 1;
const int NSP  = 108;
const int NISP = 0;
const int NE   = 0;
const int NA   = 10;
const int NIA  = 0;
const int NDO  = 5000000.0;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 500.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_kmik" << ',';
        DataFile.width(Width); DataFile << "X_s130" << ',';
        DataFile.width(Width); DataFile << "X_s137" << ',';
        DataFile.width(Width); DataFile << "X_s149" << ',';
        DataFile.width(Width); DataFile << "X_s153" << ',';
        DataFile.width(Width); DataFile << "X_s157" << ',';
        DataFile.width(Width); DataFile << "X_s161" << ',';
        DataFile.width(Width); DataFile << "X_s166" << ',';
        DataFile.width(Width); DataFile << "X_s4" << ',';
        DataFile.width(Width); DataFile << "X_s46" << ',';
        DataFile.width(Width); DataFile << "X_s47" << ',';
        DataFile.width(Width); DataFile << "X_s48" << ',';
        DataFile.width(Width); DataFile << "X_s49" << ',';
        DataFile.width(Width); DataFile << "X_s50" << ',';
        DataFile.width(Width); DataFile << "X_s51" << ',';
        DataFile.width(Width); DataFile << "X_s52" << ',';
        DataFile.width(Width); DataFile << "X_s55" << ',';
        DataFile.width(Width); DataFile << "X_s56" << ',';
        DataFile.width(Width); DataFile << "X_s57" << ',';
        DataFile.width(Width); DataFile << "X_s60" << ',';
        DataFile.width(Width); DataFile << "X_s61" << ',';
        DataFile.width(Width); DataFile << "X_s63" << ',';
        DataFile.width(Width); DataFile << "X_s64" << ',';
        DataFile.width(Width); DataFile << "X_s65" << ',';
        DataFile.width(Width); DataFile << "X_s66" << ',';
        DataFile.width(Width); DataFile << "X_s67" << ',';
        DataFile.width(Width); DataFile << "X_s70" << ',';
        DataFile.width(Width); DataFile << "X_s71" << ',';
        DataFile.width(Width); DataFile << "X_s72" << ',';
        DataFile.width(Width); DataFile << "X_s73" << ',';
        DataFile.width(Width); DataFile << "X_s74" << ',';
        DataFile.width(Width); DataFile << "X_s75" << ',';
        DataFile.width(Width); DataFile << "X_s76" << ',';
        DataFile.width(Width); DataFile << "X_s77" << ',';
        DataFile.width(Width); DataFile << "X_s78" << ',';
        DataFile.width(Width); DataFile << "X_s79" << ',';
        DataFile.width(Width); DataFile << "X_s80" << ',';
        DataFile.width(Width); DataFile << "X_s81" << ',';
        DataFile.width(Width); DataFile << "X_s82" << ',';
        DataFile.width(Width); DataFile << "X_s83" << ',';
        DataFile.width(Width); DataFile << "X_s84" << ',';
        DataFile.width(Width); DataFile << "X_s85" << ',';
        DataFile.width(Width); DataFile << "X_s88" << ',';
        DataFile.width(Width); DataFile << "X_s89" << ',';
        DataFile.width(Width); DataFile << "X_s9" << ',';
        DataFile.width(Width); DataFile << "X_s90" << ',';
        DataFile.width(Width); DataFile << "X_s91" << ',';
        DataFile.width(Width); DataFile << "X_s92" << ',';
        DataFile.width(Width); DataFile << "X_s93" << ',';
        DataFile.width(Width); DataFile << "X_s94" << ',';
        DataFile.width(Width); DataFile << "sPAR_Cdc10T" << ',';
        DataFile.width(Width); DataFile << "sPAR_Cdc25T" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ja25" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jac10" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jaie" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jaslp" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jasrw" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jawee" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ji25" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jic10" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jiie" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jipre" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jislp" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jisrw" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jiwee" << ',';
        DataFile.width(Width); DataFile << "sPAR_Puc1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Rad3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Slp1T" << ',';
        DataFile.width(Width); DataFile << "sPAR_Srw1T" << ',';
        DataFile.width(Width); DataFile << "sPAR_Va25_dash2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vamik" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vamik_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vawee_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vawee_dash2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vi25" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vi25_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vi25_dash2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vimik" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vimik_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vimik_dash2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vimik_dash3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Viwee_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_Viwee_dash2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Wee1T" << ',';
        DataFile.width(Width); DataFile << "sPAR_beta" << ',';
        DataFile.width(Width); DataFile << "sPAR_default" << ',';
        DataFile.width(Width); DataFile << "sPAR_k" << ',';
        DataFile.width(Width); DataFile << "sPAR_k255" << ',';
        DataFile.width(Width); DataFile << "sPAR_k25_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_k25_dash2" << ',';
        DataFile.width(Width); DataFile << "sPAR_k_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kac10" << ',';
        DataFile.width(Width); DataFile << "sPAR_kaie" << ',';
        DataFile.width(Width); DataFile << "sPAR_kaie_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kaslp" << ',';
        DataFile.width(Width); DataFile << "sPAR_kasrw" << ',';
        DataFile.width(Width); DataFile << "sPAR_kasrw_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdc18" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdc18c13" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdc18cig" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdc18cig_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdci1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdci1_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdci1_dash2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdcig" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdcig_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdcyc" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdcycslp" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdcycslp_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdcycsrw" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdcycsrw_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdflp" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdrum" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdrumc13" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdrumc13_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdrumci1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdrumcig" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdrumcig_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdrumpuc" << ',';
        DataFile.width(Width); DataFile << "sPAR_kic10" << ',';
        DataFile.width(Width); DataFile << "sPAR_kic10_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kiie" << ',';
        DataFile.width(Width); DataFile << "sPAR_kini_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kini_dash2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kini_dash3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kipre" << ',';
        DataFile.width(Width); DataFile << "sPAR_kipre_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kislp" << ',';
        DataFile.width(Width); DataFile << "sPAR_kisrw" << ',';
        DataFile.width(Width); DataFile << "sPAR_kisrw_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kisrw_dash2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kisrw_dash3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kisrw_dash4" << ',';
        DataFile.width(Width); DataFile << "sPAR_kmik_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kmik_dash2" << ',';
        DataFile.width(Width); DataFile << "sPAR_ko18" << ',';
        DataFile.width(Width); DataFile << "sPAR_ko18r" << ',';
        DataFile.width(Width); DataFile << "sPAR_kori" << ',';
        DataFile.width(Width); DataFile << "sPAR_kpyp" << ',';
        DataFile.width(Width); DataFile << "sPAR_kpyp2" << ',';
        DataFile.width(Width); DataFile << "sPAR_krepl" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksc18" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksc18_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksci1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kscig" << ',';
        DataFile.width(Width); DataFile << "sPAR_kscig_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kscyc" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksflp" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksflp_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksrum" << ',';
        DataFile.width(Width); DataFile << "sPAR_kwee_dash" << ',';
        DataFile.width(Width); DataFile << "sPAR_kwee_dash2" << ',';
        DataFile.width(Width); DataFile << "sPAR_lcm" << ',';
        DataFile.width(Width); DataFile << "sPAR_lcp" << ',';
        DataFile.width(Width); DataFile << "sPAR_lm" << ',';
        DataFile.width(Width); DataFile << "sPAR_lp" << ',';
        DataFile.width(Width); DataFile << "sPAR_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_oriT" << ',';
        DataFile.width(Width); DataFile << "ACC_BB" << ',';
        DataFile.width(Width); DataFile << "ACC_Cdc18" << ',';
        DataFile.width(Width); DataFile << "ACC_UDNA" << ',';
        DataFile.width(Width); DataFile << "ACC_Vdc18" << ',';
        DataFile.width(Width); DataFile << "ACC_Vdcyc" << ',';
        DataFile.width(Width); DataFile << "ACC_Vdrum" << ',';
        DataFile.width(Width); DataFile << "ACC_chrom" << ',';
        DataFile.width(Width); DataFile << "ACC_k25" << ',';
        DataFile.width(Width); DataFile << "ACC_kwee" << ',';
        DataFile.width(Width); DataFile << "ACC_preRC" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 83) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 84) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 85) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 86) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 87) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 88) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 89) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 90) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 91) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 92) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 93) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 94) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 95) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 96) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 97) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 98) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 99) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 100) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 101) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 102) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 103) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 104) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 105) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 106) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 107) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 42, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 43, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 44, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 45, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 46, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 47, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 48, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 49, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
    Solver.SolverOption(RelativeTolerance, 42, 1e-06);
    Solver.SolverOption(RelativeTolerance, 43, 1e-06);
    Solver.SolverOption(RelativeTolerance, 44, 1e-06);
    Solver.SolverOption(RelativeTolerance, 45, 1e-06);
    Solver.SolverOption(RelativeTolerance, 46, 1e-06);
    Solver.SolverOption(RelativeTolerance, 47, 1e-06);
    Solver.SolverOption(RelativeTolerance, 48, 1e-06);
    Solver.SolverOption(RelativeTolerance, 49, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD406.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

