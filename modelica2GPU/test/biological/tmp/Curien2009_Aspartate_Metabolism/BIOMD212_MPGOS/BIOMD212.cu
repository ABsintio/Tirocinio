#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD212_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 31;
const int NCP  = 1;
const int NSP  = 65;
const int NISP = 0;
const int NE   = 0;
const int NA   = 0;
const int NIA  = 0;
const int NDO  = 1000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_AK1" << ',';
        DataFile.width(Width); DataFile << "X_AK2" << ',';
        DataFile.width(Width); DataFile << "X_AKHSDHI" << ',';
        DataFile.width(Width); DataFile << "X_AKHSDHII" << ',';
        DataFile.width(Width); DataFile << "X_ASA" << ',';
        DataFile.width(Width); DataFile << "X_ASADH" << ',';
        DataFile.width(Width); DataFile << "X_AdoMet" << ',';
        DataFile.width(Width); DataFile << "X_Asp" << ',';
        DataFile.width(Width); DataFile << "X_AspP" << ',';
        DataFile.width(Width); DataFile << "X_CGS" << ',';
        DataFile.width(Width); DataFile << "X_Cys" << ',';
        DataFile.width(Width); DataFile << "X_Cysta" << ',';
        DataFile.width(Width); DataFile << "X_DHDPS1" << ',';
        DataFile.width(Width); DataFile << "X_DHDPS2" << ',';
        DataFile.width(Width); DataFile << "X_Gly" << ',';
        DataFile.width(Width); DataFile << "X_HSK" << ',';
        DataFile.width(Width); DataFile << "X_Hser" << ',';
        DataFile.width(Width); DataFile << "X_Ile" << ',';
        DataFile.width(Width); DataFile << "X_IleTRNA" << ',';
        DataFile.width(Width); DataFile << "X_LKR" << ',';
        DataFile.width(Width); DataFile << "X_Lys" << ',';
        DataFile.width(Width); DataFile << "X_LysTRNA" << ',';
        DataFile.width(Width); DataFile << "X_PHser" << ',';
        DataFile.width(Width); DataFile << "X_Phosphate" << ',';
        DataFile.width(Width); DataFile << "X_Sacc" << ',';
        DataFile.width(Width); DataFile << "X_TD" << ',';
        DataFile.width(Width); DataFile << "X_THA" << ',';
        DataFile.width(Width); DataFile << "X_TS1" << ',';
        DataFile.width(Width); DataFile << "X_Thr" << ',';
        DataFile.width(Width); DataFile << "X_ThrTRNA" << ',';
        DataFile.width(Width); DataFile << "X_Val" << ',';
        DataFile.width(Width); DataFile << "sPAR_AK1_AdoMet_Ka_app_exp_Vak1" << ',';
        DataFile.width(Width); DataFile << "sPAR_AK1_Lys_Ki_app_exp_Vak1" << ',';
        DataFile.width(Width); DataFile << "sPAR_AK1_kforward_app_exp_Vak1" << ',';
        DataFile.width(Width); DataFile << "sPAR_AK1_kreverse_app_exp_Vak1" << ',';
        DataFile.width(Width); DataFile << "sPAR_AK1_nH_exp_Vak1" << ',';
        DataFile.width(Width); DataFile << "sPAR_AK2_Lys_Ki_app_exp_Vak2" << ',';
        DataFile.width(Width); DataFile << "sPAR_AK2_kforward_app_exp_Vak2" << ',';
        DataFile.width(Width); DataFile << "sPAR_AK2_kreverse_app_exp_Vak2" << ',';
        DataFile.width(Width); DataFile << "sPAR_AK2_nH_exp_Vak2" << ',';
        DataFile.width(Width); DataFile << "sPAR_AKII_Thr_Ki_app_exp_VakII" << ',';
        DataFile.width(Width); DataFile << "sPAR_AKII_kforward_app_exp_VakII" << ',';
        DataFile.width(Width); DataFile << "sPAR_AKII_kreverse_app_exp_VakII" << ',';
        DataFile.width(Width); DataFile << "sPAR_AKII_nH_exp_VakII" << ',';
        DataFile.width(Width); DataFile << "sPAR_AKI_Thr_Ki_app_exp_VakI" << ',';
        DataFile.width(Width); DataFile << "sPAR_AKI_kforward_app_exp_VakI" << ',';
        DataFile.width(Width); DataFile << "sPAR_AKI_kreverse_app_exp_VakI" << ',';
        DataFile.width(Width); DataFile << "sPAR_AKI_nH_exp_VakI" << ',';
        DataFile.width(Width); DataFile << "sPAR_ASADH_kforward_app_exp_Vasadh" << ',';
        DataFile.width(Width); DataFile << "sPAR_ASADH_kreverse_app_exp_Vasadh" << ',';
        DataFile.width(Width); DataFile << "sPAR_CGS_Cys_Km_exp_Vcgs" << ',';
        DataFile.width(Width); DataFile << "sPAR_CGS_Phosphate_Ki_exp_Vcgs" << ',';
        DataFile.width(Width); DataFile << "sPAR_CGS_Phser_Km_exp_Vcgs" << ',';
        DataFile.width(Width); DataFile << "sPAR_CGS_kcat_exp_Vcgs" << ',';
        DataFile.width(Width); DataFile << "sPAR_DHDPS1_Lys_Ki_app_exp_Vdhdps1" << ',';
        DataFile.width(Width); DataFile << "sPAR_DHDPS1_k_app_exp_Vdhdps1" << ',';
        DataFile.width(Width); DataFile << "sPAR_DHDPS1_nH_exp_Vdhdps1" << ',';
        DataFile.width(Width); DataFile << "sPAR_DHDPS2_Lys_Ki_app_exp_Vdhdps2" << ',';
        DataFile.width(Width); DataFile << "sPAR_DHDPS2_k_app_exp_Vdhdps2" << ',';
        DataFile.width(Width); DataFile << "sPAR_DHDPS2_nH_exp_Vdhdps2" << ',';
        DataFile.width(Width); DataFile << "sPAR_HSDHII_Thr_Ki_app_exp_Vhsdh2" << ',';
        DataFile.width(Width); DataFile << "sPAR_HSDHII_Thr_relative_inhibition_app_exp_Vhsdh2" << ',';
        DataFile.width(Width); DataFile << "sPAR_HSDHII_Thr_relative_residual_activity_app_exp_Vhsdh2" << ',';
        DataFile.width(Width); DataFile << "sPAR_HSDHII_kforward_app_exp_Vhsdh2" << ',';
        DataFile.width(Width); DataFile << "sPAR_HSDHI_Thr_Ki_app_exp_Vhsdh1" << ',';
        DataFile.width(Width); DataFile << "sPAR_HSDHI_Thr_relative_inhibition_app_exp_Vhsdh1" << ',';
        DataFile.width(Width); DataFile << "sPAR_HSDHI_Thr_relative_residual_activity_app_exp_Vhsdh1" << ',';
        DataFile.width(Width); DataFile << "sPAR_HSDHI_kforward_app_exp_Vhsdh1" << ',';
        DataFile.width(Width); DataFile << "sPAR_HSK_Hser_app_exp_Vhsk" << ',';
        DataFile.width(Width); DataFile << "sPAR_HSK_kcat_app_exp_Vhsk" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ile_tRNAS_Ile_Km_VileTRNA" << ',';
        DataFile.width(Width); DataFile << "sPAR_LKR_Lys_Km_exp_VlysKR" << ',';
        DataFile.width(Width); DataFile << "sPAR_LKR_kcat_exp_VlysKR" << ',';
        DataFile.width(Width); DataFile << "sPAR_Lys_tRNAS_Lys_Km_VlysTRNA" << ',';
        DataFile.width(Width); DataFile << "sPAR_TD_Ile_Ki_no_Val_app_exp_Vtd" << ',';
        DataFile.width(Width); DataFile << "sPAR_TD_Val_Ka1_app_exp_Vtd" << ',';
        DataFile.width(Width); DataFile << "sPAR_TD_Val_Ka2_app_exp_Vtd" << ',';
        DataFile.width(Width); DataFile << "sPAR_TD_k_app_exp_Vtd" << ',';
        DataFile.width(Width); DataFile << "sPAR_TD_nH_app_exp_Vtd" << ',';
        DataFile.width(Width); DataFile << "sPAR_THA_Thr_Km_exp_Vtha" << ',';
        DataFile.width(Width); DataFile << "sPAR_THA_kcat_exp_Vtha" << ',';
        DataFile.width(Width); DataFile << "sPAR_TS1_AdoMEt_Km_no_AdoMet_exp_Vts1" << ',';
        DataFile.width(Width); DataFile << "sPAR_TS1_AdoMet_Ka1_exp_Vts1" << ',';
        DataFile.width(Width); DataFile << "sPAR_TS1_AdoMet_Ka2_exp_Vts1" << ',';
        DataFile.width(Width); DataFile << "sPAR_TS1_AdoMet_Ka3_exp_Vts1" << ',';
        DataFile.width(Width); DataFile << "sPAR_TS1_AdoMet_Ka4_exp_Vts1" << ',';
        DataFile.width(Width); DataFile << "sPAR_TS1_AdoMet_kcatmax_exp_Vts1" << ',';
        DataFile.width(Width); DataFile << "sPAR_TS1_Phosphate_Ki_exp_Vts1" << ',';
        DataFile.width(Width); DataFile << "sPAR_TS1_kcatmin_exp_Vts1" << ',';
        DataFile.width(Width); DataFile << "sPAR_TS1_nH_exp_Vts1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Thr_tRNAS_Thr_Km_VthrTRNA" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_AA_RS" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_Ile_RS" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_Lys_RS" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_Thr_RS" << ',';
        DataFile.width(Width); DataFile << "sPAR_c1" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 64) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD212.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

