#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#include "BIOMD370_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 10000;
const int SD   = 26;
const int NCP  = 1;
const int NSP  = 94;
const int NISP = 0;
const int NE   = 0;
const int NA   = 18;
const int NIA  = 0;
const int NDO  = 100;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 500.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_Cdc14n_1" << ',';
        DataFile.width(Width); DataFile << "X_Cdc15_1" << ',';
        DataFile.width(Width); DataFile << "X_Cdc20_1" << ',';
        DataFile.width(Width); DataFile << "X_Cdh1_1" << ',';
        DataFile.width(Width); DataFile << "X_Clb2T_1" << ',';
        DataFile.width(Width); DataFile << "X_Clb5T_1" << ',';
        DataFile.width(Width); DataFile << "X_Cln_1" << ',';
        DataFile.width(Width); DataFile << "X_Esp1T_1" << ',';
        DataFile.width(Width); DataFile << "X_Esp1b_1" << ',';
        DataFile.width(Width); DataFile << "X_MBF_1" << ',';
        DataFile.width(Width); DataFile << "X_MEN_1" << ',';
        DataFile.width(Width); DataFile << "X_Mcm_1" << ',';
        DataFile.width(Width); DataFile << "X_Net1dep_1" << ',';
        DataFile.width(Width); DataFile << "X_Net1p_1" << ',';
        DataFile.width(Width); DataFile << "X_Net1pp_1" << ',';
        DataFile.width(Width); DataFile << "X_Pds1T_1" << ',';
        DataFile.width(Width); DataFile << "X_PoloT_1" << ',';
        DataFile.width(Width); DataFile << "X_Polo_1" << ',';
        DataFile.width(Width); DataFile << "X_RENT_1" << ',';
        DataFile.width(Width); DataFile << "X_RENTp_1" << ',';
        DataFile.width(Width); DataFile << "X_Sic1T_1" << ',';
        DataFile.width(Width); DataFile << "X_Swi5_1" << ',';
        DataFile.width(Width); DataFile << "X_Tem1_1" << ',';
        DataFile.width(Width); DataFile << "X_Trim2_1" << ',';
        DataFile.width(Width); DataFile << "X_Trim5_1" << ',';
        DataFile.width(Width); DataFile << "X_lamen_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Cdc14T_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Clb2nd_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jcdc15_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jcdh_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jmbf_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jmcm_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jnet_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jpolo_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jswi_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Jtem1_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Net1T_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_PPT_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Swi5T_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_cell_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kac15_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kac15_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kambf_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kapolo_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kapolo_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kasic2_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kasic5_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kaswi_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kaswi_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_katem_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_katem_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd20_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd20_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdcdh_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdcdh_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdclb2_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdclb2_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdclb2_3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdclb5_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdclb5_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdcln_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdesp_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdmcm_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdpds_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdpds_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdpolo_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdpolo_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdsic2_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdsic5_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdsic_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdsic_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdsic_3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kdsic_4" << ',';
        DataFile.width(Width); DataFile << "sPAR_kexp_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kexp_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_ki_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kic15_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kic15_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kimbf_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kimbf_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kimbf_3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kimp_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kipolo_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kiswi_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kiswi_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kiswi_3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kitem_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kitem_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kitem_3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kp_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kp_3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kp_4" << ',';
        DataFile.width(Width); DataFile << "sPAR_kpcdh_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kpcdh_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kpcdh_3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kpp_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ks20_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ks20_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksclb2_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksclb2_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksclb5_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksclb5_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kscln_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kscln_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksesp_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksmcm_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksmcm_3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kspds_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kspds_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kspolo_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kspolo_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kssic_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kssic_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_lanet_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_lapds_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ldmen_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ldnet_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_ldpds_1" << ',';
        DataFile.width(Width); DataFile << "ACC_Cdc14c_1" << ',';
        DataFile.width(Width); DataFile << "ACC_Clb2_2" << ',';
        DataFile.width(Width); DataFile << "ACC_Clb5_1" << ',';
        DataFile.width(Width); DataFile << "ACC_Esp1_1" << ',';
        DataFile.width(Width); DataFile << "ACC_Net1_2" << ',';
        DataFile.width(Width); DataFile << "ACC_PP_1" << ',';
        DataFile.width(Width); DataFile << "ACC_Pds1_1" << ',';
        DataFile.width(Width); DataFile << "ACC_Sic1_1" << ',';
        DataFile.width(Width); DataFile << "ACC_V2_1" << ',';
        DataFile.width(Width); DataFile << "ACC_V6_1" << ',';
        DataFile.width(Width); DataFile << "ACC_Vacdh_1" << ',';
        DataFile.width(Width); DataFile << "ACC_Vaswi_1" << ',';
        DataFile.width(Width); DataFile << "ACC_Vd_1" << ',';
        DataFile.width(Width); DataFile << "ACC_Vdsic_1" << ',';
        DataFile.width(Width); DataFile << "ACC_Vexp_1" << ',';
        DataFile.width(Width); DataFile << "ACC_Vicdh_1" << ',';
        DataFile.width(Width); DataFile << "ACC_Viswi_1" << ',';
        DataFile.width(Width); DataFile << "ACC_Vp_1" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 83) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 84) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 85) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 86) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 87) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 88) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 89) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 90) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 91) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 92) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 93) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD370.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

