#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD219_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 13;
const int NCP  = 1;
const int NSP  = 71;
const int NISP = 0;
const int NE   = 0;
const int NA   = 0;
const int NIA  = 0;
const int NDO  = 100000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_aca" << ',';
        DataFile.width(Width); DataFile << "X_akg" << ',';
        DataFile.width(Width); DataFile << "X_biosyn" << ',';
        DataFile.width(Width); DataFile << "X_cit" << ',';
        DataFile.width(Width); DataFile << "X_coa" << ',';
        DataFile.width(Width); DataFile << "X_fa" << ',';
        DataFile.width(Width); DataFile << "X_gly" << ',';
        DataFile.width(Width); DataFile << "X_icit" << ',';
        DataFile.width(Width); DataFile << "X_mal" << ',';
        DataFile.width(Width); DataFile << "X_oaa" << ',';
        DataFile.width(Width); DataFile << "X_sca" << ',';
        DataFile.width(Width); DataFile << "X_ssa" << ',';
        DataFile.width(Width); DataFile << "X_suc" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kaca_cs_CS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kaca_ms_MS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kakg_icd1_ICD1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kakg_icd1_SYN" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kakg_icd2_ICD2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kakg_icd2_SYN" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kakg_kdh_KDH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kakg_kgd_KGD" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kcit_acn_ACN" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kcit_cs_CS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kcoa_cs_CS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kcoa_ms_MS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kfa_fum_FUM" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kfa_sdh_SDH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgly_icl1_ICL1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgly_icl2_ICL2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgly_ms_MS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kicit_acn_ACN" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kicit_icd1_ICD1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kicit_icd1_SYN" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kicit_icd2_ICD2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kicit_icd2_SYN" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kicit_icl1_ICL1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kicit_icl2_ICL2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmal_fum_FUM" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmal_mdh_MDH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmal_ms_MS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Koaa_cs_CS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Koaa_mdh_MDH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ksca_kdh_KDH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ksca_scas_ScAS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kssa_kgd_KGD" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kssa_ssadh_SSADH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ksuc_icl1_ICL1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ksuc_icl2_ICL2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ksuc_scas_ScAS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ksuc_sdh_SDH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ksuc_ssadh_SSADH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_acn_ACN" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_cs_CS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_fum_FUM" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_icd1_ICD1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_icd1_SYN" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_icd2_ICD2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_icd2_SYN" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_icl1_ICL1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_icl2_ICL2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_kdh_KDH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_kgd_KGD" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_mdh_MDH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_ms_MS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_scas_ScAS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_sdh_SDH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_ssadh_SSADH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_acn_ACN" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_cs_CS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_fum_FUM" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_icd1_ICD1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_icd1_SYN" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_icd2_ICD2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_icd2_SYN" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_icl1_ICL1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_icl2_ICL2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_kdh_KDH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_kgd_KGD" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_mdh_MDH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_ms_MS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_scas_ScAS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_sdh_SDH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_ssadh_SSADH" << ',';
        DataFile.width(Width); DataFile << "sPAR_cell" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 70) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD219.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

