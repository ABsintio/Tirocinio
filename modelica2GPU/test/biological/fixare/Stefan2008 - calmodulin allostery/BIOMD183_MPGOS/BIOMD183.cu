#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD183_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 67;
const int NCP  = 1;
const int NSP  = 16;
const int NISP = 0;
const int NE   = 0;
const int NA   = 46;
const int NIA  = 0;
const int NDO  = 100000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_species_0" << ',';
        DataFile.width(Width); DataFile << "X_species_1" << ',';
        DataFile.width(Width); DataFile << "X_species_10" << ',';
        DataFile.width(Width); DataFile << "X_species_11" << ',';
        DataFile.width(Width); DataFile << "X_species_12" << ',';
        DataFile.width(Width); DataFile << "X_species_13" << ',';
        DataFile.width(Width); DataFile << "X_species_14" << ',';
        DataFile.width(Width); DataFile << "X_species_15" << ',';
        DataFile.width(Width); DataFile << "X_species_16" << ',';
        DataFile.width(Width); DataFile << "X_species_17" << ',';
        DataFile.width(Width); DataFile << "X_species_18" << ',';
        DataFile.width(Width); DataFile << "X_species_19" << ',';
        DataFile.width(Width); DataFile << "X_species_2" << ',';
        DataFile.width(Width); DataFile << "X_species_20" << ',';
        DataFile.width(Width); DataFile << "X_species_21" << ',';
        DataFile.width(Width); DataFile << "X_species_22" << ',';
        DataFile.width(Width); DataFile << "X_species_23" << ',';
        DataFile.width(Width); DataFile << "X_species_24" << ',';
        DataFile.width(Width); DataFile << "X_species_25" << ',';
        DataFile.width(Width); DataFile << "X_species_26" << ',';
        DataFile.width(Width); DataFile << "X_species_27" << ',';
        DataFile.width(Width); DataFile << "X_species_28" << ',';
        DataFile.width(Width); DataFile << "X_species_29" << ',';
        DataFile.width(Width); DataFile << "X_species_3" << ',';
        DataFile.width(Width); DataFile << "X_species_30" << ',';
        DataFile.width(Width); DataFile << "X_species_31" << ',';
        DataFile.width(Width); DataFile << "X_species_32" << ',';
        DataFile.width(Width); DataFile << "X_species_33" << ',';
        DataFile.width(Width); DataFile << "X_species_34" << ',';
        DataFile.width(Width); DataFile << "X_species_35" << ',';
        DataFile.width(Width); DataFile << "X_species_36" << ',';
        DataFile.width(Width); DataFile << "X_species_37" << ',';
        DataFile.width(Width); DataFile << "X_species_38" << ',';
        DataFile.width(Width); DataFile << "X_species_39" << ',';
        DataFile.width(Width); DataFile << "X_species_4" << ',';
        DataFile.width(Width); DataFile << "X_species_40" << ',';
        DataFile.width(Width); DataFile << "X_species_41" << ',';
        DataFile.width(Width); DataFile << "X_species_42" << ',';
        DataFile.width(Width); DataFile << "X_species_43" << ',';
        DataFile.width(Width); DataFile << "X_species_44" << ',';
        DataFile.width(Width); DataFile << "X_species_45" << ',';
        DataFile.width(Width); DataFile << "X_species_46" << ',';
        DataFile.width(Width); DataFile << "X_species_47" << ',';
        DataFile.width(Width); DataFile << "X_species_48" << ',';
        DataFile.width(Width); DataFile << "X_species_49" << ',';
        DataFile.width(Width); DataFile << "X_species_5" << ',';
        DataFile.width(Width); DataFile << "X_species_50" << ',';
        DataFile.width(Width); DataFile << "X_species_51" << ',';
        DataFile.width(Width); DataFile << "X_species_52" << ',';
        DataFile.width(Width); DataFile << "X_species_53" << ',';
        DataFile.width(Width); DataFile << "X_species_54" << ',';
        DataFile.width(Width); DataFile << "X_species_55" << ',';
        DataFile.width(Width); DataFile << "X_species_56" << ',';
        DataFile.width(Width); DataFile << "X_species_57" << ',';
        DataFile.width(Width); DataFile << "X_species_58" << ',';
        DataFile.width(Width); DataFile << "X_species_59" << ',';
        DataFile.width(Width); DataFile << "X_species_6" << ',';
        DataFile.width(Width); DataFile << "X_species_60" << ',';
        DataFile.width(Width); DataFile << "X_species_61" << ',';
        DataFile.width(Width); DataFile << "X_species_62" << ',';
        DataFile.width(Width); DataFile << "X_species_63" << ',';
        DataFile.width(Width); DataFile << "X_species_64" << ',';
        DataFile.width(Width); DataFile << "X_species_65" << ',';
        DataFile.width(Width); DataFile << "X_species_66" << ',';
        DataFile.width(Width); DataFile << "X_species_7" << ',';
        DataFile.width(Width); DataFile << "X_species_8" << ',';
        DataFile.width(Width); DataFile << "X_species_9" << ',';
        DataFile.width(Width); DataFile << "sPAR_compartment_0" << ',';
        DataFile.width(Width); DataFile << "sPAR_parameter_0" << ',';
        DataFile.width(Width); DataFile << "sPAR_parameter_11" << ',';
        DataFile.width(Width); DataFile << "sPAR_parameter_12" << ',';
        DataFile.width(Width); DataFile << "sPAR_parameter_13" << ',';
        DataFile.width(Width); DataFile << "sPAR_parameter_14" << ',';
        DataFile.width(Width); DataFile << "sPAR_parameter_15" << ',';
        DataFile.width(Width); DataFile << "sPAR_parameter_16" << ',';
        DataFile.width(Width); DataFile << "sPAR_parameter_17" << ',';
        DataFile.width(Width); DataFile << "sPAR_parameter_18" << ',';
        DataFile.width(Width); DataFile << "sPAR_parameter_49" << ',';
        DataFile.width(Width); DataFile << "sPAR_parameter_50" << ',';
        DataFile.width(Width); DataFile << "sPAR_parameter_51" << ',';
        DataFile.width(Width); DataFile << "sPAR_parameter_52" << ',';
        DataFile.width(Width); DataFile << "sPAR_parameter_53" << ',';
        DataFile.width(Width); DataFile << "sPAR_parameter_9" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_1" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_10" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_19" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_2" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_20" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_21" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_22" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_23" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_24" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_25" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_26" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_27" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_28" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_29" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_3" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_30" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_31" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_32" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_33" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_34" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_35" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_36" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_37" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_38" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_39" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_4" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_40" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_41" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_42" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_43" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_44" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_45" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_46" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_47" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_48" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_5" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_54" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_55" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_56" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_57" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_58" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_59" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_6" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_60" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_7" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_8" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 45) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 42, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 43, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 44, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 45, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 46, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 47, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 48, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 49, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 50, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 51, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 52, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 53, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 54, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 55, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 56, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 57, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 58, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 59, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 60, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 61, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 62, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 63, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 64, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 65, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 66, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
    Solver.SolverOption(RelativeTolerance, 42, 1e-06);
    Solver.SolverOption(RelativeTolerance, 43, 1e-06);
    Solver.SolverOption(RelativeTolerance, 44, 1e-06);
    Solver.SolverOption(RelativeTolerance, 45, 1e-06);
    Solver.SolverOption(RelativeTolerance, 46, 1e-06);
    Solver.SolverOption(RelativeTolerance, 47, 1e-06);
    Solver.SolverOption(RelativeTolerance, 48, 1e-06);
    Solver.SolverOption(RelativeTolerance, 49, 1e-06);
    Solver.SolverOption(RelativeTolerance, 50, 1e-06);
    Solver.SolverOption(RelativeTolerance, 51, 1e-06);
    Solver.SolverOption(RelativeTolerance, 52, 1e-06);
    Solver.SolverOption(RelativeTolerance, 53, 1e-06);
    Solver.SolverOption(RelativeTolerance, 54, 1e-06);
    Solver.SolverOption(RelativeTolerance, 55, 1e-06);
    Solver.SolverOption(RelativeTolerance, 56, 1e-06);
    Solver.SolverOption(RelativeTolerance, 57, 1e-06);
    Solver.SolverOption(RelativeTolerance, 58, 1e-06);
    Solver.SolverOption(RelativeTolerance, 59, 1e-06);
    Solver.SolverOption(RelativeTolerance, 60, 1e-06);
    Solver.SolverOption(RelativeTolerance, 61, 1e-06);
    Solver.SolverOption(RelativeTolerance, 62, 1e-06);
    Solver.SolverOption(RelativeTolerance, 63, 1e-06);
    Solver.SolverOption(RelativeTolerance, 64, 1e-06);
    Solver.SolverOption(RelativeTolerance, 65, 1e-06);
    Solver.SolverOption(RelativeTolerance, 66, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD183.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

