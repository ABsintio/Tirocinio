#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#include "BIOMD268_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 42;
const int NCP  = 1;
const int NSP  = 156;
const int NISP = 0;
const int NE   = 0;
const int NA   = 9;
const int NIA  = 0;
const int NDO  = 100000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_BET" << ',';
        DataFile.width(Width); DataFile << "X_CO" << ',';
        DataFile.width(Width); DataFile << "X_DUMP" << ',';
        DataFile.width(Width); DataFile << "X_Fol" << ',';
        DataFile.width(Width); DataFile << "X_GAR" << ',';
        DataFile.width(Width); DataFile << "X_H2O2" << ',';
        DataFile.width(Width); DataFile << "X_HCHO" << ',';
        DataFile.width(Width); DataFile << "X_NADPH" << ',';
        DataFile.width(Width); DataFile << "X_aic" << ',';
        DataFile.width(Width); DataFile << "X_b_cys" << ',';
        DataFile.width(Width); DataFile << "X_b_glu" << ',';
        DataFile.width(Width); DataFile << "X_b_gly" << ',';
        DataFile.width(Width); DataFile << "X_b_gsg" << ',';
        DataFile.width(Width); DataFile << "X_b_gsh" << ',';
        DataFile.width(Width); DataFile << "X_c_10f" << ',';
        DataFile.width(Width); DataFile << "X_c_1cf" << ',';
        DataFile.width(Width); DataFile << "X_c_2cf" << ',';
        DataFile.width(Width); DataFile << "X_c_5mf" << ',';
        DataFile.width(Width); DataFile << "X_c_coo" << ',';
        DataFile.width(Width); DataFile << "X_c_cys" << ',';
        DataFile.width(Width); DataFile << "X_c_dhf" << ',';
        DataFile.width(Width); DataFile << "X_c_glu" << ',';
        DataFile.width(Width); DataFile << "X_c_gly" << ',';
        DataFile.width(Width); DataFile << "X_c_gsg" << ',';
        DataFile.width(Width); DataFile << "X_c_gsh" << ',';
        DataFile.width(Width); DataFile << "X_c_ser" << ',';
        DataFile.width(Width); DataFile << "X_c_thf" << ',';
        DataFile.width(Width); DataFile << "X_cyt" << ',';
        DataFile.width(Width); DataFile << "X_dmg" << ',';
        DataFile.width(Width); DataFile << "X_glc" << ',';
        DataFile.width(Width); DataFile << "X_hcy" << ',';
        DataFile.width(Width); DataFile << "X_m_10f" << ',';
        DataFile.width(Width); DataFile << "X_m_1cf" << ',';
        DataFile.width(Width); DataFile << "X_m_2cf" << ',';
        DataFile.width(Width); DataFile << "X_m_coo" << ',';
        DataFile.width(Width); DataFile << "X_m_gly" << ',';
        DataFile.width(Width); DataFile << "X_m_ser" << ',';
        DataFile.width(Width); DataFile << "X_m_thf" << ',';
        DataFile.width(Width); DataFile << "X_met" << ',';
        DataFile.width(Width); DataFile << "X_sah" << ',';
        DataFile.width(Width); DataFile << "X_sam" << ',';
        DataFile.width(Width); DataFile << "X_src" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_10f_ART" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_10f_FTD" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_10f_MTCH" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_10f_PGT" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_10f_mFTS" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_1cf_MTCH" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_1cf_MTD" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_2cf_MTD" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_2cf_MTHFR" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_2cf_SHMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_2cf_TS" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_5mf_MS" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_DUMP_TS" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_GAR_PGT" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_H2O2_GPX" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_NADPH_DHFR" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_NADPH_GR" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_NADPH_MTHFR" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_aic_ART" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_bcysc" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_bet_BHMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_bglutc" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_bglyc" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_bmetc" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_bserc" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_cgly" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_coo_cFTS" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_coo_mFTS" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_cser" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_cys_GCS" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_cyt_CTGL" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_dhf_DHFR" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_dmg_DMGD" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_glc_GS" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_glu_GCS" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_gly_GDC" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_gly_GNMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_gly_GS" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_gly_SHMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_gsgHb" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_gsgLb" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_gsg_GR" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_gshHb" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_gshLb" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_gsh_GPX" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_hcy_BHMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_hcy_CBS" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_hcy_MS" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_hcy_SAHH" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_mgly" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_mser" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_sah_SAHH" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_sam_GNMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_ser_CBS" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_ser_SHMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_src_SDH" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_thf_DMGD" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_thf_GDC" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_thf_SDH" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_thf_SHMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_thf_cFTS" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_thf_mFTS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ka_CBS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ka_GCS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ka_MAT3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ke_GCS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ke_GS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ki_BHMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ki_DNMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ki_GCS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ki_GNMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ki_MAT1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ki_MAT3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ki_MS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Km_DNMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_Km_MAT1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Km_MAT3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kp_GCS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kp_GS" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_bcysc" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_bglutc" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_bglyc" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_bmetc" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_bserc" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_cgly" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_cser" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_gsgHb" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_gsgLb" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_gshHb" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_gshLb" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_mgly" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_mser" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_oCys_b_basal" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_oGlu_b_basal" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_oGly_b_basal" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_SAHH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_cMTCH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_cMTD" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_cSHMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_mFTS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_mMTCH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_mMTD" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vf_mSHMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_ART" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_BHMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_CBS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_CTGL" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_DHFR" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_DMGD" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_DNMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_GCS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_GDC" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_GNMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_GPX" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_GR" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_GS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_MAT1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_MAT3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_MS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_MTHFR" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_PGT" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_SDH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_TS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_cFTD" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_cFTS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vm_mFTD" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_MTCH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_MTD" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_SAHH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_cMTD" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_cSHMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_mFTS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vr_mSHMT" << ',';
        DataFile.width(Width); DataFile << "sPAR_b_met_basal" << ',';
        DataFile.width(Width); DataFile << "sPAR_b_ser_basal" << ',';
        DataFile.width(Width); DataFile << "sPAR_blood" << ',';
        DataFile.width(Width); DataFile << "sPAR_breakfast" << ',';
        DataFile.width(Width); DataFile << "sPAR_cell" << ',';
        DataFile.width(Width); DataFile << "sPAR_cytosol" << ',';
        DataFile.width(Width); DataFile << "sPAR_dinner" << ',';
        DataFile.width(Width); DataFile << "sPAR_fasting" << ',';
        DataFile.width(Width); DataFile << "sPAR_h_gshLb" << ',';
        DataFile.width(Width); DataFile << "sPAR_k1_cNE" << ',';
        DataFile.width(Width); DataFile << "sPAR_k1_mNE" << ',';
        DataFile.width(Width); DataFile << "sPAR_k2_cNE" << ',';
        DataFile.width(Width); DataFile << "sPAR_k2_mNE" << ',';
        DataFile.width(Width); DataFile << "sPAR_k_in_coo" << ',';
        DataFile.width(Width); DataFile << "sPAR_k_out_coo" << ',';
        DataFile.width(Width); DataFile << "sPAR_k_out_cys" << ',';
        DataFile.width(Width); DataFile << "sPAR_k_out_glu" << ',';
        DataFile.width(Width); DataFile << "sPAR_k_out_gly" << ',';
        DataFile.width(Width); DataFile << "sPAR_k_out_met" << ',';
        DataFile.width(Width); DataFile << "sPAR_k_out_ser" << ',';
        DataFile.width(Width); DataFile << "sPAR_lunch" << ',';
        DataFile.width(Width); DataFile << "sPAR_mito" << ',';
        DataFile.width(Width); DataFile << "sPAR_ssH2O2" << ',';
        DataFile.width(Width); DataFile << "ACC_V_oCys_b" << ',';
        DataFile.width(Width); DataFile << "ACC_V_oGlu_b" << ',';
        DataFile.width(Width); DataFile << "ACC_V_oGly_b" << ',';
        DataFile.width(Width); DataFile << "ACC_aa_input" << ',';
        DataFile.width(Width); DataFile << "ACC_b_met" << ',';
        DataFile.width(Width); DataFile << "ACC_b_ser" << ',';
        DataFile.width(Width); DataFile << "ACC_daytime" << ',';
        DataFile.width(Width); DataFile << "ACC_tot_cfol" << ',';
        DataFile.width(Width); DataFile << "ACC_tot_mfol" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 83) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 84) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 85) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 86) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 87) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 88) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 89) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 90) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 91) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 92) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 93) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 94) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 95) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 96) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 97) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 98) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 99) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 100) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 101) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 102) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 103) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 104) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 105) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 106) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 107) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 108) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 109) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 110) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 111) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 112) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 113) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 114) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 115) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 116) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 117) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 118) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 119) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 120) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 121) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 122) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 123) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 124) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 125) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 126) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 127) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 128) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 129) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 130) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 131) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 132) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 133) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 134) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 135) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 136) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 137) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 138) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 139) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 140) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 141) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 142) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 143) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 144) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 145) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 146) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 147) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 148) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 149) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 150) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 151) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 152) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 153) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 154) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 155) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD268.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

