#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD013_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 27;
const int NCP  = 1;
const int NSP  = 65;
const int NISP = 0;
const int NE   = 0;
const int NA   = 0;
const int NIA  = 0;
const int NDO  = 1000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 0.1);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_ADP_ch" << ',';
        DataFile.width(Width); DataFile << "X_ATP_ch" << ',';
        DataFile.width(Width); DataFile << "X_BPGA_ch" << ',';
        DataFile.width(Width); DataFile << "X_DHAP_ch" << ',';
        DataFile.width(Width); DataFile << "X_E4P_ch" << ',';
        DataFile.width(Width); DataFile << "X_F6P_ch" << ',';
        DataFile.width(Width); DataFile << "X_FBP_ch" << ',';
        DataFile.width(Width); DataFile << "X_G1P_ch" << ',';
        DataFile.width(Width); DataFile << "X_G6P_ch" << ',';
        DataFile.width(Width); DataFile << "X_GAP_ch" << ',';
        DataFile.width(Width); DataFile << "X_PGA_ch" << ',';
        DataFile.width(Width); DataFile << "X_Pi_ch" << ',';
        DataFile.width(Width); DataFile << "X_R5P_ch" << ',';
        DataFile.width(Width); DataFile << "X_Ru5P_ch" << ',';
        DataFile.width(Width); DataFile << "X_RuBP_ch" << ',';
        DataFile.width(Width); DataFile << "X_S7P_ch" << ',';
        DataFile.width(Width); DataFile << "X_SBP_ch" << ',';
        DataFile.width(Width); DataFile << "X_X5P_ch" << ',';
        DataFile.width(Width); DataFile << "X_x_CO2" << ',';
        DataFile.width(Width); DataFile << "X_x_DHAP_cyt" << ',';
        DataFile.width(Width); DataFile << "X_x_GAP_cyt" << ',';
        DataFile.width(Width); DataFile << "X_x_NADPH_ch" << ',';
        DataFile.width(Width); DataFile << "X_x_NADP_ch" << ',';
        DataFile.width(Width); DataFile << "X_x_PGA_cyt" << ',';
        DataFile.width(Width); DataFile << "X_x_Pi_cyt" << ',';
        DataFile.width(Width); DataFile << "X_x_Proton_ch" << ',';
        DataFile.width(Width); DataFile << "X_x_Starch_ch" << ',';
        DataFile.width(Width); DataFile << "sPAR_E_Aldo_v" << ',';
        DataFile.width(Width); DataFile << "sPAR_FBPase_ch_KiF6P" << ',';
        DataFile.width(Width); DataFile << "sPAR_FBPase_ch_KiPi" << ',';
        DataFile.width(Width); DataFile << "sPAR_FBPase_ch_km" << ',';
        DataFile.width(Width); DataFile << "sPAR_FBPase_ch_vm" << ',';
        DataFile.width(Width); DataFile << "sPAR_F_Aldo_v" << ',';
        DataFile.width(Width); DataFile << "sPAR_F_TKL_v" << ',';
        DataFile.width(Width); DataFile << "sPAR_G3Pdh_v" << ',';
        DataFile.width(Width); DataFile << "sPAR_G_TKL_v" << ',';
        DataFile.width(Width); DataFile << "sPAR_LR_kmADP" << ',';
        DataFile.width(Width); DataFile << "sPAR_LR_kmPi" << ',';
        DataFile.width(Width); DataFile << "sPAR_LR_vm" << ',';
        DataFile.width(Width); DataFile << "sPAR_Light_on" << ',';
        DataFile.width(Width); DataFile << "sPAR_PGA_xpMult" << ',';
        DataFile.width(Width); DataFile << "sPAR_PGI_v" << ',';
        DataFile.width(Width); DataFile << "sPAR_PGK_v" << ',';
        DataFile.width(Width); DataFile << "sPAR_PGM_v" << ',';
        DataFile.width(Width); DataFile << "sPAR_R5Piso_v" << ',';
        DataFile.width(Width); DataFile << "sPAR_Rbco_KiFBP" << ',';
        DataFile.width(Width); DataFile << "sPAR_Rbco_KiNADPH" << ',';
        DataFile.width(Width); DataFile << "sPAR_Rbco_KiPGA" << ',';
        DataFile.width(Width); DataFile << "sPAR_Rbco_KiPi" << ',';
        DataFile.width(Width); DataFile << "sPAR_Rbco_KiSBP" << ',';
        DataFile.width(Width); DataFile << "sPAR_Rbco_km" << ',';
        DataFile.width(Width); DataFile << "sPAR_Rbco_vm" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ru5Pk_ch_KiADP1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ru5Pk_ch_KiADP2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ru5Pk_ch_KiPGA" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ru5Pk_ch_KiPi" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ru5Pk_ch_KiRuBP" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ru5Pk_ch_km1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ru5Pk_ch_km2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ru5Pk_ch_vm" << ',';
        DataFile.width(Width); DataFile << "sPAR_SBPase_ch_KiPi" << ',';
        DataFile.width(Width); DataFile << "sPAR_SBPase_ch_km" << ',';
        DataFile.width(Width); DataFile << "sPAR_SBPase_ch_vm" << ',';
        DataFile.width(Width); DataFile << "sPAR_StPase_Vm" << ',';
        DataFile.width(Width); DataFile << "sPAR_StPase_kiG1P" << ',';
        DataFile.width(Width); DataFile << "sPAR_StPase_km" << ',';
        DataFile.width(Width); DataFile << "sPAR_StSyn_vm" << ',';
        DataFile.width(Width); DataFile << "sPAR_TPI_v" << ',';
        DataFile.width(Width); DataFile << "sPAR_TP_Piap_kDHAP_ch" << ',';
        DataFile.width(Width); DataFile << "sPAR_TP_Piap_kGAP_ch" << ',';
        DataFile.width(Width); DataFile << "sPAR_TP_Piap_kPGA_ch" << ',';
        DataFile.width(Width); DataFile << "sPAR_TP_Piap_kPi_ch" << ',';
        DataFile.width(Width); DataFile << "sPAR_TP_Piap_kPi_cyt" << ',';
        DataFile.width(Width); DataFile << "sPAR_TP_Piap_vm" << ',';
        DataFile.width(Width); DataFile << "sPAR_X5Pepi_v" << ',';
        DataFile.width(Width); DataFile << "sPAR_q10" << ',';
        DataFile.width(Width); DataFile << "sPAR_q11" << ',';
        DataFile.width(Width); DataFile << "sPAR_q12" << ',';
        DataFile.width(Width); DataFile << "sPAR_q14" << ',';
        DataFile.width(Width); DataFile << "sPAR_q15" << ',';
        DataFile.width(Width); DataFile << "sPAR_q2" << ',';
        DataFile.width(Width); DataFile << "sPAR_q3" << ',';
        DataFile.width(Width); DataFile << "sPAR_q4" << ',';
        DataFile.width(Width); DataFile << "sPAR_q5" << ',';
        DataFile.width(Width); DataFile << "sPAR_q7" << ',';
        DataFile.width(Width); DataFile << "sPAR_q8" << ',';
        DataFile.width(Width); DataFile << "sPAR_stsyn_ch_Ki" << ',';
        DataFile.width(Width); DataFile << "sPAR_stsyn_ch_ka1" << ',';
        DataFile.width(Width); DataFile << "sPAR_stsyn_ch_ka2" << ',';
        DataFile.width(Width); DataFile << "sPAR_stsyn_ch_ka3" << ',';
        DataFile.width(Width); DataFile << "sPAR_stsyn_ch_km1" << ',';
        DataFile.width(Width); DataFile << "sPAR_stsyn_ch_km2" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 64) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 64; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 10.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD013.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

