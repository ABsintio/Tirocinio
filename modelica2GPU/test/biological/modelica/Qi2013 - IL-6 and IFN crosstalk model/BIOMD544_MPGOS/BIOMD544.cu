#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#include "BIOMD544_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45
#define PRECISION double
const int NT   = 1;
const int SD   = 106;
const int NCP  = 1;
const int NSP  = 0;
const int NISP = 0;
const int NE   = 0;
const int NA   = 2;
const int NIA  = 0;
const int NDO  = 100000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        ProblemNumber++;
        k_begin++;
    }
}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_s118" << ',';
        DataFile.width(Width); DataFile << "X_s119" << ',';
        DataFile.width(Width); DataFile << "X_s120" << ',';
        DataFile.width(Width); DataFile << "X_s122" << ',';
        DataFile.width(Width); DataFile << "X_s126" << ',';
        DataFile.width(Width); DataFile << "X_s135" << ',';
        DataFile.width(Width); DataFile << "X_s138" << ',';
        DataFile.width(Width); DataFile << "X_s139" << ',';
        DataFile.width(Width); DataFile << "X_s140" << ',';
        DataFile.width(Width); DataFile << "X_species_1" << ',';
        DataFile.width(Width); DataFile << "X_species_10" << ',';
        DataFile.width(Width); DataFile << "X_species_100" << ',';
        DataFile.width(Width); DataFile << "X_species_101" << ',';
        DataFile.width(Width); DataFile << "X_species_102" << ',';
        DataFile.width(Width); DataFile << "X_species_103" << ',';
        DataFile.width(Width); DataFile << "X_species_104" << ',';
        DataFile.width(Width); DataFile << "X_species_105" << ',';
        DataFile.width(Width); DataFile << "X_species_106" << ',';
        DataFile.width(Width); DataFile << "X_species_107" << ',';
        DataFile.width(Width); DataFile << "X_species_108" << ',';
        DataFile.width(Width); DataFile << "X_species_11" << ',';
        DataFile.width(Width); DataFile << "X_species_12" << ',';
        DataFile.width(Width); DataFile << "X_species_13" << ',';
        DataFile.width(Width); DataFile << "X_species_14" << ',';
        DataFile.width(Width); DataFile << "X_species_15" << ',';
        DataFile.width(Width); DataFile << "X_species_16" << ',';
        DataFile.width(Width); DataFile << "X_species_18" << ',';
        DataFile.width(Width); DataFile << "X_species_19" << ',';
        DataFile.width(Width); DataFile << "X_species_2" << ',';
        DataFile.width(Width); DataFile << "X_species_20" << ',';
        DataFile.width(Width); DataFile << "X_species_21" << ',';
        DataFile.width(Width); DataFile << "X_species_22" << ',';
        DataFile.width(Width); DataFile << "X_species_23" << ',';
        DataFile.width(Width); DataFile << "X_species_24" << ',';
        DataFile.width(Width); DataFile << "X_species_25" << ',';
        DataFile.width(Width); DataFile << "X_species_26" << ',';
        DataFile.width(Width); DataFile << "X_species_27" << ',';
        DataFile.width(Width); DataFile << "X_species_28" << ',';
        DataFile.width(Width); DataFile << "X_species_29" << ',';
        DataFile.width(Width); DataFile << "X_species_3" << ',';
        DataFile.width(Width); DataFile << "X_species_30" << ',';
        DataFile.width(Width); DataFile << "X_species_31" << ',';
        DataFile.width(Width); DataFile << "X_species_32" << ',';
        DataFile.width(Width); DataFile << "X_species_33" << ',';
        DataFile.width(Width); DataFile << "X_species_34" << ',';
        DataFile.width(Width); DataFile << "X_species_35" << ',';
        DataFile.width(Width); DataFile << "X_species_36" << ',';
        DataFile.width(Width); DataFile << "X_species_37" << ',';
        DataFile.width(Width); DataFile << "X_species_38" << ',';
        DataFile.width(Width); DataFile << "X_species_39" << ',';
        DataFile.width(Width); DataFile << "X_species_4" << ',';
        DataFile.width(Width); DataFile << "X_species_40" << ',';
        DataFile.width(Width); DataFile << "X_species_41" << ',';
        DataFile.width(Width); DataFile << "X_species_42" << ',';
        DataFile.width(Width); DataFile << "X_species_43" << ',';
        DataFile.width(Width); DataFile << "X_species_44" << ',';
        DataFile.width(Width); DataFile << "X_species_45" << ',';
        DataFile.width(Width); DataFile << "X_species_46" << ',';
        DataFile.width(Width); DataFile << "X_species_47" << ',';
        DataFile.width(Width); DataFile << "X_species_48" << ',';
        DataFile.width(Width); DataFile << "X_species_49" << ',';
        DataFile.width(Width); DataFile << "X_species_5" << ',';
        DataFile.width(Width); DataFile << "X_species_50" << ',';
        DataFile.width(Width); DataFile << "X_species_51" << ',';
        DataFile.width(Width); DataFile << "X_species_52" << ',';
        DataFile.width(Width); DataFile << "X_species_53" << ',';
        DataFile.width(Width); DataFile << "X_species_54" << ',';
        DataFile.width(Width); DataFile << "X_species_55" << ',';
        DataFile.width(Width); DataFile << "X_species_56" << ',';
        DataFile.width(Width); DataFile << "X_species_57" << ',';
        DataFile.width(Width); DataFile << "X_species_58" << ',';
        DataFile.width(Width); DataFile << "X_species_59" << ',';
        DataFile.width(Width); DataFile << "X_species_6" << ',';
        DataFile.width(Width); DataFile << "X_species_60" << ',';
        DataFile.width(Width); DataFile << "X_species_61" << ',';
        DataFile.width(Width); DataFile << "X_species_62" << ',';
        DataFile.width(Width); DataFile << "X_species_63" << ',';
        DataFile.width(Width); DataFile << "X_species_64" << ',';
        DataFile.width(Width); DataFile << "X_species_65" << ',';
        DataFile.width(Width); DataFile << "X_species_66" << ',';
        DataFile.width(Width); DataFile << "X_species_7" << ',';
        DataFile.width(Width); DataFile << "X_species_74" << ',';
        DataFile.width(Width); DataFile << "X_species_75" << ',';
        DataFile.width(Width); DataFile << "X_species_76" << ',';
        DataFile.width(Width); DataFile << "X_species_78" << ',';
        DataFile.width(Width); DataFile << "X_species_79" << ',';
        DataFile.width(Width); DataFile << "X_species_8" << ',';
        DataFile.width(Width); DataFile << "X_species_80" << ',';
        DataFile.width(Width); DataFile << "X_species_81" << ',';
        DataFile.width(Width); DataFile << "X_species_82" << ',';
        DataFile.width(Width); DataFile << "X_species_83" << ',';
        DataFile.width(Width); DataFile << "X_species_84" << ',';
        DataFile.width(Width); DataFile << "X_species_85" << ',';
        DataFile.width(Width); DataFile << "X_species_86" << ',';
        DataFile.width(Width); DataFile << "X_species_87" << ',';
        DataFile.width(Width); DataFile << "X_species_88" << ',';
        DataFile.width(Width); DataFile << "X_species_9" << ',';
        DataFile.width(Width); DataFile << "X_species_90" << ',';
        DataFile.width(Width); DataFile << "X_species_91" << ',';
        DataFile.width(Width); DataFile << "X_species_92" << ',';
        DataFile.width(Width); DataFile << "X_species_94" << ',';
        DataFile.width(Width); DataFile << "X_species_95" << ',';
        DataFile.width(Width); DataFile << "X_species_96" << ',';
        DataFile.width(Width); DataFile << "X_species_97" << ',';
        DataFile.width(Width); DataFile << "X_species_98" << ',';
        DataFile.width(Width); DataFile << "X_species_99" << ',';
        DataFile.width(Width); DataFile << "ACC_s136" << ',';
        DataFile.width(Width); DataFile << "ACC_s137" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 83) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 84) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 85) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 86) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 87) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 88) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 89) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 90) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 91) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 92) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 93) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 94) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 95) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 96) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 97) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 98) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 99) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 100) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 101) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 102) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 103) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 104) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 105) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 42, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 43, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 44, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 45, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 46, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 47, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 48, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 49, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 50, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 51, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 52, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 53, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 54, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 55, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 56, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 57, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 58, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 59, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 60, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 61, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 62, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 63, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 64, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 65, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 66, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 67, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 68, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 69, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 70, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 71, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 72, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 73, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 74, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 75, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 76, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 77, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 78, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 79, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 80, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 81, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 82, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 83, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 84, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 85, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 86, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 87, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 88, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 89, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 90, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 91, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 92, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 93, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 94, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 95, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 96, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 97, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 98, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 99, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 100, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 101, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 102, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 103, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 104, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 105, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
    Solver.SolverOption(RelativeTolerance, 42, 1e-06);
    Solver.SolverOption(RelativeTolerance, 43, 1e-06);
    Solver.SolverOption(RelativeTolerance, 44, 1e-06);
    Solver.SolverOption(RelativeTolerance, 45, 1e-06);
    Solver.SolverOption(RelativeTolerance, 46, 1e-06);
    Solver.SolverOption(RelativeTolerance, 47, 1e-06);
    Solver.SolverOption(RelativeTolerance, 48, 1e-06);
    Solver.SolverOption(RelativeTolerance, 49, 1e-06);
    Solver.SolverOption(RelativeTolerance, 50, 1e-06);
    Solver.SolverOption(RelativeTolerance, 51, 1e-06);
    Solver.SolverOption(RelativeTolerance, 52, 1e-06);
    Solver.SolverOption(RelativeTolerance, 53, 1e-06);
    Solver.SolverOption(RelativeTolerance, 54, 1e-06);
    Solver.SolverOption(RelativeTolerance, 55, 1e-06);
    Solver.SolverOption(RelativeTolerance, 56, 1e-06);
    Solver.SolverOption(RelativeTolerance, 57, 1e-06);
    Solver.SolverOption(RelativeTolerance, 58, 1e-06);
    Solver.SolverOption(RelativeTolerance, 59, 1e-06);
    Solver.SolverOption(RelativeTolerance, 60, 1e-06);
    Solver.SolverOption(RelativeTolerance, 61, 1e-06);
    Solver.SolverOption(RelativeTolerance, 62, 1e-06);
    Solver.SolverOption(RelativeTolerance, 63, 1e-06);
    Solver.SolverOption(RelativeTolerance, 64, 1e-06);
    Solver.SolverOption(RelativeTolerance, 65, 1e-06);
    Solver.SolverOption(RelativeTolerance, 66, 1e-06);
    Solver.SolverOption(RelativeTolerance, 67, 1e-06);
    Solver.SolverOption(RelativeTolerance, 68, 1e-06);
    Solver.SolverOption(RelativeTolerance, 69, 1e-06);
    Solver.SolverOption(RelativeTolerance, 70, 1e-06);
    Solver.SolverOption(RelativeTolerance, 71, 1e-06);
    Solver.SolverOption(RelativeTolerance, 72, 1e-06);
    Solver.SolverOption(RelativeTolerance, 73, 1e-06);
    Solver.SolverOption(RelativeTolerance, 74, 1e-06);
    Solver.SolverOption(RelativeTolerance, 75, 1e-06);
    Solver.SolverOption(RelativeTolerance, 76, 1e-06);
    Solver.SolverOption(RelativeTolerance, 77, 1e-06);
    Solver.SolverOption(RelativeTolerance, 78, 1e-06);
    Solver.SolverOption(RelativeTolerance, 79, 1e-06);
    Solver.SolverOption(RelativeTolerance, 80, 1e-06);
    Solver.SolverOption(RelativeTolerance, 81, 1e-06);
    Solver.SolverOption(RelativeTolerance, 82, 1e-06);
    Solver.SolverOption(RelativeTolerance, 83, 1e-06);
    Solver.SolverOption(RelativeTolerance, 84, 1e-06);
    Solver.SolverOption(RelativeTolerance, 85, 1e-06);
    Solver.SolverOption(RelativeTolerance, 86, 1e-06);
    Solver.SolverOption(RelativeTolerance, 87, 1e-06);
    Solver.SolverOption(RelativeTolerance, 88, 1e-06);
    Solver.SolverOption(RelativeTolerance, 89, 1e-06);
    Solver.SolverOption(RelativeTolerance, 90, 1e-06);
    Solver.SolverOption(RelativeTolerance, 91, 1e-06);
    Solver.SolverOption(RelativeTolerance, 92, 1e-06);
    Solver.SolverOption(RelativeTolerance, 93, 1e-06);
    Solver.SolverOption(RelativeTolerance, 94, 1e-06);
    Solver.SolverOption(RelativeTolerance, 95, 1e-06);
    Solver.SolverOption(RelativeTolerance, 96, 1e-06);
    Solver.SolverOption(RelativeTolerance, 97, 1e-06);
    Solver.SolverOption(RelativeTolerance, 98, 1e-06);
    Solver.SolverOption(RelativeTolerance, 99, 1e-06);
    Solver.SolverOption(RelativeTolerance, 100, 1e-06);
    Solver.SolverOption(RelativeTolerance, 101, 1e-06);
    Solver.SolverOption(RelativeTolerance, 102, 1e-06);
    Solver.SolverOption(RelativeTolerance, 103, 1e-06);
    Solver.SolverOption(RelativeTolerance, 104, 1e-06);
    Solver.SolverOption(RelativeTolerance, 105, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD544.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

