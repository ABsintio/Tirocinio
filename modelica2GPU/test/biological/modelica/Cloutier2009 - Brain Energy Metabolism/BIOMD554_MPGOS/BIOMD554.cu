#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#include "BIOMD554_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45
#define PRECISION double
const int NT   = 1;
const int SD   = 45;
const int NCP  = 1;
const int NSP  = 0;
const int NISP = 0;
const int NE   = 0;
const int NA   = 64;
const int NIA  = 0;
const int NDO  = 100000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        ProblemNumber++;
        k_begin++;
    }
}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_ATPg" << ',';
        DataFile.width(Width); DataFile << "X_ATPn" << ',';
        DataFile.width(Width); DataFile << "X_CO2a" << ',';
        DataFile.width(Width); DataFile << "X_CO2c" << ',';
        DataFile.width(Width); DataFile << "X_F6Pg" << ',';
        DataFile.width(Width); DataFile << "X_F6Pn" << ',';
        DataFile.width(Width); DataFile << "X_G6Pg" << ',';
        DataFile.width(Width); DataFile << "X_G6Pn" << ',';
        DataFile.width(Width); DataFile << "X_GAPg" << ',';
        DataFile.width(Width); DataFile << "X_GAPn" << ',';
        DataFile.width(Width); DataFile << "X_GLCa" << ',';
        DataFile.width(Width); DataFile << "X_GLCc" << ',';
        DataFile.width(Width); DataFile << "X_GLCe" << ',';
        DataFile.width(Width); DataFile << "X_GLCg" << ',';
        DataFile.width(Width); DataFile << "X_GLCn" << ',';
        DataFile.width(Width); DataFile << "X_GLUe" << ',';
        DataFile.width(Width); DataFile << "X_GLUg" << ',';
        DataFile.width(Width); DataFile << "X_GLUn" << ',';
        DataFile.width(Width); DataFile << "X_GLYg" << ',';
        DataFile.width(Width); DataFile << "X_LACa" << ',';
        DataFile.width(Width); DataFile << "X_LACc" << ',';
        DataFile.width(Width); DataFile << "X_LACe" << ',';
        DataFile.width(Width); DataFile << "X_LACg" << ',';
        DataFile.width(Width); DataFile << "X_LACn" << ',';
        DataFile.width(Width); DataFile << "X_NADH_g_tot" << ',';
        DataFile.width(Width); DataFile << "X_NADH_n_tot" << ',';
        DataFile.width(Width); DataFile << "X_NADHg" << ',';
        DataFile.width(Width); DataFile << "X_NADHn" << ',';
        DataFile.width(Width); DataFile << "X_NAe" << ',';
        DataFile.width(Width); DataFile << "X_NAg" << ',';
        DataFile.width(Width); DataFile << "X_NAn" << ',';
        DataFile.width(Width); DataFile << "X_O2a" << ',';
        DataFile.width(Width); DataFile << "X_O2c" << ',';
        DataFile.width(Width); DataFile << "X_O2g" << ',';
        DataFile.width(Width); DataFile << "X_O2n" << ',';
        DataFile.width(Width); DataFile << "X_PCrg" << ',';
        DataFile.width(Width); DataFile << "X_PCrg_tot" << ',';
        DataFile.width(Width); DataFile << "X_PCrn" << ',';
        DataFile.width(Width); DataFile << "X_PCrn_tot" << ',';
        DataFile.width(Width); DataFile << "X_PEPg" << ',';
        DataFile.width(Width); DataFile << "X_PEPn" << ',';
        DataFile.width(Width); DataFile << "X_PYRg" << ',';
        DataFile.width(Width); DataFile << "X_PYRn" << ',';
        DataFile.width(Width); DataFile << "X_Vv" << ',';
        DataFile.width(Width); DataFile << "X_dHb" << ',';
        DataFile.width(Width); DataFile << "ACC_ADPg" << ',';
        DataFile.width(Width); DataFile << "ACC_ADPn" << ',';
        DataFile.width(Width); DataFile << "ACC_AMPg" << ',';
        DataFile.width(Width); DataFile << "ACC_AMPn" << ',';
        DataFile.width(Width); DataFile << "ACC_BOLD" << ',';
        DataFile.width(Width); DataFile << "ACC_CRg" << ',';
        DataFile.width(Width); DataFile << "ACC_CRn" << ',';
        DataFile.width(Width); DataFile << "ACC_Fin_t" << ',';
        DataFile.width(Width); DataFile << "ACC_Fout_t" << ',';
        DataFile.width(Width); DataFile << "ACC_NADg" << ',';
        DataFile.width(Width); DataFile << "ACC_NADn" << ',';
        DataFile.width(Width); DataFile << "ACC_V_en_GLC" << ',';
        DataFile.width(Width); DataFile << "ACC_Vc_CO2" << ',';
        DataFile.width(Width); DataFile << "ACC_Vc_GLC" << ',';
        DataFile.width(Width); DataFile << "ACC_Vc_LAC" << ',';
        DataFile.width(Width); DataFile << "ACC_Vc_O2" << ',';
        DataFile.width(Width); DataFile << "ACC_Vce_GLC" << ',';
        DataFile.width(Width); DataFile << "ACC_Vcg_GLC" << ',';
        DataFile.width(Width); DataFile << "ACC_Vcg_O2" << ',';
        DataFile.width(Width); DataFile << "ACC_Vcn_O2" << ',';
        DataFile.width(Width); DataFile << "ACC_Vec_LAC" << ',';
        DataFile.width(Width); DataFile << "ACC_Veg_GLC" << ',';
        DataFile.width(Width); DataFile << "ACC_Veg_GLU" << ',';
        DataFile.width(Width); DataFile << "ACC_Vg_ATPase" << ',';
        DataFile.width(Width); DataFile << "ACC_Vg_ck" << ',';
        DataFile.width(Width); DataFile << "ACC_Vg_glyp" << ',';
        DataFile.width(Width); DataFile << "ACC_Vg_glys" << ',';
        DataFile.width(Width); DataFile << "ACC_Vg_gs" << ',';
        DataFile.width(Width); DataFile << "ACC_Vg_hk" << ',';
        DataFile.width(Width); DataFile << "ACC_Vg_ldh" << ',';
        DataFile.width(Width); DataFile << "ACC_Vg_leak_Na" << ',';
        DataFile.width(Width); DataFile << "ACC_Vg_mito" << ',';
        DataFile.width(Width); DataFile << "ACC_Vg_pfk" << ',';
        DataFile.width(Width); DataFile << "ACC_Vg_pgi" << ',';
        DataFile.width(Width); DataFile << "ACC_Vg_pgk" << ',';
        DataFile.width(Width); DataFile << "ACC_Vg_pk" << ',';
        DataFile.width(Width); DataFile << "ACC_Vg_pump" << ',';
        DataFile.width(Width); DataFile << "ACC_Vgc_CO2" << ',';
        DataFile.width(Width); DataFile << "ACC_Vgc_LAC" << ',';
        DataFile.width(Width); DataFile << "ACC_Vge_LAC" << ',';
        DataFile.width(Width); DataFile << "ACC_Vn_ATPase" << ',';
        DataFile.width(Width); DataFile << "ACC_Vn_ck" << ',';
        DataFile.width(Width); DataFile << "ACC_Vn_hk" << ',';
        DataFile.width(Width); DataFile << "ACC_Vn_ldh" << ',';
        DataFile.width(Width); DataFile << "ACC_Vn_leak_Na" << ',';
        DataFile.width(Width); DataFile << "ACC_Vn_mito" << ',';
        DataFile.width(Width); DataFile << "ACC_Vn_pfk" << ',';
        DataFile.width(Width); DataFile << "ACC_Vn_pgi" << ',';
        DataFile.width(Width); DataFile << "ACC_Vn_pgk" << ',';
        DataFile.width(Width); DataFile << "ACC_Vn_pk" << ',';
        DataFile.width(Width); DataFile << "ACC_Vn_pump" << ',';
        DataFile.width(Width); DataFile << "ACC_Vn_stim" << ',';
        DataFile.width(Width); DataFile << "ACC_Vn_stim_GLU" << ',';
        DataFile.width(Width); DataFile << "ACC_Vnc_CO2" << ',';
        DataFile.width(Width); DataFile << "ACC_Vne_LAC" << ',';
        DataFile.width(Width); DataFile << "ACC_dAMP_dATPg" << ',';
        DataFile.width(Width); DataFile << "ACC_dAMP_dATPn" << ',';
        DataFile.width(Width); DataFile << "ACC_deltaVt_GLY" << ',';
        DataFile.width(Width); DataFile << "ACC_u_g" << ',';
        DataFile.width(Width); DataFile << "ACC_u_n" << ',';
        DataFile.width(Width); DataFile << "ACC_unitpulseSB" << ',';
        DataFile.width(Width); DataFile << "ACC_unitstepSB" << ',';
        DataFile.width(Width); DataFile << "ACC_unitstepSB2" << ',';
        DataFile.width(Width); DataFile << "ACC_v_stim" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 63) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 42, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 43, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 44, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
    Solver.SolverOption(RelativeTolerance, 42, 1e-06);
    Solver.SolverOption(RelativeTolerance, 43, 1e-06);
    Solver.SolverOption(RelativeTolerance, 44, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD554.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

