#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD124_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 8;
const int NCP  = 1;
const int NSP  = 29;
const int NISP = 0;
const int NE   = 1;
const int NA   = 24;
const int NIA  = 0;
const int NDO  = 30000000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 30000.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_V" << ',';
        DataFile.width(Width); DataFile << "X_a" << ',';
        DataFile.width(Width); DataFile << "X_c" << ',';
        DataFile.width(Width); DataFile << "X_cer" << ',';
        DataFile.width(Width); DataFile << "X_gkatp" << ',';
        DataFile.width(Width); DataFile << "X_n" << ',';
        DataFile.width(Width); DataFile << "X_nIR" << ',';
        DataFile.width(Width); DataFile << "X_rIR" << ',';
        DataFile.width(Width); DataFile << "sPAR_alpha" << ',';
        DataFile.width(Width); DataFile << "sPAR_cell" << ',';
        DataFile.width(Width); DataFile << "sPAR_cm" << ',';
        DataFile.width(Width); DataFile << "sPAR_dact" << ',';
        DataFile.width(Width); DataFile << "sPAR_dinact" << ',';
        DataFile.width(Width); DataFile << "sPAR_dip3" << ',';
        DataFile.width(Width); DataFile << "sPAR_fcyt" << ',';
        DataFile.width(Width); DataFile << "sPAR_fer" << ',';
        DataFile.width(Width); DataFile << "sPAR_gca" << ',';
        DataFile.width(Width); DataFile << "sPAR_gir" << ',';
        DataFile.width(Width); DataFile << "sPAR_gk" << ',';
        DataFile.width(Width); DataFile << "sPAR_gkca" << ',';
        DataFile.width(Width); DataFile << "sPAR_ip3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd" << ',';
        DataFile.width(Width); DataFile << "sPAR_kpmca" << ',';
        DataFile.width(Width); DataFile << "sPAR_kserca" << ',';
        DataFile.width(Width); DataFile << "sPAR_pleak" << ',';
        DataFile.width(Width); DataFile << "sPAR_r" << ',';
        DataFile.width(Width); DataFile << "sPAR_sa" << ',';
        DataFile.width(Width); DataFile << "sPAR_sm" << ',';
        DataFile.width(Width); DataFile << "sPAR_sn" << ',';
        DataFile.width(Width); DataFile << "sPAR_taua" << ',';
        DataFile.width(Width); DataFile << "sPAR_taun" << ',';
        DataFile.width(Width); DataFile << "sPAR_vca" << ',';
        DataFile.width(Width); DataFile << "sPAR_vcytver" << ',';
        DataFile.width(Width); DataFile << "sPAR_vir" << ',';
        DataFile.width(Width); DataFile << "sPAR_vk" << ',';
        DataFile.width(Width); DataFile << "sPAR_vm" << ',';
        DataFile.width(Width); DataFile << "sPAR_vn" << ',';
        DataFile.width(Width); DataFile << "ACC_ainf" << ',';
        DataFile.width(Width); DataFile << "ACC_alphaIRn" << ',';
        DataFile.width(Width); DataFile << "ACC_alphaIRr" << ',';
        DataFile.width(Width); DataFile << "ACC_betaIRn" << ',';
        DataFile.width(Width); DataFile << "ACC_betaIRr" << ',';
        DataFile.width(Width); DataFile << "ACC_ica" << ',';
        DataFile.width(Width); DataFile << "ACC_iir" << ',';
        DataFile.width(Width); DataFile << "ACC_ik" << ',';
        DataFile.width(Width); DataFile << "ACC_ikatp" << ',';
        DataFile.width(Width); DataFile << "ACC_ikca" << ',';
        DataFile.width(Width); DataFile << "ACC_jer" << ',';
        DataFile.width(Width); DataFile << "ACC_jip3" << ',';
        DataFile.width(Width); DataFile << "ACC_jleak" << ',';
        DataFile.width(Width); DataFile << "ACC_jmem" << ',';
        DataFile.width(Width); DataFile << "ACC_jserca" << ',';
        DataFile.width(Width); DataFile << "ACC_minf" << ',';
        DataFile.width(Width); DataFile << "ACC_nIRinf" << ',';
        DataFile.width(Width); DataFile << "ACC_ninf" << ',';
        DataFile.width(Width); DataFile << "ACC_oinf" << ',';
        DataFile.width(Width); DataFile << "ACC_rIRinf" << ',';
        DataFile.width(Width); DataFile << "ACC_tauIRn" << ',';
        DataFile.width(Width); DataFile << "ACC_tauIRr" << ',';
        DataFile.width(Width); DataFile << "ACC_w" << ',';
        DataFile.width(Width); DataFile << "ACC_$whenCondition1" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 23) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);


    Solver.SolverOption(EventDirection, 0, 0);

    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD124.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

