#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD400_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 13;
const int NCP  = 1;
const int NSP  = 32;
const int NISP = 0;
const int NE   = 0;
const int NA   = 24;
const int NIA  = 0;
const int NDO  = 10000000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 1000.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_Ca" << ',';
        DataFile.width(Width); DataFile << "X_Gd" << ',';
        DataFile.width(Width); DataFile << "X_Gt" << ',';
        DataFile.width(Width); DataFile << "X_IP3" << ',';
        DataFile.width(Width); DataFile << "X_P" << ',';
        DataFile.width(Width); DataFile << "X_Pc" << ',';
        DataFile.width(Width); DataFile << "X_Pcg" << ',';
        DataFile.width(Width); DataFile << "X_Pg" << ',';
        DataFile.width(Width); DataFile << "X_R" << ',';
        DataFile.width(Width); DataFile << "X_Rg" << ',';
        DataFile.width(Width); DataFile << "X_Rl" << ',';
        DataFile.width(Width); DataFile << "X_Rlg" << ',';
        DataFile.width(Width); DataFile << "X_Rlgp" << ',';
        DataFile.width(Width); DataFile << "sPAR_Compartment" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kd1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kd11" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kd2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kd4" << ',';
        DataFile.width(Width); DataFile << "sPAR_Km14" << ',';
        DataFile.width(Width); DataFile << "sPAR_Km15" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ls" << ',';
        DataFile.width(Width); DataFile << "sPAR_PIP2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Rpc" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vc" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf10" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf11" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf12" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf13" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf14" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf15" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf16" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf2" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf4" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf5" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf6" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf7" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf8" << ',';
        DataFile.width(Width); DataFile << "sPAR_kf9" << ',';
        DataFile.width(Width); DataFile << "sPAR_kr10" << ',';
        DataFile.width(Width); DataFile << "sPAR_kr3" << ',';
        DataFile.width(Width); DataFile << "sPAR_kr8" << ',';
        DataFile.width(Width); DataFile << "sPAR_kr9" << ',';
        DataFile.width(Width); DataFile << "sPAR_ts" << ',';
        DataFile.width(Width); DataFile << "ACC_Cc" << ',';
        DataFile.width(Width); DataFile << "ACC_Cp" << ',';
        DataFile.width(Width); DataFile << "ACC_Cpc" << ',';
        DataFile.width(Width); DataFile << "ACC_J1" << ',';
        DataFile.width(Width); DataFile << "ACC_J10" << ',';
        DataFile.width(Width); DataFile << "ACC_J11" << ',';
        DataFile.width(Width); DataFile << "ACC_J12" << ',';
        DataFile.width(Width); DataFile << "ACC_J13" << ',';
        DataFile.width(Width); DataFile << "ACC_J14" << ',';
        DataFile.width(Width); DataFile << "ACC_J15" << ',';
        DataFile.width(Width); DataFile << "ACC_J16" << ',';
        DataFile.width(Width); DataFile << "ACC_J2" << ',';
        DataFile.width(Width); DataFile << "ACC_J3" << ',';
        DataFile.width(Width); DataFile << "ACC_J4" << ',';
        DataFile.width(Width); DataFile << "ACC_J5" << ',';
        DataFile.width(Width); DataFile << "ACC_J6" << ',';
        DataFile.width(Width); DataFile << "ACC_J7" << ',';
        DataFile.width(Width); DataFile << "ACC_J8" << ',';
        DataFile.width(Width); DataFile << "ACC_J9" << ',';
        DataFile.width(Width); DataFile << "ACC_L" << ',';
        DataFile.width(Width); DataFile << "ACC_kr1" << ',';
        DataFile.width(Width); DataFile << "ACC_kr11" << ',';
        DataFile.width(Width); DataFile << "ACC_kr2" << ',';
        DataFile.width(Width); DataFile << "ACC_kr4" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 23) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD400.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

