#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#include "BIOMD452_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45
#define PRECISION double
const int NT   = 1;
const int SD   = 108;
const int NCP  = 1;
const int NSP  = 0;
const int NISP = 0;
const int NE   = 0;
const int NA   = 1;
const int NIA  = 0;
const int NDO  = 1000000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 1000.0);  

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        ProblemNumber++;
        k_begin++;
    }
}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_mw014cc419_b720_4b90_9192_2ec6e706c87d" << ',';
        DataFile.width(Width); DataFile << "X_mw06b8aada_c92a_48eb_8ee7_af3778cfe62f" << ',';
        DataFile.width(Width); DataFile << "X_mw0834731b_0477_4217_a53b_30cef851191b" << ',';
        DataFile.width(Width); DataFile << "X_mw0a0ca6ba_cb28_44c7_a0c0_1593cb720966" << ',';
        DataFile.width(Width); DataFile << "X_mw0dc4e5eb_4366_4799_bebc_cfcffe5c06f5" << ',';
        DataFile.width(Width); DataFile << "X_mw0e1be972_fded_4bff_a93d_091ec942485f" << ',';
        DataFile.width(Width); DataFile << "X_mw0facb8f2_95cf_4ddf_a959_b24ba64f320b" << ',';
        DataFile.width(Width); DataFile << "X_mw1093b3af_1864_4ba3_a541_6009a9921282" << ',';
        DataFile.width(Width); DataFile << "X_mw11a8b702_b8ac_4513_b4aa_063e51089812" << ',';
        DataFile.width(Width); DataFile << "X_mw12ba4000_d452_420c_be63_96d2848aca32" << ',';
        DataFile.width(Width); DataFile << "X_mw13abe2a6_9905_40e5_8c23_3fc8834b572a" << ',';
        DataFile.width(Width); DataFile << "X_mw142e6dc4_ec15_459d_a184_6b20be04f08d" << ',';
        DataFile.width(Width); DataFile << "X_mw16796ffe_4764_4a9f_942e_149f42c1cd28" << ',';
        DataFile.width(Width); DataFile << "X_mw19122f7d_f92e_4dc0_922f_6b681db65b0b" << ',';
        DataFile.width(Width); DataFile << "X_mw19a33ad5_5ba4_46c7_84eb_c1287f02bcd5" << ',';
        DataFile.width(Width); DataFile << "X_mw1a0cb97a_b657_430b_963c_92217f643081" << ',';
        DataFile.width(Width); DataFile << "X_mw1d5948e7_5504_4224_9d71_227911b4f1ee" << ',';
        DataFile.width(Width); DataFile << "X_mw1e591998_65c0_484e_8a3b_537a38d94de1" << ',';
        DataFile.width(Width); DataFile << "X_mw236a3250_4c96_4f6e_b94c_ab3d12852801" << ',';
        DataFile.width(Width); DataFile << "X_mw28464aad_8013_4a23_ae09_a406954859a6" << ',';
        DataFile.width(Width); DataFile << "X_mw2ba1db9a_4483_44fa_a3a2_b4a5ea66898c" << ',';
        DataFile.width(Width); DataFile << "X_mw2c47ae3f_06d9_40ec_a252_535db0ae5caa" << ',';
        DataFile.width(Width); DataFile << "X_mw2fd710a6_7fe2_4484_bca6_59c187bade8b" << ',';
        DataFile.width(Width); DataFile << "X_mw31261227_9cd6_4059_a0bb_04dbf4888080" << ',';
        DataFile.width(Width); DataFile << "X_mw31ac308f_da36_4f73_830f_67f3e5b945d9" << ',';
        DataFile.width(Width); DataFile << "X_mw341082a0_8017_4cc7_9d00_b1211a196072" << ',';
        DataFile.width(Width); DataFile << "X_mw35f5adaa_d1c0_433c_817d_76e317f4cb15" << ',';
        DataFile.width(Width); DataFile << "X_mw3c2e1b43_29ca_491a_93e9_c723a993d6fb" << ',';
        DataFile.width(Width); DataFile << "X_mw3d81860d_d786_4fcc_b8bb_64f1a2d7739d" << ',';
        DataFile.width(Width); DataFile << "X_mw4110f531_7513_4786_8896_7c9d969ff558" << ',';
        DataFile.width(Width); DataFile << "X_mw45ab688a_6467_4a3e_a779_2118fa84d69e" << ',';
        DataFile.width(Width); DataFile << "X_mw4628f984_eb87_4922_9760_4975095ce6eb" << ',';
        DataFile.width(Width); DataFile << "X_mw472d5cb9_120e_4f60_bbae_1ae2552837dd" << ',';
        DataFile.width(Width); DataFile << "X_mw481cd12b_61ba_44e5_93bf_8b88c6c4a4e7" << ',';
        DataFile.width(Width); DataFile << "X_mw4f575c55_7dff_45d7_94ad_cda9621d5b63" << ',';
        DataFile.width(Width); DataFile << "X_mw504578d8_96c3_471f_8a7e_8c14e7535d3d" << ',';
        DataFile.width(Width); DataFile << "X_mw5198d3c2_879c_4f0d_b4f8_cd40efe0b1cf" << ',';
        DataFile.width(Width); DataFile << "X_mw548c81c2_c626_4df8_9177_a1a6fc3d4ce8" << ',';
        DataFile.width(Width); DataFile << "X_mw57a44eb0_ace7_4294_905a_219e87d3c281" << ',';
        DataFile.width(Width); DataFile << "X_mw5babe3d5_a9af_4dfd_ac01_35474ef64af2" << ',';
        DataFile.width(Width); DataFile << "X_mw62bf5275_ce02_4e86_b3b6_3f87a335e1de" << ',';
        DataFile.width(Width); DataFile << "X_mw6353aa36_d4a4_4254_8a1f_1f7f571d4233" << ',';
        DataFile.width(Width); DataFile << "X_mw66ac98c4_7e7b_4071_954d_43eb17584220" << ',';
        DataFile.width(Width); DataFile << "X_mw6cb74b27_ffef_49bb_8ffb_622d552caa9e" << ',';
        DataFile.width(Width); DataFile << "X_mw6e01967b_3e2a_433d_bec6_9f9cf3ba243c" << ',';
        DataFile.width(Width); DataFile << "X_mw7033dfd6_53c5_433b_a132_f8cb34dea20f" << ',';
        DataFile.width(Width); DataFile << "X_mw78e207c4_4faf_4b48_8e22_1ee666e9cc4c" << ',';
        DataFile.width(Width); DataFile << "X_mw7cff9a0e_094d_498e_bf7f_7b162c61d63a" << ',';
        DataFile.width(Width); DataFile << "X_mw7e23b961_186b_47a0_a8b5_5e9957766792" << ',';
        DataFile.width(Width); DataFile << "X_mw7eacabf9_d68c_491a_aba2_ec0809a8ecc8" << ',';
        DataFile.width(Width); DataFile << "X_mw83de7813_4941_45a6_a320_a551165bf22a" << ',';
        DataFile.width(Width); DataFile << "X_mw8c85ff7f_6368_4b11_a2ed_ce83481b55e6" << ',';
        DataFile.width(Width); DataFile << "X_mw8f5a7b5c_ca4c_4a4c_85b1_e5d640c426bf" << ',';
        DataFile.width(Width); DataFile << "X_mw925b938a_fe73_4664_ba6f_e72e57780891" << ',';
        DataFile.width(Width); DataFile << "X_mw93907b2d_53db_4080_9e3f_3eb304441ab9" << ',';
        DataFile.width(Width); DataFile << "X_mw954e8fcb_ac0a_459d_8878_f19080208a17" << ',';
        DataFile.width(Width); DataFile << "X_mw960bddeb_e567_46dd_b2f3_ed5e6a5c7972" << ',';
        DataFile.width(Width); DataFile << "X_mw9686f53e_d343_45fd_b441_9c992219546a" << ',';
        DataFile.width(Width); DataFile << "X_mw9b25f809_18a1_4c14_8f4b_cf18e6d93c28" << ',';
        DataFile.width(Width); DataFile << "X_mw9b937ca3_0d82_46d5_8f5a_0f9701002797" << ',';
        DataFile.width(Width); DataFile << "X_mw9dcaa655_a755_426e_a3fa_1ad7c3c45575" << ',';
        DataFile.width(Width); DataFile << "X_mwa0349407_8187_48fc_9e94_5698ccc4e06d" << ',';
        DataFile.width(Width); DataFile << "X_mwa0acc0ac_5fac_4a42_a3be_e36db44994b0" << ',';
        DataFile.width(Width); DataFile << "X_mwa455ec7e_1a12_4659_95a2_a5695d09ca60" << ',';
        DataFile.width(Width); DataFile << "X_mwa54a9c38_c98b_45e5_8432_4119fb777e44" << ',';
        DataFile.width(Width); DataFile << "X_mwa6e82fc9_a0ce_461c_93c8_17f3c807c1a1" << ',';
        DataFile.width(Width); DataFile << "X_mwa7e3103a_6394_472c_b0f4_8ed527f68604" << ',';
        DataFile.width(Width); DataFile << "X_mwa8f2e7b2_0927_4ab4_a817_dddc43bb4fa3" << ',';
        DataFile.width(Width); DataFile << "X_mwa98802cb_c977_4fe0_9e67_5000904c2c36" << ',';
        DataFile.width(Width); DataFile << "X_mwaff92910_ed3d_40b9_a29c_e4866167e828" << ',';
        DataFile.width(Width); DataFile << "X_mwb1bc2058_e6d8_4680_9e6c_d27bb366cde0" << ',';
        DataFile.width(Width); DataFile << "X_mwb2366216_0b3c_4f28_8303_fec92c68dd57" << ',';
        DataFile.width(Width); DataFile << "X_mwb561d9f3_a9ed_4bdb_8d40_87be5cc3237a" << ',';
        DataFile.width(Width); DataFile << "X_mwb6a9aa2c_62e7_410f_9c33_dbe36dfcc4af" << ',';
        DataFile.width(Width); DataFile << "X_mwbaaeb210_4806_4076_9d60_219f4ed945b6" << ',';
        DataFile.width(Width); DataFile << "X_mwbd6bb050_89bd_41df_8cea_d2e1fb77bafe" << ',';
        DataFile.width(Width); DataFile << "X_mwbf5cb039_b830_4282_aa22_a3dda6272ec1" << ',';
        DataFile.width(Width); DataFile << "X_mwbfcf6773_1915_432c_b1d2_1f246094cc74" << ',';
        DataFile.width(Width); DataFile << "X_mwc1935afc_56b1_4a87_923c_ae6d82455d80" << ',';
        DataFile.width(Width); DataFile << "X_mwcb572fe2_c3ac_40e7_8141_da7d55fce18a" << ',';
        DataFile.width(Width); DataFile << "X_mwcc894c94_0ddf_42cc_913e_cdcc4d471d94" << ',';
        DataFile.width(Width); DataFile << "X_mwcea1f1c1_2f85_4af1_98ea_ef14cf580c09" << ',';
        DataFile.width(Width); DataFile << "X_mwcedf8ecd_67bd_4b91_aa04_d58782dec2a4" << ',';
        DataFile.width(Width); DataFile << "X_mwcef73e0e_d195_4077_ae71_723664ee1602" << ',';
        DataFile.width(Width); DataFile << "X_mwd087f76b_65dc_47f1_ba21_c43774457686" << ',';
        DataFile.width(Width); DataFile << "X_mwd2c465fb_eea7_499a_8ea4_f318a64cb9ee" << ',';
        DataFile.width(Width); DataFile << "X_mwd32d108b_49c2_4df2_9b67_d6c6b84f54b9" << ',';
        DataFile.width(Width); DataFile << "X_mwd39388fd_4f85_4d1c_b2a3_37857c595a2d" << ',';
        DataFile.width(Width); DataFile << "X_mwd746a5d5_5e65_4a4c_9f84_0e4a3cb7d2fc" << ',';
        DataFile.width(Width); DataFile << "X_mwd784228d_0cb5_468a_ac70_02d8f04b3d9c" << ',';
        DataFile.width(Width); DataFile << "X_mwd7bf31ba_b05c_4c45_bb2f_6a2468a2a507" << ',';
        DataFile.width(Width); DataFile << "X_mwd7f41594_8377_4e2e_9528_45d5a82ffdb4" << ',';
        DataFile.width(Width); DataFile << "X_mwd9462e5b_a272_4b66_ab66_fde9266b1a43" << ',';
        DataFile.width(Width); DataFile << "X_mwdc34472c_a6f9_4002_951d_e0e8da64eb42" << ',';
        DataFile.width(Width); DataFile << "X_mwdf82303e_323f_4c51_a858_56a59233cd98" << ',';
        DataFile.width(Width); DataFile << "X_mwdf92bdc0_f426_45b0_9ad0_876521f41312" << ',';
        DataFile.width(Width); DataFile << "X_mwe2fff28d_182c_4a1c_9882_f17774c0958a" << ',';
        DataFile.width(Width); DataFile << "X_mwe3fd7f65_b0d1_44d9_b6f3_d2f7d332f664" << ',';
        DataFile.width(Width); DataFile << "X_mwe57c3282_5935_405c_8c0b_7fadb7a5de17" << ',';
        DataFile.width(Width); DataFile << "X_mwec1b368b_8f73_47eb_9636_9956389836eb" << ',';
        DataFile.width(Width); DataFile << "X_mwf40d6176_abfc_4a30_886f_83a19fcffc48" << ',';
        DataFile.width(Width); DataFile << "X_mwf430a579_ecbf_48ba_80c2_06e455808f2a" << ',';
        DataFile.width(Width); DataFile << "X_mwf816df4c_4593_4d23_990f_0d7c15ddde5d" << ',';
        DataFile.width(Width); DataFile << "X_mwf8cc7834_bf4f_4ccd_8235_d0890badf0f6" << ',';
        DataFile.width(Width); DataFile << "X_mwf9999977_6f0e_4e35_9b73_75587f3448e9" << ',';
        DataFile.width(Width); DataFile << "X_mwf9e2a044_7774_400b_a74e_a111b4a21f30" << ',';
        DataFile.width(Width); DataFile << "X_mwfbda4e09_0cbb_49bc_ae69_f88b7a79ed21" << ',';
        DataFile.width(Width); DataFile << "X_mwfc4a9c3d_3ebb_4033_8b7d_f4d7613d2078" << ',';
        DataFile.width(Width); DataFile << "ACC_mwa6994523_5d45_4000_af0c_3e94073bf183" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 83) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 84) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 85) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 86) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 87) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 88) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 89) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 90) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 91) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 92) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 93) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 94) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 95) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 96) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 97) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 98) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 99) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 100) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 101) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 102) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 103) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 104) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 105) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 106) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 107) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 42, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 43, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 44, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 45, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 46, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 47, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 48, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 49, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 50, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 51, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 52, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 53, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 54, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 55, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 56, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 57, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 58, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 59, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 60, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 61, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 62, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 63, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 64, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 65, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 66, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 67, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 68, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 69, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 70, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 71, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 72, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 73, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 74, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 75, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 76, 1e-10);
    Solver.SolverOption(AbsoluteTolerance, 77, 1e-10);
    Solver.SolverOption(AbsoluteTolerance, 78, 1e-10);
    Solver.SolverOption(AbsoluteTolerance, 79, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 80, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 81, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 82, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 83, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 84, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 85, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 86, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 87, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 88, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 89, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 90, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 91, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 92, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 93, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 94, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 95, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 96, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 97, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 98, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 99, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 100, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 101, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 102, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 103, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 104, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 105, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 106, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 107, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
    Solver.SolverOption(RelativeTolerance, 42, 1e-06);
    Solver.SolverOption(RelativeTolerance, 43, 1e-06);
    Solver.SolverOption(RelativeTolerance, 44, 1e-06);
    Solver.SolverOption(RelativeTolerance, 45, 1e-06);
    Solver.SolverOption(RelativeTolerance, 46, 1e-06);
    Solver.SolverOption(RelativeTolerance, 47, 1e-06);
    Solver.SolverOption(RelativeTolerance, 48, 1e-06);
    Solver.SolverOption(RelativeTolerance, 49, 1e-06);
    Solver.SolverOption(RelativeTolerance, 50, 1e-06);
    Solver.SolverOption(RelativeTolerance, 51, 1e-06);
    Solver.SolverOption(RelativeTolerance, 52, 1e-06);
    Solver.SolverOption(RelativeTolerance, 53, 1e-06);
    Solver.SolverOption(RelativeTolerance, 54, 1e-06);
    Solver.SolverOption(RelativeTolerance, 55, 1e-06);
    Solver.SolverOption(RelativeTolerance, 56, 1e-06);
    Solver.SolverOption(RelativeTolerance, 57, 1e-06);
    Solver.SolverOption(RelativeTolerance, 58, 1e-06);
    Solver.SolverOption(RelativeTolerance, 59, 1e-06);
    Solver.SolverOption(RelativeTolerance, 60, 1e-06);
    Solver.SolverOption(RelativeTolerance, 61, 1e-06);
    Solver.SolverOption(RelativeTolerance, 62, 1e-06);
    Solver.SolverOption(RelativeTolerance, 63, 1e-06);
    Solver.SolverOption(RelativeTolerance, 64, 1e-06);
    Solver.SolverOption(RelativeTolerance, 65, 1e-06);
    Solver.SolverOption(RelativeTolerance, 66, 1e-06);
    Solver.SolverOption(RelativeTolerance, 67, 1e-06);
    Solver.SolverOption(RelativeTolerance, 68, 1e-06);
    Solver.SolverOption(RelativeTolerance, 69, 1e-06);
    Solver.SolverOption(RelativeTolerance, 70, 1e-06);
    Solver.SolverOption(RelativeTolerance, 71, 1e-06);
    Solver.SolverOption(RelativeTolerance, 72, 1e-06);
    Solver.SolverOption(RelativeTolerance, 73, 1e-06);
    Solver.SolverOption(RelativeTolerance, 74, 1e-06);
    Solver.SolverOption(RelativeTolerance, 75, 1e-06);
    Solver.SolverOption(RelativeTolerance, 76, 1e-06);
    Solver.SolverOption(RelativeTolerance, 77, 1e-06);
    Solver.SolverOption(RelativeTolerance, 78, 1e-06);
    Solver.SolverOption(RelativeTolerance, 79, 1e-06);
    Solver.SolverOption(RelativeTolerance, 80, 1e-06);
    Solver.SolverOption(RelativeTolerance, 81, 1e-06);
    Solver.SolverOption(RelativeTolerance, 82, 1e-06);
    Solver.SolverOption(RelativeTolerance, 83, 1e-06);
    Solver.SolverOption(RelativeTolerance, 84, 1e-06);
    Solver.SolverOption(RelativeTolerance, 85, 1e-06);
    Solver.SolverOption(RelativeTolerance, 86, 1e-06);
    Solver.SolverOption(RelativeTolerance, 87, 1e-06);
    Solver.SolverOption(RelativeTolerance, 88, 1e-06);
    Solver.SolverOption(RelativeTolerance, 89, 1e-06);
    Solver.SolverOption(RelativeTolerance, 90, 1e-06);
    Solver.SolverOption(RelativeTolerance, 91, 1e-06);
    Solver.SolverOption(RelativeTolerance, 92, 1e-06);
    Solver.SolverOption(RelativeTolerance, 93, 1e-06);
    Solver.SolverOption(RelativeTolerance, 94, 1e-06);
    Solver.SolverOption(RelativeTolerance, 95, 1e-06);
    Solver.SolverOption(RelativeTolerance, 96, 1e-06);
    Solver.SolverOption(RelativeTolerance, 97, 1e-06);
    Solver.SolverOption(RelativeTolerance, 98, 1e-06);
    Solver.SolverOption(RelativeTolerance, 99, 1e-06);
    Solver.SolverOption(RelativeTolerance, 100, 1e-06);
    Solver.SolverOption(RelativeTolerance, 101, 1e-06);
    Solver.SolverOption(RelativeTolerance, 102, 1e-06);
    Solver.SolverOption(RelativeTolerance, 103, 1e-06);
    Solver.SolverOption(RelativeTolerance, 104, 1e-06);
    Solver.SolverOption(RelativeTolerance, 105, 1e-06);
    Solver.SolverOption(RelativeTolerance, 106, 1e-06);
    Solver.SolverOption(RelativeTolerance, 107, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD452.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

