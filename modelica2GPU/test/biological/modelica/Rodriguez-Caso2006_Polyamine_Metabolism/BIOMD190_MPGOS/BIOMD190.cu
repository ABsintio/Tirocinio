#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD190_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 15;
const int NCP  = 1;
const int NSP  = 51;
const int NISP = 0;
const int NE   = 0;
const int NA   = 2;
const int NIA  = 0;
const int NDO  = 1000000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_A" << ',';
        DataFile.width(Width); DataFile << "X_AcCoA" << ',';
        DataFile.width(Width); DataFile << "X_Antz" << ',';
        DataFile.width(Width); DataFile << "X_CoA" << ',';
        DataFile.width(Width); DataFile << "X_D" << ',';
        DataFile.width(Width); DataFile << "X_Met" << ',';
        DataFile.width(Width); DataFile << "X_ORN" << ',';
        DataFile.width(Width); DataFile << "X_P" << ',';
        DataFile.width(Width); DataFile << "X_S" << ',';
        DataFile.width(Width); DataFile << "X_SAM" << ',';
        DataFile.width(Width); DataFile << "X_Vmaxodc" << ',';
        DataFile.width(Width); DataFile << "X_Vmaxsamdc" << ',';
        DataFile.width(Width); DataFile << "X_Vmaxssat" << ',';
        DataFile.width(Width); DataFile << "X_aD" << ',';
        DataFile.width(Width); DataFile << "X_aS" << ',';
        DataFile.width(Width); DataFile << "sPAR_C" << ',';
        DataFile.width(Width); DataFile << "sPAR_KaSpds_SpdS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kadefflux_aD_efflux" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kapsamdc_SAMdc" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kaspms_SpmS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kdantz" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kdodc" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kdsamdc" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kdspms_SpmS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kdssat" << ',';
        DataFile.width(Width); DataFile << "sPAR_Keq" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kiasamdc_SAMdc" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kiaspds_SpdS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kiaspms_SpmS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kidspds_SpdS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kimetmat_MAT" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kipodc_ODC" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kissamdc_SAMdc" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kisspms_SpmS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmaccoassat_SSAT_for_D" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmaccoassat_SSAT_for_S" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmadpao_PAO_for_aD" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmadpao_PAO_for_aS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmaspao_PAO_for_aD" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmaspao_PAO_for_aS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmcoassat_SSAT_for_D" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmcoassat_SSAT_for_S" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmdpao_PAO_for_aD" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmdpao_PAO_for_aS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmdssat_SSAT_for_D" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmdssat_SSAT_for_S" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmmat_MAT" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmodc_ODC" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmsamdc_SAMdc" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmspao_PAO_for_aD" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmspao_PAO_for_aS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmsssat_SSAT_for_D" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kmsssat_SSAT_for_S" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kpefflux_P_efflux" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kpspds_SpdS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ksantz" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ksodc" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kssamdc" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ksssat" << ',';
        DataFile.width(Width); DataFile << "sPAR_R" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vmmat_MAT" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vmpao_PAO_for_aD" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vmpao_PAO_for_aS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vmspds_SpdS" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vmspms_SpmS" << ',';
        DataFile.width(Width); DataFile << "sPAR_cytosol" << ',';
        DataFile.width(Width); DataFile << "ACC_Kaccoa" << ',';
        DataFile.width(Width); DataFile << "ACC_Kcoa" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD190.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

