#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#include "BIOMD246_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 10000;
const int SD   = 26;
const int NCP  = 1;
const int NSP  = 136;
const int NISP = 0;
const int NE   = 0;
const int NA   = 34;
const int NIA  = 0;
const int NDO  = 1000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_B_C" << ',';
        DataFile.width(Width); DataFile << "X_B_CP" << ',';
        DataFile.width(Width); DataFile << "X_B_N" << ',';
        DataFile.width(Width); DataFile << "X_B_NP" << ',';
        DataFile.width(Width); DataFile << "X_CB" << ',';
        DataFile.width(Width); DataFile << "X_C_C" << ',';
        DataFile.width(Width); DataFile << "X_C_CP" << ',';
        DataFile.width(Width); DataFile << "X_Ca_ex" << ',';
        DataFile.width(Width); DataFile << "X_Ca_in" << ',';
        DataFile.width(Width); DataFile << "X_Ca_store" << ',';
        DataFile.width(Width); DataFile << "X_Cl_ex" << ',';
        DataFile.width(Width); DataFile << "X_Cl_o" << ',';
        DataFile.width(Width); DataFile << "X_GABA_o" << ',';
        DataFile.width(Width); DataFile << "X_I_N" << ',';
        DataFile.width(Width); DataFile << "X_K_ex" << ',';
        DataFile.width(Width); DataFile << "X_M_B" << ',';
        DataFile.width(Width); DataFile << "X_M_C" << ',';
        DataFile.width(Width); DataFile << "X_M_P" << ',';
        DataFile.width(Width); DataFile << "X_Na_ex" << ',';
        DataFile.width(Width); DataFile << "X_PC_C" << ',';
        DataFile.width(Width); DataFile << "X_PC_CP" << ',';
        DataFile.width(Width); DataFile << "X_PC_N" << ',';
        DataFile.width(Width); DataFile << "X_PC_NP" << ',';
        DataFile.width(Width); DataFile << "X_P_C" << ',';
        DataFile.width(Width); DataFile << "X_P_CP" << ',';
        DataFile.width(Width); DataFile << "X_VIP" << ',';
        DataFile.width(Width); DataFile << "sPAR_C_T" << ',';
        DataFile.width(Width); DataFile << "sPAR_Cm" << ',';
        DataFile.width(Width); DataFile << "sPAR_E_K_0" << ',';
        DataFile.width(Width); DataFile << "sPAR_E_L_0" << ',';
        DataFile.width(Width); DataFile << "sPAR_E_Na_0" << ',';
        DataFile.width(Width); DataFile << "sPAR_E_ex" << ',';
        DataFile.width(Width); DataFile << "sPAR_Faraday" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_1_CB" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_2_CB" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_A" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_AP" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_C" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_Ca" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_Cl1" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_Cl2" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_D" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_GABA" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_IB" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_KCa" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_PK" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_R" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_R_Ca" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_VIP" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_dp" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_ex1" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_ex2" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_gk" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_kk" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_mB" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_mC" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_mP" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_p" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_sC" << ',';
        DataFile.width(Width); DataFile << "sPAR_K_vo" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kd" << ',';
        DataFile.width(Width); DataFile << "sPAR_PK_o" << ',';
        DataFile.width(Width); DataFile << "sPAR_P_Ca" << ',';
        DataFile.width(Width); DataFile << "sPAR_P_Cl" << ',';
        DataFile.width(Width); DataFile << "sPAR_P_Na" << ',';
        DataFile.width(Width); DataFile << "sPAR_R_g" << ',';
        DataFile.width(Width); DataFile << "sPAR_T" << ',';
        DataFile.width(Width); DataFile << "sPAR_T_abs" << ',';
        DataFile.width(Width); DataFile << "sPAR_T_room" << ',';
        DataFile.width(Width); DataFile << "sPAR_V1_B" << ',';
        DataFile.width(Width); DataFile << "sPAR_V1_C" << ',';
        DataFile.width(Width); DataFile << "sPAR_V1_P" << ',';
        DataFile.width(Width); DataFile << "sPAR_V1_PC" << ',';
        DataFile.width(Width); DataFile << "sPAR_V2_B" << ',';
        DataFile.width(Width); DataFile << "sPAR_V2_C" << ',';
        DataFile.width(Width); DataFile << "sPAR_V2_P" << ',';
        DataFile.width(Width); DataFile << "sPAR_V2_PC" << ',';
        DataFile.width(Width); DataFile << "sPAR_V3_B" << ',';
        DataFile.width(Width); DataFile << "sPAR_V3_PC" << ',';
        DataFile.width(Width); DataFile << "sPAR_V4_B" << ',';
        DataFile.width(Width); DataFile << "sPAR_V4_PC" << ',';
        DataFile.width(Width); DataFile << "sPAR_VK_Ca" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_M1" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_M2" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_M3" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_MK" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_R" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_b" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_ex1" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_ex2" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_phos" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_theta" << ',';
        DataFile.width(Width); DataFile << "sPAR_WT" << ',';
        DataFile.width(Width); DataFile << "sPAR_beta_IP3" << ',';
        DataFile.width(Width); DataFile << "sPAR_cytoplasm" << ',';
        DataFile.width(Width); DataFile << "sPAR_extra" << ',';
        DataFile.width(Width); DataFile << "sPAR_g_K_0" << ',';
        DataFile.width(Width); DataFile << "sPAR_g_Na" << ',';
        DataFile.width(Width); DataFile << "sPAR_g_inhib" << ',';
        DataFile.width(Width); DataFile << "sPAR_k1" << ',';
        DataFile.width(Width); DataFile << "sPAR_k2" << ',';
        DataFile.width(Width); DataFile << "sPAR_k3" << ',';
        DataFile.width(Width); DataFile << "sPAR_k4" << ',';
        DataFile.width(Width); DataFile << "sPAR_k5" << ',';
        DataFile.width(Width); DataFile << "sPAR_k6" << ',';
        DataFile.width(Width); DataFile << "sPAR_k7" << ',';
        DataFile.width(Width); DataFile << "sPAR_k8" << ',';
        DataFile.width(Width); DataFile << "sPAR_k_MK" << ',';
        DataFile.width(Width); DataFile << "sPAR_k_b" << ',';
        DataFile.width(Width); DataFile << "sPAR_k_dVIP" << ',';
        DataFile.width(Width); DataFile << "sPAR_k_f" << ',';
        DataFile.width(Width); DataFile << "sPAR_k_q" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd_mB" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd_mC" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd_mP" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_kd_nc" << ',';
        DataFile.width(Width); DataFile << "sPAR_ksB" << ',';
        DataFile.width(Width); DataFile << "sPAR_ks_C" << ',';
        DataFile.width(Width); DataFile << "sPAR_ks_P" << ',';
        DataFile.width(Width); DataFile << "sPAR_m_BN" << ',';
        DataFile.width(Width); DataFile << "sPAR_n_BN" << ',';
        DataFile.width(Width); DataFile << "sPAR_n_Ca" << ',';
        DataFile.width(Width); DataFile << "sPAR_n_Cl" << ',';
        DataFile.width(Width); DataFile << "sPAR_n_KCa" << ',';
        DataFile.width(Width); DataFile << "sPAR_n_M2" << ',';
        DataFile.width(Width); DataFile << "sPAR_n_M3" << ',';
        DataFile.width(Width); DataFile << "sPAR_n_PK" << ',';
        DataFile.width(Width); DataFile << "sPAR_n_VIP" << ',';
        DataFile.width(Width); DataFile << "sPAR_n_dVIP" << ',';
        DataFile.width(Width); DataFile << "sPAR_n_ex1" << ',';
        DataFile.width(Width); DataFile << "sPAR_n_ex2" << ',';
        DataFile.width(Width); DataFile << "sPAR_n_kCa" << ',';
        DataFile.width(Width); DataFile << "sPAR_n_kk" << ',';
        DataFile.width(Width); DataFile << "sPAR_n_vo" << ',';
        DataFile.width(Width); DataFile << "sPAR_nucleus" << ',';
        DataFile.width(Width); DataFile << "sPAR_p_A" << ',';
        DataFile.width(Width); DataFile << "sPAR_store" << ',';
        DataFile.width(Width); DataFile << "sPAR_vP" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_Ca" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_Cl1" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_Cl2" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_GABA" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_PK" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_VIP" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_dCC" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_dPC" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_gk" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_kk" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_mB" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_mC" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_mP" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_sB" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_sC" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_sP0" << ',';
        DataFile.width(Width); DataFile << "sPAR_v_vo" << ',';
        DataFile.width(Width); DataFile << "sPAR_vd_BC" << ',';
        DataFile.width(Width); DataFile << "sPAR_vd_BN" << ',';
        DataFile.width(Width); DataFile << "sPAR_vd_IN" << ',';
        DataFile.width(Width); DataFile << "sPAR_vd_PCC" << ',';
        DataFile.width(Width); DataFile << "sPAR_vd_PCN" << ',';
        DataFile.width(Width); DataFile << "ACC_Cl_in" << ',';
        DataFile.width(Width); DataFile << "ACC_E_Ca" << ',';
        DataFile.width(Width); DataFile << "ACC_E_K" << ',';
        DataFile.width(Width); DataFile << "ACC_E_L" << ',';
        DataFile.width(Width); DataFile << "ACC_E_Na" << ',';
        DataFile.width(Width); DataFile << "ACC_E_inhib" << ',';
        DataFile.width(Width); DataFile << "ACC_GABA" << ',';
        DataFile.width(Width); DataFile << "ACC_I_Na" << ',';
        DataFile.width(Width); DataFile << "ACC_I_Na_abs" << ',';
        DataFile.width(Width); DataFile << "ACC_I_star" << ',';
        DataFile.width(Width); DataFile << "ACC_K_in" << ',';
        DataFile.width(Width); DataFile << "ACC_Na_in" << ',';
        DataFile.width(Width); DataFile << "ACC_P_K" << ',';
        DataFile.width(Width); DataFile << "ACC_R" << ',';
        DataFile.width(Width); DataFile << "ACC_R_star" << ',';
        DataFile.width(Width); DataFile << "ACC_V_reset" << ',';
        DataFile.width(Width); DataFile << "ACC_V_rest" << ',';
        DataFile.width(Width); DataFile << "ACC_alpha" << ',';
        DataFile.width(Width); DataFile << "ACC_beta" << ',';
        DataFile.width(Width); DataFile << "ACC_beta_a" << ',';
        DataFile.width(Width); DataFile << "ACC_c" << ',';
        DataFile.width(Width); DataFile << "ACC_f_r" << ',';
        DataFile.width(Width); DataFile << "ACC_gK_Ca" << ',';
        DataFile.width(Width); DataFile << "ACC_g_Ca" << ',';
        DataFile.width(Width); DataFile << "ACC_g_K" << ',';
        DataFile.width(Width); DataFile << "ACC_g_L" << ',';
        DataFile.width(Width); DataFile << "ACC_g_ex" << ',';
        DataFile.width(Width); DataFile << "ACC_psi" << ',';
        DataFile.width(Width); DataFile << "ACC_tau_m" << ',';
        DataFile.width(Width); DataFile << "ACC_theta" << ',';
        DataFile.width(Width); DataFile << "ACC_theta_K" << ',';
        DataFile.width(Width); DataFile << "ACC_theta_Na" << ',';
        DataFile.width(Width); DataFile << "ACC_v_K" << ',';
        DataFile.width(Width); DataFile << "ACC_v_sPc" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 83) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 84) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 85) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 86) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 87) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 88) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 89) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 90) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 91) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 92) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 93) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 94) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 95) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 96) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 97) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 98) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 99) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 100) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 101) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 102) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 103) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 104) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 105) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 106) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 107) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 108) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 109) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 110) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 111) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 112) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 113) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 114) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 115) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 116) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 117) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 118) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 119) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 120) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 121) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 122) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 123) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 124) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 125) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 126) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 127) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 128) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 129) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 130) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 131) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 132) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 133) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 134) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 135) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 33) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-10);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-10);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD246.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

