#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD217_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 22;
const int NCP  = 1;
const int NSP  = 95;
const int NISP = 0;
const int NE   = 0;
const int NA   = 10;
const int NIA  = 0;
const int NDO  = 1000000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_ADP" << ',';
        DataFile.width(Width); DataFile << "X_AMP" << ',';
        DataFile.width(Width); DataFile << "X_ATP" << ',';
        DataFile.width(Width); DataFile << "X_AZGLN" << ',';
        DataFile.width(Width); DataFile << "X_AZGLU" << ',';
        DataFile.width(Width); DataFile << "X_AZgln" << ',';
        DataFile.width(Width); DataFile << "X_AZglu" << ',';
        DataFile.width(Width); DataFile << "X_GLN" << ',';
        DataFile.width(Width); DataFile << "X_GLU" << ',';
        DataFile.width(Width); DataFile << "X_GS" << ',';
        DataFile.width(Width); DataFile << "X_KG" << ',';
        DataFile.width(Width); DataFile << "X_NADP" << ',';
        DataFile.width(Width); DataFile << "X_NADPH" << ',';
        DataFile.width(Width); DataFile << "X_NH4" << ',';
        DataFile.width(Width); DataFile << "X_PII" << ',';
        DataFile.width(Width); DataFile << "X_PIIUMP" << ',';
        DataFile.width(Width); DataFile << "X_PIIUMP2" << ',';
        DataFile.width(Width); DataFile << "X_PIIUMP3" << ',';
        DataFile.width(Width); DataFile << "X_PPi" << ',';
        DataFile.width(Width); DataFile << "X_P_i" << ',';
        DataFile.width(Width); DataFile << "X_UMP" << ',';
        DataFile.width(Width); DataFile << "X_UTP" << ',';
        DataFile.width(Width); DataFile << "sPAR_ATPtot" << ',';
        DataFile.width(Width); DataFile << "sPAR_GStot" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kadgln" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kadgs" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kadp" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kadpiikg" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kazglndemazgln" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kazglndemazinter" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kazglndemeq" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kazgludemazglu" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kazgludemazinter" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kazgludemeq" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kd1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kd1piiump" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kd2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kd2piiump" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kd3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kd3piiump" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kdeadgln" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kdeadgsa" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kdeadpiikg" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kdeadpiiu" << ',';
        DataFile.width(Width); DataFile << "sPAR_Keq" << ',';
        DataFile.width(Width); DataFile << "sPAR_Keqgdh" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgdhazglu" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgdhglu" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgdhkg" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgdhnadp" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgdhnadph" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgdhnh" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kglndemazgln" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kglndemeq" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kglndemgln" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kglnur" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kglnut" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgludemazglu" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgludemeq" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgludemglu" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgogaz" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgoggln" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgogglu" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgogkg" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgognadp" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgognadph" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgsadp" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgsatp" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgseq" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgsgln" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgsglu" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgsnh" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kgspi" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kurpiiump" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kurump" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kutipii" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kutippi" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kutpii" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kutpiiump" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kututp" << ',';
        DataFile.width(Width); DataFile << "sPAR_PIItot" << ',';
        DataFile.width(Width); DataFile << "sPAR_UR" << ',';
        DataFile.width(Width); DataFile << "sPAR_UT" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vad" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vadp" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vazglndem" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vazgludem" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vdead" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vgdh" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vglndem" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vgludem" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vgog" << ',';
        DataFile.width(Width); DataFile << "sPAR_Vgs" << ',';
        DataFile.width(Width); DataFile << "sPAR_a1" << ',';
        DataFile.width(Width); DataFile << "sPAR_aamp" << ',';
        DataFile.width(Width); DataFile << "sPAR_b1" << ',';
        DataFile.width(Width); DataFile << "sPAR_bamp" << ',';
        DataFile.width(Width); DataFile << "sPAR_c1" << ',';
        DataFile.width(Width); DataFile << "sPAR_camp" << ',';
        DataFile.width(Width); DataFile << "sPAR_compartment" << ',';
        DataFile.width(Width); DataFile << "sPAR_d1" << ',';
        DataFile.width(Width); DataFile << "sPAR_damp" << ',';
        DataFile.width(Width); DataFile << "sPAR_e1" << ',';
        DataFile.width(Width); DataFile << "sPAR_f1" << ',';
        DataFile.width(Width); DataFile << "sPAR_g1" << ',';
        DataFile.width(Width); DataFile << "sPAR_h1" << ',';
        DataFile.width(Width); DataFile << "sPAR_i1" << ',';
        DataFile.width(Width); DataFile << "sPAR_j1" << ',';
        DataFile.width(Width); DataFile << "sPAR_k1" << ',';
        DataFile.width(Width); DataFile << "sPAR_kcatur" << ',';
        DataFile.width(Width); DataFile << "sPAR_kcatut" << ',';
        DataFile.width(Width); DataFile << "sPAR_l1" << ',';
        DataFile.width(Width); DataFile << "sPAR_m1" << ',';
        DataFile.width(Width); DataFile << "sPAR_n1" << ',';
        DataFile.width(Width); DataFile << "sPAR_n1amp" << ',';
        DataFile.width(Width); DataFile << "sPAR_n2amp" << ',';
        DataFile.width(Width); DataFile << "sPAR_o1" << ',';
        DataFile.width(Width); DataFile << "ACC_$cse1" << ',';
        DataFile.width(Width); DataFile << "ACC_$cse2" << ',';
        DataFile.width(Width); DataFile << "ACC_$cse3" << ',';
        DataFile.width(Width); DataFile << "ACC_$cse4" << ',';
        DataFile.width(Width); DataFile << "ACC_$cse5" << ',';
        DataFile.width(Width); DataFile << "ACC_$cse6" << ',';
        DataFile.width(Width); DataFile << "ACC_PIIKG1" << ',';
        DataFile.width(Width); DataFile << "ACC_PIIUMP3KG3" << ',';
        DataFile.width(Width); DataFile << "ACC_nAMP" << ',';
        DataFile.width(Width); DataFile << "ACC_vAPP_GS" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 83) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 84) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 85) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 86) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 87) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 88) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 89) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 90) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 91) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 92) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 93) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 94) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD217.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

