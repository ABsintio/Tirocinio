#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD332_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 10000;
const int SD   = 78;
const int NCP  = 1;
const int NSP  = 114;
const int NISP = 0;
const int NE   = 0;
const int NA   = 0;
const int NIA  = 0;
const int NDO  = 100;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_APC_PS_VIIIa_l" << ',';
        DataFile.width(Width); DataFile << "X_APC_PS_Va_l" << ',';
        DataFile.width(Width); DataFile << "X_APC_PS_l" << ',';
        DataFile.width(Width); DataFile << "X_APC_f" << ',';
        DataFile.width(Width); DataFile << "X_APC_l" << ',';
        DataFile.width(Width); DataFile << "X_AT_XIa_l" << ',';
        DataFile.width(Width); DataFile << "X_AT_f" << ',';
        DataFile.width(Width); DataFile << "X_II_f" << ',';
        DataFile.width(Width); DataFile << "X_II_l" << ',';
        DataFile.width(Width); DataFile << "X_IIa_AT_f" << ',';
        DataFile.width(Width); DataFile << "X_IIa_TM_PC_l" << ',';
        DataFile.width(Width); DataFile << "X_IIa_TM_l" << ',';
        DataFile.width(Width); DataFile << "X_IIa_f" << ',';
        DataFile.width(Width); DataFile << "X_IX_f" << ',';
        DataFile.width(Width); DataFile << "X_IX_l" << ',';
        DataFile.width(Width); DataFile << "X_IXa_AT_f" << ',';
        DataFile.width(Width); DataFile << "X_IXa_VIIIa_X_l" << ',';
        DataFile.width(Width); DataFile << "X_IXa_VIIIa_l" << ',';
        DataFile.width(Width); DataFile << "X_IXa_f" << ',';
        DataFile.width(Width); DataFile << "X_IXa_l" << ',';
        DataFile.width(Width); DataFile << "X_LIPID" << ',';
        DataFile.width(Width); DataFile << "X_PC_f" << ',';
        DataFile.width(Width); DataFile << "X_PC_l" << ',';
        DataFile.width(Width); DataFile << "X_PS_f" << ',';
        DataFile.width(Width); DataFile << "X_PS_l" << ',';
        DataFile.width(Width); DataFile << "X_TFPI_Xa_TF_VIIa_l" << ',';
        DataFile.width(Width); DataFile << "X_TFPI_Xa_l" << ',';
        DataFile.width(Width); DataFile << "X_TFPI_f" << ',';
        DataFile.width(Width); DataFile << "X_TF_VII_Xa_l" << ',';
        DataFile.width(Width); DataFile << "X_TF_VII_l" << ',';
        DataFile.width(Width); DataFile << "X_TF_VIIa_IX_l" << ',';
        DataFile.width(Width); DataFile << "X_TF_VIIa_IXa_l" << ',';
        DataFile.width(Width); DataFile << "X_TF_VIIa_X_l" << ',';
        DataFile.width(Width); DataFile << "X_TF_VIIa_Xa_l" << ',';
        DataFile.width(Width); DataFile << "X_TF_VIIa_l" << ',';
        DataFile.width(Width); DataFile << "X_TF_l" << ',';
        DataFile.width(Width); DataFile << "X_TM_l" << ',';
        DataFile.width(Width); DataFile << "X_VIII_IIa_l" << ',';
        DataFile.width(Width); DataFile << "X_VIII_Xa_l" << ',';
        DataFile.width(Width); DataFile << "X_VIII_f" << ',';
        DataFile.width(Width); DataFile << "X_VIII_l" << ',';
        DataFile.width(Width); DataFile << "X_VIII_mIIa_l" << ',';
        DataFile.width(Width); DataFile << "X_VIIIa_f" << ',';
        DataFile.width(Width); DataFile << "X_VIIIa_l" << ',';
        DataFile.width(Width); DataFile << "X_VIIIai_f" << ',';
        DataFile.width(Width); DataFile << "X_VIIIai_l" << ',';
        DataFile.width(Width); DataFile << "X_VII_Xa_l" << ',';
        DataFile.width(Width); DataFile << "X_VII_f" << ',';
        DataFile.width(Width); DataFile << "X_VII_l" << ',';
        DataFile.width(Width); DataFile << "X_VIIa_f" << ',';
        DataFile.width(Width); DataFile << "X_VIIa_l" << ',';
        DataFile.width(Width); DataFile << "X_V_IIa_l" << ',';
        DataFile.width(Width); DataFile << "X_V_Xa_l" << ',';
        DataFile.width(Width); DataFile << "X_V_f" << ',';
        DataFile.width(Width); DataFile << "X_V_l" << ',';
        DataFile.width(Width); DataFile << "X_V_mIIa_l" << ',';
        DataFile.width(Width); DataFile << "X_Va_f" << ',';
        DataFile.width(Width); DataFile << "X_Va_l" << ',';
        DataFile.width(Width); DataFile << "X_Vai_f" << ',';
        DataFile.width(Width); DataFile << "X_Vai_l" << ',';
        DataFile.width(Width); DataFile << "X_XI_IIa_l" << ',';
        DataFile.width(Width); DataFile << "X_XI_f" << ',';
        DataFile.width(Width); DataFile << "X_XIa_IX_l" << ',';
        DataFile.width(Width); DataFile << "X_XIa_l" << ',';
        DataFile.width(Width); DataFile << "X_X_f" << ',';
        DataFile.width(Width); DataFile << "X_X_l" << ',';
        DataFile.width(Width); DataFile << "X_Xa_AT_f" << ',';
        DataFile.width(Width); DataFile << "X_Xa_Va_II_l" << ',';
        DataFile.width(Width); DataFile << "X_Xa_Va_l" << ',';
        DataFile.width(Width); DataFile << "X_Xa_Va_mIIa_l" << ',';
        DataFile.width(Width); DataFile << "X_Xa_f" << ',';
        DataFile.width(Width); DataFile << "X_Xa_l" << ',';
        DataFile.width(Width); DataFile << "X_alpha2M_IIa_l" << ',';
        DataFile.width(Width); DataFile << "X_alpha2M_Xa_l" << ',';
        DataFile.width(Width); DataFile << "X_alpha2M_l" << ',';
        DataFile.width(Width); DataFile << "X_mIIa_AT_l" << ',';
        DataFile.width(Width); DataFile << "X_mIIa_f" << ',';
        DataFile.width(Width); DataFile << "X_mIIa_l" << ',';
        DataFile.width(Width); DataFile << "sPAR_compartment" << ',';
        DataFile.width(Width); DataFile << "sPAR_k1" << ',';
        DataFile.width(Width); DataFile << "sPAR_k10" << ',';
        DataFile.width(Width); DataFile << "sPAR_k11" << ',';
        DataFile.width(Width); DataFile << "sPAR_k12" << ',';
        DataFile.width(Width); DataFile << "sPAR_k13" << ',';
        DataFile.width(Width); DataFile << "sPAR_k14" << ',';
        DataFile.width(Width); DataFile << "sPAR_k15" << ',';
        DataFile.width(Width); DataFile << "sPAR_k16" << ',';
        DataFile.width(Width); DataFile << "sPAR_k17" << ',';
        DataFile.width(Width); DataFile << "sPAR_k18" << ',';
        DataFile.width(Width); DataFile << "sPAR_k19" << ',';
        DataFile.width(Width); DataFile << "sPAR_k2" << ',';
        DataFile.width(Width); DataFile << "sPAR_k20" << ',';
        DataFile.width(Width); DataFile << "sPAR_k21" << ',';
        DataFile.width(Width); DataFile << "sPAR_k22" << ',';
        DataFile.width(Width); DataFile << "sPAR_k23" << ',';
        DataFile.width(Width); DataFile << "sPAR_k24" << ',';
        DataFile.width(Width); DataFile << "sPAR_k25" << ',';
        DataFile.width(Width); DataFile << "sPAR_k26" << ',';
        DataFile.width(Width); DataFile << "sPAR_k27" << ',';
        DataFile.width(Width); DataFile << "sPAR_k28" << ',';
        DataFile.width(Width); DataFile << "sPAR_k29" << ',';
        DataFile.width(Width); DataFile << "sPAR_k3" << ',';
        DataFile.width(Width); DataFile << "sPAR_k30" << ',';
        DataFile.width(Width); DataFile << "sPAR_k31" << ',';
        DataFile.width(Width); DataFile << "sPAR_k32" << ',';
        DataFile.width(Width); DataFile << "sPAR_k33" << ',';
        DataFile.width(Width); DataFile << "sPAR_k34" << ',';
        DataFile.width(Width); DataFile << "sPAR_k35" << ',';
        DataFile.width(Width); DataFile << "sPAR_k36" << ',';
        DataFile.width(Width); DataFile << "sPAR_k37" << ',';
        DataFile.width(Width); DataFile << "sPAR_k38" << ',';
        DataFile.width(Width); DataFile << "sPAR_k39" << ',';
        DataFile.width(Width); DataFile << "sPAR_k4" << ',';
        DataFile.width(Width); DataFile << "sPAR_k40" << ',';
        DataFile.width(Width); DataFile << "sPAR_k41" << ',';
        DataFile.width(Width); DataFile << "sPAR_k42" << ',';
        DataFile.width(Width); DataFile << "sPAR_k43" << ',';
        DataFile.width(Width); DataFile << "sPAR_k44" << ',';
        DataFile.width(Width); DataFile << "sPAR_k45" << ',';
        DataFile.width(Width); DataFile << "sPAR_k46" << ',';
        DataFile.width(Width); DataFile << "sPAR_k47" << ',';
        DataFile.width(Width); DataFile << "sPAR_k48" << ',';
        DataFile.width(Width); DataFile << "sPAR_k49" << ',';
        DataFile.width(Width); DataFile << "sPAR_k5" << ',';
        DataFile.width(Width); DataFile << "sPAR_k50" << ',';
        DataFile.width(Width); DataFile << "sPAR_k51" << ',';
        DataFile.width(Width); DataFile << "sPAR_k52" << ',';
        DataFile.width(Width); DataFile << "sPAR_k53" << ',';
        DataFile.width(Width); DataFile << "sPAR_k54" << ',';
        DataFile.width(Width); DataFile << "sPAR_k55" << ',';
        DataFile.width(Width); DataFile << "sPAR_k56" << ',';
        DataFile.width(Width); DataFile << "sPAR_k57" << ',';
        DataFile.width(Width); DataFile << "sPAR_k58" << ',';
        DataFile.width(Width); DataFile << "sPAR_k59" << ',';
        DataFile.width(Width); DataFile << "sPAR_k6" << ',';
        DataFile.width(Width); DataFile << "sPAR_k60" << ',';
        DataFile.width(Width); DataFile << "sPAR_k61" << ',';
        DataFile.width(Width); DataFile << "sPAR_k62" << ',';
        DataFile.width(Width); DataFile << "sPAR_k63" << ',';
        DataFile.width(Width); DataFile << "sPAR_k64" << ',';
        DataFile.width(Width); DataFile << "sPAR_k65" << ',';
        DataFile.width(Width); DataFile << "sPAR_k66" << ',';
        DataFile.width(Width); DataFile << "sPAR_k67" << ',';
        DataFile.width(Width); DataFile << "sPAR_k68" << ',';
        DataFile.width(Width); DataFile << "sPAR_k69" << ',';
        DataFile.width(Width); DataFile << "sPAR_k7" << ',';
        DataFile.width(Width); DataFile << "sPAR_k70" << ',';
        DataFile.width(Width); DataFile << "sPAR_k71" << ',';
        DataFile.width(Width); DataFile << "sPAR_k72" << ',';
        DataFile.width(Width); DataFile << "sPAR_k73" << ',';
        DataFile.width(Width); DataFile << "sPAR_k74" << ',';
        DataFile.width(Width); DataFile << "sPAR_k75" << ',';
        DataFile.width(Width); DataFile << "sPAR_k76" << ',';
        DataFile.width(Width); DataFile << "sPAR_k77" << ',';
        DataFile.width(Width); DataFile << "sPAR_k78" << ',';
        DataFile.width(Width); DataFile << "sPAR_k8" << ',';
        DataFile.width(Width); DataFile << "sPAR_k9" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffAPC" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffII" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffIX" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffIXa" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffPC" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffPS" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffV" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffVII" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffVIII" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffVIIIa" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffVIIIai" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffVIIa" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffVa" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffVai" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffX" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffXa" << ',';
        DataFile.width(Width); DataFile << "sPAR_koffmIIa" << ',';
        DataFile.width(Width); DataFile << "sPAR_konAPC" << ',';
        DataFile.width(Width); DataFile << "sPAR_konII" << ',';
        DataFile.width(Width); DataFile << "sPAR_konIX" << ',';
        DataFile.width(Width); DataFile << "sPAR_konIXa" << ',';
        DataFile.width(Width); DataFile << "sPAR_konPC" << ',';
        DataFile.width(Width); DataFile << "sPAR_konPS" << ',';
        DataFile.width(Width); DataFile << "sPAR_konV" << ',';
        DataFile.width(Width); DataFile << "sPAR_konVII" << ',';
        DataFile.width(Width); DataFile << "sPAR_konVIII" << ',';
        DataFile.width(Width); DataFile << "sPAR_konVIIIa" << ',';
        DataFile.width(Width); DataFile << "sPAR_konVIIIai" << ',';
        DataFile.width(Width); DataFile << "sPAR_konVIIa" << ',';
        DataFile.width(Width); DataFile << "sPAR_konVa" << ',';
        DataFile.width(Width); DataFile << "sPAR_konVai" << ',';
        DataFile.width(Width); DataFile << "sPAR_konX" << ',';
        DataFile.width(Width); DataFile << "sPAR_konXa" << ',';
        DataFile.width(Width); DataFile << "sPAR_konmIIa" << ',';
        DataFile.width(Width); DataFile << "sPAR_nva" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 83) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 84) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 85) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 86) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 87) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 88) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 89) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 90) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 91) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 92) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 93) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 94) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 95) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 96) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 97) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 98) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 99) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 100) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 101) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 102) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 103) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 104) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 105) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 106) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 107) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 108) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 109) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 110) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 111) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 112) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 113) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 42, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 43, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 44, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 45, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 46, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 47, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 48, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 49, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 50, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 51, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 52, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 53, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 54, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 55, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 56, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 57, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 58, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 59, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 60, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 61, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 62, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 63, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 64, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 65, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 66, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 67, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 68, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 69, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 70, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 71, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 72, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 73, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 74, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 75, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 76, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 77, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
    Solver.SolverOption(RelativeTolerance, 42, 1e-06);
    Solver.SolverOption(RelativeTolerance, 43, 1e-06);
    Solver.SolverOption(RelativeTolerance, 44, 1e-06);
    Solver.SolverOption(RelativeTolerance, 45, 1e-06);
    Solver.SolverOption(RelativeTolerance, 46, 1e-06);
    Solver.SolverOption(RelativeTolerance, 47, 1e-06);
    Solver.SolverOption(RelativeTolerance, 48, 1e-06);
    Solver.SolverOption(RelativeTolerance, 49, 1e-06);
    Solver.SolverOption(RelativeTolerance, 50, 1e-06);
    Solver.SolverOption(RelativeTolerance, 51, 1e-06);
    Solver.SolverOption(RelativeTolerance, 52, 1e-06);
    Solver.SolverOption(RelativeTolerance, 53, 1e-06);
    Solver.SolverOption(RelativeTolerance, 54, 1e-06);
    Solver.SolverOption(RelativeTolerance, 55, 1e-06);
    Solver.SolverOption(RelativeTolerance, 56, 1e-06);
    Solver.SolverOption(RelativeTolerance, 57, 1e-06);
    Solver.SolverOption(RelativeTolerance, 58, 1e-06);
    Solver.SolverOption(RelativeTolerance, 59, 1e-06);
    Solver.SolverOption(RelativeTolerance, 60, 1e-06);
    Solver.SolverOption(RelativeTolerance, 61, 1e-06);
    Solver.SolverOption(RelativeTolerance, 62, 1e-06);
    Solver.SolverOption(RelativeTolerance, 63, 1e-06);
    Solver.SolverOption(RelativeTolerance, 64, 1e-06);
    Solver.SolverOption(RelativeTolerance, 65, 1e-06);
    Solver.SolverOption(RelativeTolerance, 66, 1e-06);
    Solver.SolverOption(RelativeTolerance, 67, 1e-06);
    Solver.SolverOption(RelativeTolerance, 68, 1e-06);
    Solver.SolverOption(RelativeTolerance, 69, 1e-06);
    Solver.SolverOption(RelativeTolerance, 70, 1e-06);
    Solver.SolverOption(RelativeTolerance, 71, 1e-06);
    Solver.SolverOption(RelativeTolerance, 72, 1e-06);
    Solver.SolverOption(RelativeTolerance, 73, 1e-06);
    Solver.SolverOption(RelativeTolerance, 74, 1e-06);
    Solver.SolverOption(RelativeTolerance, 75, 1e-06);
    Solver.SolverOption(RelativeTolerance, 76, 1e-06);
    Solver.SolverOption(RelativeTolerance, 77, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD332.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

