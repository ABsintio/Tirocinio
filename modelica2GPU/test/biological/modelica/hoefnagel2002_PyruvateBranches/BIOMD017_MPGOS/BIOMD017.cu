#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD017_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 19;
const int NCP  = 1;
const int NSP  = 71;
const int NISP = 0;
const int NE   = 0;
const int NA   = 0;
const int NIA  = 0;
const int NDO  = 10000000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 0.2);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_ADP" << ',';
        DataFile.width(Width); DataFile << "X_ATP" << ',';
        DataFile.width(Width); DataFile << "X_Ac" << ',';
        DataFile.width(Width); DataFile << "X_AcCoA" << ',';
        DataFile.width(Width); DataFile << "X_AcLac" << ',';
        DataFile.width(Width); DataFile << "X_AcO" << ',';
        DataFile.width(Width); DataFile << "X_AcP" << ',';
        DataFile.width(Width); DataFile << "X_AcetoinIn" << ',';
        DataFile.width(Width); DataFile << "X_AcetoinOut" << ',';
        DataFile.width(Width); DataFile << "X_Butanediol" << ',';
        DataFile.width(Width); DataFile << "X_CoA" << ',';
        DataFile.width(Width); DataFile << "X_EtOH" << ',';
        DataFile.width(Width); DataFile << "X_NAD" << ',';
        DataFile.width(Width); DataFile << "X_NADH" << ',';
        DataFile.width(Width); DataFile << "X_O2" << ',';
        DataFile.width(Width); DataFile << "X_PO4" << ',';
        DataFile.width(Width); DataFile << "X_halfglucose" << ',';
        DataFile.width(Width); DataFile << "X_lactate" << ',';
        DataFile.width(Width); DataFile << "X_pyruvate" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kac_5" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kaccoa_3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kaccoa_6" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kacet_10" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kacet_11" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kacet_9" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kaclac_8" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kaclac_9" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kaco_6" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kaco_7" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kacp_4" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kacp_5" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kadp_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kadp_5" << ',';
        DataFile.width(Width); DataFile << "sPAR_Katp_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Katp_12" << ',';
        DataFile.width(Width); DataFile << "sPAR_Katp_5" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kbut_11" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kcoa_3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kcoa_6" << ',';
        DataFile.width(Width); DataFile << "sPAR_Keq_11" << ',';
        DataFile.width(Width); DataFile << "sPAR_Keq_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Keq_4" << ',';
        DataFile.width(Width); DataFile << "sPAR_Keq_5" << ',';
        DataFile.width(Width); DataFile << "sPAR_Keq_6" << ',';
        DataFile.width(Width); DataFile << "sPAR_Keq_7" << ',';
        DataFile.width(Width); DataFile << "sPAR_Keq_8" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ketoh_7" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kglc_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ki_3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kiaccoa_4" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kiacp_4" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kicoa_4" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kipi_4" << ',';
        DataFile.width(Width); DataFile << "sPAR_Klac_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Knad_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Knad_11" << ',';
        DataFile.width(Width); DataFile << "sPAR_Knad_13" << ',';
        DataFile.width(Width); DataFile << "sPAR_Knad_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Knad_3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Knad_6" << ',';
        DataFile.width(Width); DataFile << "sPAR_Knad_7" << ',';
        DataFile.width(Width); DataFile << "sPAR_Knadh_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Knadh_11" << ',';
        DataFile.width(Width); DataFile << "sPAR_Knadh_13" << ',';
        DataFile.width(Width); DataFile << "sPAR_Knadh_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Knadh_3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Knadh_6" << ',';
        DataFile.width(Width); DataFile << "sPAR_Knadh_7" << ',';
        DataFile.width(Width); DataFile << "sPAR_Ko_13" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kpi_4" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kpyr_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kpyr_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kpyr_3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kpyr_8" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_1" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_10" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_11" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_12" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_13" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_2" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_3" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_4" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_5" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_6" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_7" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_8" << ',';
        DataFile.width(Width); DataFile << "sPAR_V_9" << ',';
        DataFile.width(Width); DataFile << "sPAR_k_14" << ',';
        DataFile.width(Width); DataFile << "sPAR_n_12" << ',';
        DataFile.width(Width); DataFile << "sPAR_n_8" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 70) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 64; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 1.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD017.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

