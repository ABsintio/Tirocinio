#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD345_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 12;
const int NCP  = 1;
const int NSP  = 19;
const int NISP = 0;
const int NE   = 0;
const int NA   = 41;
const int NIA  = 0;
const int NDO  = 1000000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_I2R" << ',';
        DataFile.width(Width); DataFile << "X_I2Ren" << ',';
        DataFile.width(Width); DataFile << "X_I2Rp" << ',';
        DataFile.width(Width); DataFile << "X_IR" << ',';
        DataFile.width(Width); DataFile << "X_IRPen" << ',';
        DataFile.width(Width); DataFile << "X_IRen" << ',';
        DataFile.width(Width); DataFile << "X_IRp" << ',';
        DataFile.width(Width); DataFile << "X_R" << ',';
        DataFile.width(Width); DataFile << "X_RPen" << ',';
        DataFile.width(Width); DataFile << "X_Ren" << ',';
        DataFile.width(Width); DataFile << "X_Rp" << ',';
        DataFile.width(Width); DataFile << "X_ins" << ',';
        DataFile.width(Width); DataFile << "sPAR_Kpan" << ',';
        DataFile.width(Width); DataFile << "sPAR_Rtotal" << ',';
        DataFile.width(Width); DataFile << "sPAR_bw" << ',';
        DataFile.width(Width); DataFile << "sPAR_compartment1" << ',';
        DataFile.width(Width); DataFile << "sPAR_intk1" << ',';
        DataFile.width(Width); DataFile << "sPAR_intk2" << ',';
        DataFile.width(Width); DataFile << "sPAR_k1ub" << ',';
        DataFile.width(Width); DataFile << "sPAR_k2ub" << ',';
        DataFile.width(Width); DataFile << "sPAR_kins" << ',';
        DataFile.width(Width); DataFile << "sPAR_kins1d" << ',';
        DataFile.width(Width); DataFile << "sPAR_kins1den" << ',';
        DataFile.width(Width); DataFile << "sPAR_kins2d" << ',';
        DataFile.width(Width); DataFile << "sPAR_kins2den" << ',';
        DataFile.width(Width); DataFile << "sPAR_kyd" << ',';
        DataFile.width(Width); DataFile << "sPAR_kyden" << ',';
        DataFile.width(Width); DataFile << "sPAR_kyp" << ',';
        DataFile.width(Width); DataFile << "sPAR_pansec" << ',';
        DataFile.width(Width); DataFile << "sPAR_reck1" << ',';
        DataFile.width(Width); DataFile << "sPAR_rholiver" << ',';
        DataFile.width(Width); DataFile << "ACC_I2RPen" << ',';
        DataFile.width(Width); DataFile << "ACC_Kkidney" << ',';
        DataFile.width(Width); DataFile << "ACC_f1" << ',';
        DataFile.width(Width); DataFile << "ACC_f2" << ',';
        DataFile.width(Width); DataFile << "ACC_f3" << ',';
        DataFile.width(Width); DataFile << "ACC_f4" << ',';
        DataFile.width(Width); DataFile << "ACC_f5" << ',';
        DataFile.width(Width); DataFile << "ACC_f6" << ',';
        DataFile.width(Width); DataFile << "ACC_i1" << ',';
        DataFile.width(Width); DataFile << "ACC_i2" << ',';
        DataFile.width(Width); DataFile << "ACC_i3" << ',';
        DataFile.width(Width); DataFile << "ACC_i4" << ',';
        DataFile.width(Width); DataFile << "ACC_i5" << ',';
        DataFile.width(Width); DataFile << "ACC_i6" << ',';
        DataFile.width(Width); DataFile << "ACC_i7" << ',';
        DataFile.width(Width); DataFile << "ACC_mkidney" << ',';
        DataFile.width(Width); DataFile << "ACC_mliver" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_1" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_10" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_11" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_12" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_13" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_14" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_2" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_3" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_4" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_5" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_6" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_7" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_8" << ',';
        DataFile.width(Width); DataFile << "ACC_parameter_9" << ',';
        DataFile.width(Width); DataFile << "ACC_r1" << ',';
        DataFile.width(Width); DataFile << "ACC_r2" << ',';
        DataFile.width(Width); DataFile << "ACC_r3" << ',';
        DataFile.width(Width); DataFile << "ACC_r4" << ',';
        DataFile.width(Width); DataFile << "ACC_r5" << ',';
        DataFile.width(Width); DataFile << "ACC_r6" << ',';
        DataFile.width(Width); DataFile << "ACC_r7" << ',';
        DataFile.width(Width); DataFile << "ACC_vd" << ',';
        DataFile.width(Width); DataFile << "ACC_vhep" << ',';
        DataFile.width(Width); DataFile << "ACC_vp" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 40) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD345.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

