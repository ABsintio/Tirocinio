#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD015_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 18;
const int NCP  = 1;
const int NSP  = 104;
const int NISP = 0;
const int NE   = 0;
const int NA   = 0;
const int NIA  = 0;
const int NDO  = 1000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 20.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_ATP" << ',';
        DataFile.width(Width); DataFile << "X_Ade" << ',';
        DataFile.width(Width); DataFile << "X_DNA" << ',';
        DataFile.width(Width); DataFile << "X_GTP" << ',';
        DataFile.width(Width); DataFile << "X_Gua" << ',';
        DataFile.width(Width); DataFile << "X_HX" << ',';
        DataFile.width(Width); DataFile << "X_IMP" << ',';
        DataFile.width(Width); DataFile << "X_PRPP" << ',';
        DataFile.width(Width); DataFile << "X_Pi" << ',';
        DataFile.width(Width); DataFile << "X_R5P" << ',';
        DataFile.width(Width); DataFile << "X_RNA" << ',';
        DataFile.width(Width); DataFile << "X_SAM" << ',';
        DataFile.width(Width); DataFile << "X_SAMP" << ',';
        DataFile.width(Width); DataFile << "X_UA" << ',';
        DataFile.width(Width); DataFile << "X_XMP" << ',';
        DataFile.width(Width); DataFile << "X_Xa" << ',';
        DataFile.width(Width); DataFile << "X_dATP" << ',';
        DataFile.width(Width); DataFile << "X_dGTP" << ',';
        DataFile.width(Width); DataFile << "sPAR_aada" << ',';
        DataFile.width(Width); DataFile << "sPAR_aade" << ',';
        DataFile.width(Width); DataFile << "sPAR_aadna" << ',';
        DataFile.width(Width); DataFile << "sPAR_aadrnr" << ',';
        DataFile.width(Width); DataFile << "sPAR_aampd" << ',';
        DataFile.width(Width); DataFile << "sPAR_aaprt" << ',';
        DataFile.width(Width); DataFile << "sPAR_aarna" << ',';
        DataFile.width(Width); DataFile << "sPAR_aasli" << ',';
        DataFile.width(Width); DataFile << "sPAR_aasuc" << ',';
        DataFile.width(Width); DataFile << "sPAR_adada" << ',';
        DataFile.width(Width); DataFile << "sPAR_aden" << ',';
        DataFile.width(Width); DataFile << "sPAR_adgnuc" << ',';
        DataFile.width(Width); DataFile << "sPAR_adnaa" << ',';
        DataFile.width(Width); DataFile << "sPAR_adnag" << ',';
        DataFile.width(Width); DataFile << "sPAR_agdna" << ',';
        DataFile.width(Width); DataFile << "sPAR_agdrnr" << ',';
        DataFile.width(Width); DataFile << "sPAR_agmpr" << ',';
        DataFile.width(Width); DataFile << "sPAR_agmps" << ',';
        DataFile.width(Width); DataFile << "sPAR_agnuc" << ',';
        DataFile.width(Width); DataFile << "sPAR_agprt" << ',';
        DataFile.width(Width); DataFile << "sPAR_agrna" << ',';
        DataFile.width(Width); DataFile << "sPAR_agua" << ',';
        DataFile.width(Width); DataFile << "sPAR_ahprt" << ',';
        DataFile.width(Width); DataFile << "sPAR_ahx" << ',';
        DataFile.width(Width); DataFile << "sPAR_ahxd" << ',';
        DataFile.width(Width); DataFile << "sPAR_aimpd" << ',';
        DataFile.width(Width); DataFile << "sPAR_ainuc" << ',';
        DataFile.width(Width); DataFile << "sPAR_amat" << ',';
        DataFile.width(Width); DataFile << "sPAR_apolyam" << ',';
        DataFile.width(Width); DataFile << "sPAR_aprpps" << ',';
        DataFile.width(Width); DataFile << "sPAR_apyr" << ',';
        DataFile.width(Width); DataFile << "sPAR_arnaa" << ',';
        DataFile.width(Width); DataFile << "sPAR_arnag" << ',';
        DataFile.width(Width); DataFile << "sPAR_atrans" << ',';
        DataFile.width(Width); DataFile << "sPAR_aua" << ',';
        DataFile.width(Width); DataFile << "sPAR_ax" << ',';
        DataFile.width(Width); DataFile << "sPAR_axd" << ',';
        DataFile.width(Width); DataFile << "sPAR_fada4" << ',';
        DataFile.width(Width); DataFile << "sPAR_fade6" << ',';
        DataFile.width(Width); DataFile << "sPAR_fadrnr10" << ',';
        DataFile.width(Width); DataFile << "sPAR_fadrnr4" << ',';
        DataFile.width(Width); DataFile << "sPAR_fadrnr9" << ',';
        DataFile.width(Width); DataFile << "sPAR_fampd18" << ',';
        DataFile.width(Width); DataFile << "sPAR_fampd4" << ',';
        DataFile.width(Width); DataFile << "sPAR_fampd8" << ',';
        DataFile.width(Width); DataFile << "sPAR_faprt1" << ',';
        DataFile.width(Width); DataFile << "sPAR_faprt4" << ',';
        DataFile.width(Width); DataFile << "sPAR_faprt6" << ',';
        DataFile.width(Width); DataFile << "sPAR_fasli3" << ',';
        DataFile.width(Width); DataFile << "sPAR_fasli4" << ',';
        DataFile.width(Width); DataFile << "sPAR_fasuc18" << ',';
        DataFile.width(Width); DataFile << "sPAR_fasuc2" << ',';
        DataFile.width(Width); DataFile << "sPAR_fasuc4" << ',';
        DataFile.width(Width); DataFile << "sPAR_fasuc8" << ',';
        DataFile.width(Width); DataFile << "sPAR_fdada9" << ',';
        DataFile.width(Width); DataFile << "sPAR_fden1" << ',';
        DataFile.width(Width); DataFile << "sPAR_fden18" << ',';
        DataFile.width(Width); DataFile << "sPAR_fden2" << ',';
        DataFile.width(Width); DataFile << "sPAR_fden4" << ',';
        DataFile.width(Width); DataFile << "sPAR_fden8" << ',';
        DataFile.width(Width); DataFile << "sPAR_fdgnuc10" << ',';
        DataFile.width(Width); DataFile << "sPAR_fdnan12" << ',';
        DataFile.width(Width); DataFile << "sPAR_fdnap10" << ',';
        DataFile.width(Width); DataFile << "sPAR_fdnap9" << ',';
        DataFile.width(Width); DataFile << "sPAR_fgdrnr10" << ',';
        DataFile.width(Width); DataFile << "sPAR_fgdrnr8" << ',';
        DataFile.width(Width); DataFile << "sPAR_fgdrnr9" << ',';
        DataFile.width(Width); DataFile << "sPAR_fgmpr2" << ',';
        DataFile.width(Width); DataFile << "sPAR_fgmpr4" << ',';
        DataFile.width(Width); DataFile << "sPAR_fgmpr7" << ',';
        DataFile.width(Width); DataFile << "sPAR_fgmpr8" << ',';
        DataFile.width(Width); DataFile << "sPAR_fgmps4" << ',';
        DataFile.width(Width); DataFile << "sPAR_fgmps7" << ',';
        DataFile.width(Width); DataFile << "sPAR_fgnuc18" << ',';
        DataFile.width(Width); DataFile << "sPAR_fgnuc8" << ',';
        DataFile.width(Width); DataFile << "sPAR_fgprt1" << ',';
        DataFile.width(Width); DataFile << "sPAR_fgprt15" << ',';
        DataFile.width(Width); DataFile << "sPAR_fgprt8" << ',';
        DataFile.width(Width); DataFile << "sPAR_fgua15" << ',';
        DataFile.width(Width); DataFile << "sPAR_fhprt1" << ',';
        DataFile.width(Width); DataFile << "sPAR_fhprt13" << ',';
        DataFile.width(Width); DataFile << "sPAR_fhprt2" << ',';
        DataFile.width(Width); DataFile << "sPAR_fhx13" << ',';
        DataFile.width(Width); DataFile << "sPAR_fhxd13" << ',';
        DataFile.width(Width); DataFile << "sPAR_fimpd2" << ',';
        DataFile.width(Width); DataFile << "sPAR_fimpd7" << ',';
        DataFile.width(Width); DataFile << "sPAR_fimpd8" << ',';
        DataFile.width(Width); DataFile << "sPAR_finuc18" << ',';
        DataFile.width(Width); DataFile << "sPAR_finuc2" << ',';
        DataFile.width(Width); DataFile << "sPAR_fmat4" << ',';
        DataFile.width(Width); DataFile << "sPAR_fmat5" << ',';
        DataFile.width(Width); DataFile << "sPAR_fpolyam5" << ',';
        DataFile.width(Width); DataFile << "sPAR_fprpps1" << ',';
        DataFile.width(Width); DataFile << "sPAR_fprpps17" << ',';
        DataFile.width(Width); DataFile << "sPAR_fprpps18" << ',';
        DataFile.width(Width); DataFile << "sPAR_fprpps4" << ',';
        DataFile.width(Width); DataFile << "sPAR_fprpps8" << ',';
        DataFile.width(Width); DataFile << "sPAR_fpyr1" << ',';
        DataFile.width(Width); DataFile << "sPAR_franp8" << ',';
        DataFile.width(Width); DataFile << "sPAR_frnap4" << ',';
        DataFile.width(Width); DataFile << "sPAR_frnap8" << ',';
        DataFile.width(Width); DataFile << "sPAR_ftrans5" << ',';
        DataFile.width(Width); DataFile << "sPAR_fua16" << ',';
        DataFile.width(Width); DataFile << "sPAR_fxd14" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 83) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 84) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 85) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 86) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 87) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 88) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 89) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 90) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 91) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 92) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 93) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 94) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 95) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 96) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 97) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 98) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 99) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 100) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 101) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 102) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 103) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 64; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 1.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD015.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

