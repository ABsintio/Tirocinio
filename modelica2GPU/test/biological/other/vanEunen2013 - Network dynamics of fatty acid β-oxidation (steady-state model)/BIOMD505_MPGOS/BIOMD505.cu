#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#include "BIOMD505_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45
#define PRECISION double
const int NT   = 1;
const int SD   = 53;
const int NCP  = 1;
const int NSP  = 0;
const int NISP = 0;
const int NE   = 0;
const int NA   = 1;
const int NIA  = 0;
const int NDO  = 1000000.0;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 1.0);  

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        ProblemNumber++;
        k_begin++;
    }
}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_AcetylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C10AcylCarCYT" << ',';
        DataFile.width(Width); DataFile << "X_C10AcylCarMAT" << ',';
        DataFile.width(Width); DataFile << "X_C10AcylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C10EnoylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C10HydroxyacylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C10KetoacylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C12AcylCarCYT" << ',';
        DataFile.width(Width); DataFile << "X_C12AcylCarMAT" << ',';
        DataFile.width(Width); DataFile << "X_C12AcylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C12EnoylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C12HydroxyacylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C12KetoacylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C14AcylCarCYT" << ',';
        DataFile.width(Width); DataFile << "X_C14AcylCarMAT" << ',';
        DataFile.width(Width); DataFile << "X_C14AcylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C14EnoylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C14HydroxyacylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C14KetoacylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C16AcylCarCYT" << ',';
        DataFile.width(Width); DataFile << "X_C16AcylCarMAT" << ',';
        DataFile.width(Width); DataFile << "X_C16AcylCoACYT" << ',';
        DataFile.width(Width); DataFile << "X_C16AcylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C16EnoylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C16HydroxyacylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C16KetoacylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C4AcetoacylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C4AcylCarCYT" << ',';
        DataFile.width(Width); DataFile << "X_C4AcylCarMAT" << ',';
        DataFile.width(Width); DataFile << "X_C4AcylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C4EnoylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C4HydroxyacylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C6AcylCarCYT" << ',';
        DataFile.width(Width); DataFile << "X_C6AcylCarMAT" << ',';
        DataFile.width(Width); DataFile << "X_C6AcylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C6EnoylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C6HydroxyacylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C6KetoacylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C8AcylCarCYT" << ',';
        DataFile.width(Width); DataFile << "X_C8AcylCarMAT" << ',';
        DataFile.width(Width); DataFile << "X_C8AcylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C8EnoylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C8HydroxyacylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_C8KetoacylCoAMAT" << ',';
        DataFile.width(Width); DataFile << "X_CarCYT" << ',';
        DataFile.width(Width); DataFile << "X_CarMAT" << ',';
        DataFile.width(Width); DataFile << "X_CoACYT" << ',';
        DataFile.width(Width); DataFile << "X_CoAMATt" << ',';
        DataFile.width(Width); DataFile << "X_FADHMAT" << ',';
        DataFile.width(Width); DataFile << "X_FADtMAT" << ',';
        DataFile.width(Width); DataFile << "X_MalCoACYT" << ',';
        DataFile.width(Width); DataFile << "X_NADHMAT" << ',';
        DataFile.width(Width); DataFile << "X_NADtMAT" << ',';
        DataFile.width(Width); DataFile << "ACC_CoAMAT" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 10.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.1);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 42, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 43, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 44, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 45, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 46, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 47, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 48, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 49, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 50, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 51, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 52, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
    Solver.SolverOption(RelativeTolerance, 42, 1e-06);
    Solver.SolverOption(RelativeTolerance, 43, 1e-06);
    Solver.SolverOption(RelativeTolerance, 44, 1e-06);
    Solver.SolverOption(RelativeTolerance, 45, 1e-06);
    Solver.SolverOption(RelativeTolerance, 46, 1e-06);
    Solver.SolverOption(RelativeTolerance, 47, 1e-06);
    Solver.SolverOption(RelativeTolerance, 48, 1e-06);
    Solver.SolverOption(RelativeTolerance, 49, 1e-06);
    Solver.SolverOption(RelativeTolerance, 50, 1e-06);
    Solver.SolverOption(RelativeTolerance, 51, 1e-06);
    Solver.SolverOption(RelativeTolerance, 52, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD505.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

