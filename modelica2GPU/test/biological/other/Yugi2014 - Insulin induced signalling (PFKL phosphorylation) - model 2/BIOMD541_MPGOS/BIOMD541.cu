#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#include "BIOMD541_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45
#define PRECISION double
const int NT   = 1;
const int SD   = 43;
const int NCP  = 1;
const int NSP  = 0;
const int NISP = 0;
const int NE   = 7;
const int NA   = 8;
const int NIA  = 0;
const int NDO  = 1000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100.0);  

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        ProblemNumber++;
        k_begin++;
    }
}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_k_akg" << ',';
        DataFile.width(Width); DataFile << "X_k_cit" << ',';
        DataFile.width(Width); DataFile << "X_k_f26bp" << ',';
        DataFile.width(Width); DataFile << "X_k_f6p" << ',';
        DataFile.width(Width); DataFile << "X_k_icit" << ',';
        DataFile.width(Width); DataFile << "X_k_mal" << ',';
        DataFile.width(Width); DataFile << "X_k_pep" << ',';
        DataFile.width(Width); DataFile << "X_s1" << ',';
        DataFile.width(Width); DataFile << "X_s10" << ',';
        DataFile.width(Width); DataFile << "X_s11" << ',';
        DataFile.width(Width); DataFile << "X_s12" << ',';
        DataFile.width(Width); DataFile << "X_s13" << ',';
        DataFile.width(Width); DataFile << "X_s14" << ',';
        DataFile.width(Width); DataFile << "X_s15" << ',';
        DataFile.width(Width); DataFile << "X_s16" << ',';
        DataFile.width(Width); DataFile << "X_s17" << ',';
        DataFile.width(Width); DataFile << "X_s18" << ',';
        DataFile.width(Width); DataFile << "X_s19" << ',';
        DataFile.width(Width); DataFile << "X_s2" << ',';
        DataFile.width(Width); DataFile << "X_s21" << ',';
        DataFile.width(Width); DataFile << "X_s23" << ',';
        DataFile.width(Width); DataFile << "X_s24" << ',';
        DataFile.width(Width); DataFile << "X_s25" << ',';
        DataFile.width(Width); DataFile << "X_s26" << ',';
        DataFile.width(Width); DataFile << "X_s27" << ',';
        DataFile.width(Width); DataFile << "X_s28" << ',';
        DataFile.width(Width); DataFile << "X_s29" << ',';
        DataFile.width(Width); DataFile << "X_s3" << ',';
        DataFile.width(Width); DataFile << "X_s30" << ',';
        DataFile.width(Width); DataFile << "X_s31" << ',';
        DataFile.width(Width); DataFile << "X_s32" << ',';
        DataFile.width(Width); DataFile << "X_s33" << ',';
        DataFile.width(Width); DataFile << "X_s34" << ',';
        DataFile.width(Width); DataFile << "X_s35" << ',';
        DataFile.width(Width); DataFile << "X_s36" << ',';
        DataFile.width(Width); DataFile << "X_s37" << ',';
        DataFile.width(Width); DataFile << "X_s38" << ',';
        DataFile.width(Width); DataFile << "X_s4" << ',';
        DataFile.width(Width); DataFile << "X_s5" << ',';
        DataFile.width(Width); DataFile << "X_s6" << ',';
        DataFile.width(Width); DataFile << "X_s7" << ',';
        DataFile.width(Width); DataFile << "X_s8" << ',';
        DataFile.width(Width); DataFile << "X_s9" << ',';
        DataFile.width(Width); DataFile << "ACC_s22" << ',';
        DataFile.width(Width); DataFile << "ACC_$whenCondition1" << ',';
        DataFile.width(Width); DataFile << "ACC_$whenCondition2" << ',';
        DataFile.width(Width); DataFile << "ACC_$whenCondition3" << ',';
        DataFile.width(Width); DataFile << "ACC_$whenCondition4" << ',';
        DataFile.width(Width); DataFile << "ACC_$whenCondition5" << ',';
        DataFile.width(Width); DataFile << "ACC_$whenCondition6" << ',';
        DataFile.width(Width); DataFile << "ACC_$whenCondition7" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);


    Solver.SolverOption(EventDirection, 0, 0);
    Solver.SolverOption(EventDirection, 1, 0);
    Solver.SolverOption(EventDirection, 2, 0);
    Solver.SolverOption(EventDirection, 3, 0);
    Solver.SolverOption(EventDirection, 4, 0);
    Solver.SolverOption(EventDirection, 5, 0);
    Solver.SolverOption(EventDirection, 6, 0);

    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 42, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
    Solver.SolverOption(RelativeTolerance, 42, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD541.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

