#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#include "TransmissionLineEquations_N_80_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 160;
const int NCP  = 1;
const int NSP  = 9;
const int NISP = 1;
const int NE   = 0;
const int NA   = 2;
const int NIA  = 0;
const int NDO  = 100000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 4e-06);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_cur[1]" << ',';
        DataFile.width(Width); DataFile << "X_cur[2]" << ',';
        DataFile.width(Width); DataFile << "X_cur[3]" << ',';
        DataFile.width(Width); DataFile << "X_cur[4]" << ',';
        DataFile.width(Width); DataFile << "X_cur[5]" << ',';
        DataFile.width(Width); DataFile << "X_cur[6]" << ',';
        DataFile.width(Width); DataFile << "X_cur[7]" << ',';
        DataFile.width(Width); DataFile << "X_cur[8]" << ',';
        DataFile.width(Width); DataFile << "X_cur[9]" << ',';
        DataFile.width(Width); DataFile << "X_cur[10]" << ',';
        DataFile.width(Width); DataFile << "X_cur[11]" << ',';
        DataFile.width(Width); DataFile << "X_cur[12]" << ',';
        DataFile.width(Width); DataFile << "X_cur[13]" << ',';
        DataFile.width(Width); DataFile << "X_cur[14]" << ',';
        DataFile.width(Width); DataFile << "X_cur[15]" << ',';
        DataFile.width(Width); DataFile << "X_cur[16]" << ',';
        DataFile.width(Width); DataFile << "X_cur[17]" << ',';
        DataFile.width(Width); DataFile << "X_cur[18]" << ',';
        DataFile.width(Width); DataFile << "X_cur[19]" << ',';
        DataFile.width(Width); DataFile << "X_cur[20]" << ',';
        DataFile.width(Width); DataFile << "X_cur[21]" << ',';
        DataFile.width(Width); DataFile << "X_cur[22]" << ',';
        DataFile.width(Width); DataFile << "X_cur[23]" << ',';
        DataFile.width(Width); DataFile << "X_cur[24]" << ',';
        DataFile.width(Width); DataFile << "X_cur[25]" << ',';
        DataFile.width(Width); DataFile << "X_cur[26]" << ',';
        DataFile.width(Width); DataFile << "X_cur[27]" << ',';
        DataFile.width(Width); DataFile << "X_cur[28]" << ',';
        DataFile.width(Width); DataFile << "X_cur[29]" << ',';
        DataFile.width(Width); DataFile << "X_cur[30]" << ',';
        DataFile.width(Width); DataFile << "X_cur[31]" << ',';
        DataFile.width(Width); DataFile << "X_cur[32]" << ',';
        DataFile.width(Width); DataFile << "X_cur[33]" << ',';
        DataFile.width(Width); DataFile << "X_cur[34]" << ',';
        DataFile.width(Width); DataFile << "X_cur[35]" << ',';
        DataFile.width(Width); DataFile << "X_cur[36]" << ',';
        DataFile.width(Width); DataFile << "X_cur[37]" << ',';
        DataFile.width(Width); DataFile << "X_cur[38]" << ',';
        DataFile.width(Width); DataFile << "X_cur[39]" << ',';
        DataFile.width(Width); DataFile << "X_cur[40]" << ',';
        DataFile.width(Width); DataFile << "X_cur[41]" << ',';
        DataFile.width(Width); DataFile << "X_cur[42]" << ',';
        DataFile.width(Width); DataFile << "X_cur[43]" << ',';
        DataFile.width(Width); DataFile << "X_cur[44]" << ',';
        DataFile.width(Width); DataFile << "X_cur[45]" << ',';
        DataFile.width(Width); DataFile << "X_cur[46]" << ',';
        DataFile.width(Width); DataFile << "X_cur[47]" << ',';
        DataFile.width(Width); DataFile << "X_cur[48]" << ',';
        DataFile.width(Width); DataFile << "X_cur[49]" << ',';
        DataFile.width(Width); DataFile << "X_cur[50]" << ',';
        DataFile.width(Width); DataFile << "X_cur[51]" << ',';
        DataFile.width(Width); DataFile << "X_cur[52]" << ',';
        DataFile.width(Width); DataFile << "X_cur[53]" << ',';
        DataFile.width(Width); DataFile << "X_cur[54]" << ',';
        DataFile.width(Width); DataFile << "X_cur[55]" << ',';
        DataFile.width(Width); DataFile << "X_cur[56]" << ',';
        DataFile.width(Width); DataFile << "X_cur[57]" << ',';
        DataFile.width(Width); DataFile << "X_cur[58]" << ',';
        DataFile.width(Width); DataFile << "X_cur[59]" << ',';
        DataFile.width(Width); DataFile << "X_cur[60]" << ',';
        DataFile.width(Width); DataFile << "X_cur[61]" << ',';
        DataFile.width(Width); DataFile << "X_cur[62]" << ',';
        DataFile.width(Width); DataFile << "X_cur[63]" << ',';
        DataFile.width(Width); DataFile << "X_cur[64]" << ',';
        DataFile.width(Width); DataFile << "X_cur[65]" << ',';
        DataFile.width(Width); DataFile << "X_cur[66]" << ',';
        DataFile.width(Width); DataFile << "X_cur[67]" << ',';
        DataFile.width(Width); DataFile << "X_cur[68]" << ',';
        DataFile.width(Width); DataFile << "X_cur[69]" << ',';
        DataFile.width(Width); DataFile << "X_cur[70]" << ',';
        DataFile.width(Width); DataFile << "X_cur[71]" << ',';
        DataFile.width(Width); DataFile << "X_cur[72]" << ',';
        DataFile.width(Width); DataFile << "X_cur[73]" << ',';
        DataFile.width(Width); DataFile << "X_cur[74]" << ',';
        DataFile.width(Width); DataFile << "X_cur[75]" << ',';
        DataFile.width(Width); DataFile << "X_cur[76]" << ',';
        DataFile.width(Width); DataFile << "X_cur[77]" << ',';
        DataFile.width(Width); DataFile << "X_cur[78]" << ',';
        DataFile.width(Width); DataFile << "X_cur[79]" << ',';
        DataFile.width(Width); DataFile << "X_vol[1]" << ',';
        DataFile.width(Width); DataFile << "X_vol[2]" << ',';
        DataFile.width(Width); DataFile << "X_vol[3]" << ',';
        DataFile.width(Width); DataFile << "X_vol[4]" << ',';
        DataFile.width(Width); DataFile << "X_vol[5]" << ',';
        DataFile.width(Width); DataFile << "X_vol[6]" << ',';
        DataFile.width(Width); DataFile << "X_vol[7]" << ',';
        DataFile.width(Width); DataFile << "X_vol[8]" << ',';
        DataFile.width(Width); DataFile << "X_vol[9]" << ',';
        DataFile.width(Width); DataFile << "X_vol[10]" << ',';
        DataFile.width(Width); DataFile << "X_vol[11]" << ',';
        DataFile.width(Width); DataFile << "X_vol[12]" << ',';
        DataFile.width(Width); DataFile << "X_vol[13]" << ',';
        DataFile.width(Width); DataFile << "X_vol[14]" << ',';
        DataFile.width(Width); DataFile << "X_vol[15]" << ',';
        DataFile.width(Width); DataFile << "X_vol[16]" << ',';
        DataFile.width(Width); DataFile << "X_vol[17]" << ',';
        DataFile.width(Width); DataFile << "X_vol[18]" << ',';
        DataFile.width(Width); DataFile << "X_vol[19]" << ',';
        DataFile.width(Width); DataFile << "X_vol[20]" << ',';
        DataFile.width(Width); DataFile << "X_vol[21]" << ',';
        DataFile.width(Width); DataFile << "X_vol[22]" << ',';
        DataFile.width(Width); DataFile << "X_vol[23]" << ',';
        DataFile.width(Width); DataFile << "X_vol[24]" << ',';
        DataFile.width(Width); DataFile << "X_vol[25]" << ',';
        DataFile.width(Width); DataFile << "X_vol[26]" << ',';
        DataFile.width(Width); DataFile << "X_vol[27]" << ',';
        DataFile.width(Width); DataFile << "X_vol[28]" << ',';
        DataFile.width(Width); DataFile << "X_vol[29]" << ',';
        DataFile.width(Width); DataFile << "X_vol[30]" << ',';
        DataFile.width(Width); DataFile << "X_vol[31]" << ',';
        DataFile.width(Width); DataFile << "X_vol[32]" << ',';
        DataFile.width(Width); DataFile << "X_vol[33]" << ',';
        DataFile.width(Width); DataFile << "X_vol[34]" << ',';
        DataFile.width(Width); DataFile << "X_vol[35]" << ',';
        DataFile.width(Width); DataFile << "X_vol[36]" << ',';
        DataFile.width(Width); DataFile << "X_vol[37]" << ',';
        DataFile.width(Width); DataFile << "X_vol[38]" << ',';
        DataFile.width(Width); DataFile << "X_vol[39]" << ',';
        DataFile.width(Width); DataFile << "X_vol[40]" << ',';
        DataFile.width(Width); DataFile << "X_vol[41]" << ',';
        DataFile.width(Width); DataFile << "X_vol[42]" << ',';
        DataFile.width(Width); DataFile << "X_vol[43]" << ',';
        DataFile.width(Width); DataFile << "X_vol[44]" << ',';
        DataFile.width(Width); DataFile << "X_vol[45]" << ',';
        DataFile.width(Width); DataFile << "X_vol[46]" << ',';
        DataFile.width(Width); DataFile << "X_vol[47]" << ',';
        DataFile.width(Width); DataFile << "X_vol[48]" << ',';
        DataFile.width(Width); DataFile << "X_vol[49]" << ',';
        DataFile.width(Width); DataFile << "X_vol[50]" << ',';
        DataFile.width(Width); DataFile << "X_vol[51]" << ',';
        DataFile.width(Width); DataFile << "X_vol[52]" << ',';
        DataFile.width(Width); DataFile << "X_vol[53]" << ',';
        DataFile.width(Width); DataFile << "X_vol[54]" << ',';
        DataFile.width(Width); DataFile << "X_vol[55]" << ',';
        DataFile.width(Width); DataFile << "X_vol[56]" << ',';
        DataFile.width(Width); DataFile << "X_vol[57]" << ',';
        DataFile.width(Width); DataFile << "X_vol[58]" << ',';
        DataFile.width(Width); DataFile << "X_vol[59]" << ',';
        DataFile.width(Width); DataFile << "X_vol[60]" << ',';
        DataFile.width(Width); DataFile << "X_vol[61]" << ',';
        DataFile.width(Width); DataFile << "X_vol[62]" << ',';
        DataFile.width(Width); DataFile << "X_vol[63]" << ',';
        DataFile.width(Width); DataFile << "X_vol[64]" << ',';
        DataFile.width(Width); DataFile << "X_vol[65]" << ',';
        DataFile.width(Width); DataFile << "X_vol[66]" << ',';
        DataFile.width(Width); DataFile << "X_vol[67]" << ',';
        DataFile.width(Width); DataFile << "X_vol[68]" << ',';
        DataFile.width(Width); DataFile << "X_vol[69]" << ',';
        DataFile.width(Width); DataFile << "X_vol[70]" << ',';
        DataFile.width(Width); DataFile << "X_vol[71]" << ',';
        DataFile.width(Width); DataFile << "X_vol[72]" << ',';
        DataFile.width(Width); DataFile << "X_vol[73]" << ',';
        DataFile.width(Width); DataFile << "X_vol[74]" << ',';
        DataFile.width(Width); DataFile << "X_vol[75]" << ',';
        DataFile.width(Width); DataFile << "X_vol[76]" << ',';
        DataFile.width(Width); DataFile << "X_vol[77]" << ',';
        DataFile.width(Width); DataFile << "X_vol[78]" << ',';
        DataFile.width(Width); DataFile << "X_vol[79]" << ',';
        DataFile.width(Width); DataFile << "X_vol[80]" << ',';
        DataFile.width(Width); DataFile << "X_vvol" << ',';
        DataFile.width(Width); DataFile << "sPAR_L" << ',';
        DataFile.width(Width); DataFile << "sPAR_RL" << ',';
        DataFile.width(Width); DataFile << "sPAR_TD" << ',';
        DataFile.width(Width); DataFile << "sPAR_cap" << ',';
        DataFile.width(Width); DataFile << "sPAR_ind" << ',';
        DataFile.width(Width); DataFile << "sPAR_l" << ',';
        DataFile.width(Width); DataFile << "sPAR_res" << ',';
        DataFile.width(Width); DataFile << "sPAR_v" << ',';
        DataFile.width(Width); DataFile << "sPAR_w" << ',';
        DataFile.width(Width); DataFile << "sPARi_N" << ',';
        DataFile.width(Width); DataFile << "ACC_Vstep" << ',';
        DataFile.width(Width); DataFile << "ACC_cur[80]" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 83) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 84) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 85) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 86) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 87) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 88) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 89) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 90) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 91) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 92) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 93) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 94) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 95) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 96) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 97) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 98) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 99) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 100) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 101) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 102) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 103) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 104) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 105) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 106) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 107) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 108) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 109) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 110) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 111) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 112) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 113) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 114) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 115) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 116) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 117) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 118) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 119) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 120) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 121) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 122) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 123) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 124) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 125) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 126) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 127) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 128) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 129) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 130) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 131) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 132) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 133) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 134) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 135) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 136) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 137) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 138) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 139) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 140) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 141) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 142) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 143) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 144) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 145) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 146) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 147) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 148) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 149) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 150) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 151) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 152) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 153) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 154) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 155) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 156) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 157) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 158) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 159) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(IntegerSharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-30);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 42, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 43, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 44, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 45, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 46, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 47, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 48, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 49, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 50, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 51, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 52, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 53, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 54, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 55, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 56, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 57, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 58, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 59, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 60, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 61, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 62, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 63, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 64, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 65, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 66, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 67, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 68, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 69, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 70, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 71, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 72, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 73, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 74, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 75, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 76, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 77, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 78, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 79, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 80, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 81, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 82, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 83, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 84, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 85, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 86, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 87, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 88, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 89, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 90, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 91, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 92, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 93, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 94, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 95, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 96, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 97, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 98, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 99, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 100, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 101, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 102, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 103, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 104, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 105, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 106, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 107, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 108, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 109, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 110, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 111, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 112, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 113, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 114, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 115, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 116, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 117, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 118, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 119, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 120, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 121, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 122, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 123, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 124, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 125, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 126, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 127, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 128, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 129, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 130, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 131, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 132, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 133, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 134, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 135, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 136, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 137, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 138, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 139, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 140, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 141, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 142, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 143, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 144, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 145, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 146, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 147, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 148, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 149, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 150, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 151, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 152, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 153, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 154, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 155, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 156, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 157, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 158, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 159, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
    Solver.SolverOption(RelativeTolerance, 42, 1e-06);
    Solver.SolverOption(RelativeTolerance, 43, 1e-06);
    Solver.SolverOption(RelativeTolerance, 44, 1e-06);
    Solver.SolverOption(RelativeTolerance, 45, 1e-06);
    Solver.SolverOption(RelativeTolerance, 46, 1e-06);
    Solver.SolverOption(RelativeTolerance, 47, 1e-06);
    Solver.SolverOption(RelativeTolerance, 48, 1e-06);
    Solver.SolverOption(RelativeTolerance, 49, 1e-06);
    Solver.SolverOption(RelativeTolerance, 50, 1e-06);
    Solver.SolverOption(RelativeTolerance, 51, 1e-06);
    Solver.SolverOption(RelativeTolerance, 52, 1e-06);
    Solver.SolverOption(RelativeTolerance, 53, 1e-06);
    Solver.SolverOption(RelativeTolerance, 54, 1e-06);
    Solver.SolverOption(RelativeTolerance, 55, 1e-06);
    Solver.SolverOption(RelativeTolerance, 56, 1e-06);
    Solver.SolverOption(RelativeTolerance, 57, 1e-06);
    Solver.SolverOption(RelativeTolerance, 58, 1e-06);
    Solver.SolverOption(RelativeTolerance, 59, 1e-06);
    Solver.SolverOption(RelativeTolerance, 60, 1e-06);
    Solver.SolverOption(RelativeTolerance, 61, 1e-06);
    Solver.SolverOption(RelativeTolerance, 62, 1e-06);
    Solver.SolverOption(RelativeTolerance, 63, 1e-06);
    Solver.SolverOption(RelativeTolerance, 64, 1e-06);
    Solver.SolverOption(RelativeTolerance, 65, 1e-06);
    Solver.SolverOption(RelativeTolerance, 66, 1e-06);
    Solver.SolverOption(RelativeTolerance, 67, 1e-06);
    Solver.SolverOption(RelativeTolerance, 68, 1e-06);
    Solver.SolverOption(RelativeTolerance, 69, 1e-06);
    Solver.SolverOption(RelativeTolerance, 70, 1e-06);
    Solver.SolverOption(RelativeTolerance, 71, 1e-06);
    Solver.SolverOption(RelativeTolerance, 72, 1e-06);
    Solver.SolverOption(RelativeTolerance, 73, 1e-06);
    Solver.SolverOption(RelativeTolerance, 74, 1e-06);
    Solver.SolverOption(RelativeTolerance, 75, 1e-06);
    Solver.SolverOption(RelativeTolerance, 76, 1e-06);
    Solver.SolverOption(RelativeTolerance, 77, 1e-06);
    Solver.SolverOption(RelativeTolerance, 78, 1e-06);
    Solver.SolverOption(RelativeTolerance, 79, 1e-06);
    Solver.SolverOption(RelativeTolerance, 80, 1e-06);
    Solver.SolverOption(RelativeTolerance, 81, 1e-06);
    Solver.SolverOption(RelativeTolerance, 82, 1e-06);
    Solver.SolverOption(RelativeTolerance, 83, 1e-06);
    Solver.SolverOption(RelativeTolerance, 84, 1e-06);
    Solver.SolverOption(RelativeTolerance, 85, 1e-06);
    Solver.SolverOption(RelativeTolerance, 86, 1e-06);
    Solver.SolverOption(RelativeTolerance, 87, 1e-06);
    Solver.SolverOption(RelativeTolerance, 88, 1e-06);
    Solver.SolverOption(RelativeTolerance, 89, 1e-06);
    Solver.SolverOption(RelativeTolerance, 90, 1e-06);
    Solver.SolverOption(RelativeTolerance, 91, 1e-06);
    Solver.SolverOption(RelativeTolerance, 92, 1e-06);
    Solver.SolverOption(RelativeTolerance, 93, 1e-06);
    Solver.SolverOption(RelativeTolerance, 94, 1e-06);
    Solver.SolverOption(RelativeTolerance, 95, 1e-06);
    Solver.SolverOption(RelativeTolerance, 96, 1e-06);
    Solver.SolverOption(RelativeTolerance, 97, 1e-06);
    Solver.SolverOption(RelativeTolerance, 98, 1e-06);
    Solver.SolverOption(RelativeTolerance, 99, 1e-06);
    Solver.SolverOption(RelativeTolerance, 100, 1e-06);
    Solver.SolverOption(RelativeTolerance, 101, 1e-06);
    Solver.SolverOption(RelativeTolerance, 102, 1e-06);
    Solver.SolverOption(RelativeTolerance, 103, 1e-06);
    Solver.SolverOption(RelativeTolerance, 104, 1e-06);
    Solver.SolverOption(RelativeTolerance, 105, 1e-06);
    Solver.SolverOption(RelativeTolerance, 106, 1e-06);
    Solver.SolverOption(RelativeTolerance, 107, 1e-06);
    Solver.SolverOption(RelativeTolerance, 108, 1e-06);
    Solver.SolverOption(RelativeTolerance, 109, 1e-06);
    Solver.SolverOption(RelativeTolerance, 110, 1e-06);
    Solver.SolverOption(RelativeTolerance, 111, 1e-06);
    Solver.SolverOption(RelativeTolerance, 112, 1e-06);
    Solver.SolverOption(RelativeTolerance, 113, 1e-06);
    Solver.SolverOption(RelativeTolerance, 114, 1e-06);
    Solver.SolverOption(RelativeTolerance, 115, 1e-06);
    Solver.SolverOption(RelativeTolerance, 116, 1e-06);
    Solver.SolverOption(RelativeTolerance, 117, 1e-06);
    Solver.SolverOption(RelativeTolerance, 118, 1e-06);
    Solver.SolverOption(RelativeTolerance, 119, 1e-06);
    Solver.SolverOption(RelativeTolerance, 120, 1e-06);
    Solver.SolverOption(RelativeTolerance, 121, 1e-06);
    Solver.SolverOption(RelativeTolerance, 122, 1e-06);
    Solver.SolverOption(RelativeTolerance, 123, 1e-06);
    Solver.SolverOption(RelativeTolerance, 124, 1e-06);
    Solver.SolverOption(RelativeTolerance, 125, 1e-06);
    Solver.SolverOption(RelativeTolerance, 126, 1e-06);
    Solver.SolverOption(RelativeTolerance, 127, 1e-06);
    Solver.SolverOption(RelativeTolerance, 128, 1e-06);
    Solver.SolverOption(RelativeTolerance, 129, 1e-06);
    Solver.SolverOption(RelativeTolerance, 130, 1e-06);
    Solver.SolverOption(RelativeTolerance, 131, 1e-06);
    Solver.SolverOption(RelativeTolerance, 132, 1e-06);
    Solver.SolverOption(RelativeTolerance, 133, 1e-06);
    Solver.SolverOption(RelativeTolerance, 134, 1e-06);
    Solver.SolverOption(RelativeTolerance, 135, 1e-06);
    Solver.SolverOption(RelativeTolerance, 136, 1e-06);
    Solver.SolverOption(RelativeTolerance, 137, 1e-06);
    Solver.SolverOption(RelativeTolerance, 138, 1e-06);
    Solver.SolverOption(RelativeTolerance, 139, 1e-06);
    Solver.SolverOption(RelativeTolerance, 140, 1e-06);
    Solver.SolverOption(RelativeTolerance, 141, 1e-06);
    Solver.SolverOption(RelativeTolerance, 142, 1e-06);
    Solver.SolverOption(RelativeTolerance, 143, 1e-06);
    Solver.SolverOption(RelativeTolerance, 144, 1e-06);
    Solver.SolverOption(RelativeTolerance, 145, 1e-06);
    Solver.SolverOption(RelativeTolerance, 146, 1e-06);
    Solver.SolverOption(RelativeTolerance, 147, 1e-06);
    Solver.SolverOption(RelativeTolerance, 148, 1e-06);
    Solver.SolverOption(RelativeTolerance, 149, 1e-06);
    Solver.SolverOption(RelativeTolerance, 150, 1e-06);
    Solver.SolverOption(RelativeTolerance, 151, 1e-06);
    Solver.SolverOption(RelativeTolerance, 152, 1e-06);
    Solver.SolverOption(RelativeTolerance, 153, 1e-06);
    Solver.SolverOption(RelativeTolerance, 154, 1e-06);
    Solver.SolverOption(RelativeTolerance, 155, 1e-06);
    Solver.SolverOption(RelativeTolerance, 156, 1e-06);
    Solver.SolverOption(RelativeTolerance, 157, 1e-06);
    Solver.SolverOption(RelativeTolerance, 158, 1e-06);
    Solver.SolverOption(RelativeTolerance, 159, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "TransmissionLineEquations_N_80.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

