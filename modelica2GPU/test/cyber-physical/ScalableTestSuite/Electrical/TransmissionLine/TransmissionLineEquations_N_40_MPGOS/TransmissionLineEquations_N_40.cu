#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#include "TransmissionLineEquations_N_40_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 80;
const int NCP  = 1;
const int NSP  = 9;
const int NISP = 1;
const int NE   = 0;
const int NA   = 2;
const int NIA  = 0;
const int NDO  = 1000000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 4e-06);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_cur[1]" << ',';
        DataFile.width(Width); DataFile << "X_cur[2]" << ',';
        DataFile.width(Width); DataFile << "X_cur[3]" << ',';
        DataFile.width(Width); DataFile << "X_cur[4]" << ',';
        DataFile.width(Width); DataFile << "X_cur[5]" << ',';
        DataFile.width(Width); DataFile << "X_cur[6]" << ',';
        DataFile.width(Width); DataFile << "X_cur[7]" << ',';
        DataFile.width(Width); DataFile << "X_cur[8]" << ',';
        DataFile.width(Width); DataFile << "X_cur[9]" << ',';
        DataFile.width(Width); DataFile << "X_cur[10]" << ',';
        DataFile.width(Width); DataFile << "X_cur[11]" << ',';
        DataFile.width(Width); DataFile << "X_cur[12]" << ',';
        DataFile.width(Width); DataFile << "X_cur[13]" << ',';
        DataFile.width(Width); DataFile << "X_cur[14]" << ',';
        DataFile.width(Width); DataFile << "X_cur[15]" << ',';
        DataFile.width(Width); DataFile << "X_cur[16]" << ',';
        DataFile.width(Width); DataFile << "X_cur[17]" << ',';
        DataFile.width(Width); DataFile << "X_cur[18]" << ',';
        DataFile.width(Width); DataFile << "X_cur[19]" << ',';
        DataFile.width(Width); DataFile << "X_cur[20]" << ',';
        DataFile.width(Width); DataFile << "X_cur[21]" << ',';
        DataFile.width(Width); DataFile << "X_cur[22]" << ',';
        DataFile.width(Width); DataFile << "X_cur[23]" << ',';
        DataFile.width(Width); DataFile << "X_cur[24]" << ',';
        DataFile.width(Width); DataFile << "X_cur[25]" << ',';
        DataFile.width(Width); DataFile << "X_cur[26]" << ',';
        DataFile.width(Width); DataFile << "X_cur[27]" << ',';
        DataFile.width(Width); DataFile << "X_cur[28]" << ',';
        DataFile.width(Width); DataFile << "X_cur[29]" << ',';
        DataFile.width(Width); DataFile << "X_cur[30]" << ',';
        DataFile.width(Width); DataFile << "X_cur[31]" << ',';
        DataFile.width(Width); DataFile << "X_cur[32]" << ',';
        DataFile.width(Width); DataFile << "X_cur[33]" << ',';
        DataFile.width(Width); DataFile << "X_cur[34]" << ',';
        DataFile.width(Width); DataFile << "X_cur[35]" << ',';
        DataFile.width(Width); DataFile << "X_cur[36]" << ',';
        DataFile.width(Width); DataFile << "X_cur[37]" << ',';
        DataFile.width(Width); DataFile << "X_cur[38]" << ',';
        DataFile.width(Width); DataFile << "X_cur[39]" << ',';
        DataFile.width(Width); DataFile << "X_vol[1]" << ',';
        DataFile.width(Width); DataFile << "X_vol[2]" << ',';
        DataFile.width(Width); DataFile << "X_vol[3]" << ',';
        DataFile.width(Width); DataFile << "X_vol[4]" << ',';
        DataFile.width(Width); DataFile << "X_vol[5]" << ',';
        DataFile.width(Width); DataFile << "X_vol[6]" << ',';
        DataFile.width(Width); DataFile << "X_vol[7]" << ',';
        DataFile.width(Width); DataFile << "X_vol[8]" << ',';
        DataFile.width(Width); DataFile << "X_vol[9]" << ',';
        DataFile.width(Width); DataFile << "X_vol[10]" << ',';
        DataFile.width(Width); DataFile << "X_vol[11]" << ',';
        DataFile.width(Width); DataFile << "X_vol[12]" << ',';
        DataFile.width(Width); DataFile << "X_vol[13]" << ',';
        DataFile.width(Width); DataFile << "X_vol[14]" << ',';
        DataFile.width(Width); DataFile << "X_vol[15]" << ',';
        DataFile.width(Width); DataFile << "X_vol[16]" << ',';
        DataFile.width(Width); DataFile << "X_vol[17]" << ',';
        DataFile.width(Width); DataFile << "X_vol[18]" << ',';
        DataFile.width(Width); DataFile << "X_vol[19]" << ',';
        DataFile.width(Width); DataFile << "X_vol[20]" << ',';
        DataFile.width(Width); DataFile << "X_vol[21]" << ',';
        DataFile.width(Width); DataFile << "X_vol[22]" << ',';
        DataFile.width(Width); DataFile << "X_vol[23]" << ',';
        DataFile.width(Width); DataFile << "X_vol[24]" << ',';
        DataFile.width(Width); DataFile << "X_vol[25]" << ',';
        DataFile.width(Width); DataFile << "X_vol[26]" << ',';
        DataFile.width(Width); DataFile << "X_vol[27]" << ',';
        DataFile.width(Width); DataFile << "X_vol[28]" << ',';
        DataFile.width(Width); DataFile << "X_vol[29]" << ',';
        DataFile.width(Width); DataFile << "X_vol[30]" << ',';
        DataFile.width(Width); DataFile << "X_vol[31]" << ',';
        DataFile.width(Width); DataFile << "X_vol[32]" << ',';
        DataFile.width(Width); DataFile << "X_vol[33]" << ',';
        DataFile.width(Width); DataFile << "X_vol[34]" << ',';
        DataFile.width(Width); DataFile << "X_vol[35]" << ',';
        DataFile.width(Width); DataFile << "X_vol[36]" << ',';
        DataFile.width(Width); DataFile << "X_vol[37]" << ',';
        DataFile.width(Width); DataFile << "X_vol[38]" << ',';
        DataFile.width(Width); DataFile << "X_vol[39]" << ',';
        DataFile.width(Width); DataFile << "X_vol[40]" << ',';
        DataFile.width(Width); DataFile << "X_vvol" << ',';
        DataFile.width(Width); DataFile << "sPAR_L" << ',';
        DataFile.width(Width); DataFile << "sPAR_RL" << ',';
        DataFile.width(Width); DataFile << "sPAR_TD" << ',';
        DataFile.width(Width); DataFile << "sPAR_cap" << ',';
        DataFile.width(Width); DataFile << "sPAR_ind" << ',';
        DataFile.width(Width); DataFile << "sPAR_l" << ',';
        DataFile.width(Width); DataFile << "sPAR_res" << ',';
        DataFile.width(Width); DataFile << "sPAR_v" << ',';
        DataFile.width(Width); DataFile << "sPAR_w" << ',';
        DataFile.width(Width); DataFile << "sPARi_N" << ',';
        DataFile.width(Width); DataFile << "ACC_Vstep" << ',';
        DataFile.width(Width); DataFile << "ACC_cur[40]" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(IntegerSharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-30);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 42, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 43, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 44, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 45, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 46, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 47, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 48, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 49, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 50, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 51, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 52, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 53, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 54, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 55, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 56, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 57, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 58, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 59, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 60, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 61, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 62, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 63, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 64, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 65, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 66, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 67, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 68, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 69, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 70, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 71, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 72, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 73, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 74, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 75, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 76, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 77, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 78, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 79, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
    Solver.SolverOption(RelativeTolerance, 42, 1e-06);
    Solver.SolverOption(RelativeTolerance, 43, 1e-06);
    Solver.SolverOption(RelativeTolerance, 44, 1e-06);
    Solver.SolverOption(RelativeTolerance, 45, 1e-06);
    Solver.SolverOption(RelativeTolerance, 46, 1e-06);
    Solver.SolverOption(RelativeTolerance, 47, 1e-06);
    Solver.SolverOption(RelativeTolerance, 48, 1e-06);
    Solver.SolverOption(RelativeTolerance, 49, 1e-06);
    Solver.SolverOption(RelativeTolerance, 50, 1e-06);
    Solver.SolverOption(RelativeTolerance, 51, 1e-06);
    Solver.SolverOption(RelativeTolerance, 52, 1e-06);
    Solver.SolverOption(RelativeTolerance, 53, 1e-06);
    Solver.SolverOption(RelativeTolerance, 54, 1e-06);
    Solver.SolverOption(RelativeTolerance, 55, 1e-06);
    Solver.SolverOption(RelativeTolerance, 56, 1e-06);
    Solver.SolverOption(RelativeTolerance, 57, 1e-06);
    Solver.SolverOption(RelativeTolerance, 58, 1e-06);
    Solver.SolverOption(RelativeTolerance, 59, 1e-06);
    Solver.SolverOption(RelativeTolerance, 60, 1e-06);
    Solver.SolverOption(RelativeTolerance, 61, 1e-06);
    Solver.SolverOption(RelativeTolerance, 62, 1e-06);
    Solver.SolverOption(RelativeTolerance, 63, 1e-06);
    Solver.SolverOption(RelativeTolerance, 64, 1e-06);
    Solver.SolverOption(RelativeTolerance, 65, 1e-06);
    Solver.SolverOption(RelativeTolerance, 66, 1e-06);
    Solver.SolverOption(RelativeTolerance, 67, 1e-06);
    Solver.SolverOption(RelativeTolerance, 68, 1e-06);
    Solver.SolverOption(RelativeTolerance, 69, 1e-06);
    Solver.SolverOption(RelativeTolerance, 70, 1e-06);
    Solver.SolverOption(RelativeTolerance, 71, 1e-06);
    Solver.SolverOption(RelativeTolerance, 72, 1e-06);
    Solver.SolverOption(RelativeTolerance, 73, 1e-06);
    Solver.SolverOption(RelativeTolerance, 74, 1e-06);
    Solver.SolverOption(RelativeTolerance, 75, 1e-06);
    Solver.SolverOption(RelativeTolerance, 76, 1e-06);
    Solver.SolverOption(RelativeTolerance, 77, 1e-06);
    Solver.SolverOption(RelativeTolerance, 78, 1e-06);
    Solver.SolverOption(RelativeTolerance, 79, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "TransmissionLineEquations_N_40.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

