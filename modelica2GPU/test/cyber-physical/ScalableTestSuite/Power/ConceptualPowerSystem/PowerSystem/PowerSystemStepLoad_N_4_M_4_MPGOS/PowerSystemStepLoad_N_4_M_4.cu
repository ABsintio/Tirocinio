#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#include "PowerSystemStepLoad_N_4_M_4_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 49;
const int NCP  = 1;
const int NSP  = 83;
const int NISP = 6;
const int NE   = 0;
const int NA   = 122;
const int NIA  = 0;
const int NDO  = 10000000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 200.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_P_sfc" << ',';
        DataFile.width(Width); DataFile << "X_generator[1].T_s[1]" << ',';
        DataFile.width(Width); DataFile << "X_generator[1].T_s[2]" << ',';
        DataFile.width(Width); DataFile << "X_generator[1].T_s[3]" << ',';
        DataFile.width(Width); DataFile << "X_generator[1].T_s[4]" << ',';
        DataFile.width(Width); DataFile << "X_generator[2].T_s[1]" << ',';
        DataFile.width(Width); DataFile << "X_generator[2].T_s[2]" << ',';
        DataFile.width(Width); DataFile << "X_generator[2].T_s[3]" << ',';
        DataFile.width(Width); DataFile << "X_generator[2].T_s[4]" << ',';
        DataFile.width(Width); DataFile << "X_generator[3].T_s[1]" << ',';
        DataFile.width(Width); DataFile << "X_generator[3].T_s[2]" << ',';
        DataFile.width(Width); DataFile << "X_generator[3].T_s[3]" << ',';
        DataFile.width(Width); DataFile << "X_generator[3].T_s[4]" << ',';
        DataFile.width(Width); DataFile << "X_generator[4].T_s[1]" << ',';
        DataFile.width(Width); DataFile << "X_generator[4].T_s[2]" << ',';
        DataFile.width(Width); DataFile << "X_generator[4].T_s[3]" << ',';
        DataFile.width(Width); DataFile << "X_generator[4].T_s[4]" << ',';
        DataFile.width(Width); DataFile << "X_generator[1].err_p_int" << ',';
        DataFile.width(Width); DataFile << "X_generator[2].err_p_int" << ',';
        DataFile.width(Width); DataFile << "X_generator[3].err_p_int" << ',';
        DataFile.width(Width); DataFile << "X_generator[4].err_p_int" << ',';
        DataFile.width(Width); DataFile << "X_generator[1].err_p_t_int" << ',';
        DataFile.width(Width); DataFile << "X_generator[2].err_p_t_int" << ',';
        DataFile.width(Width); DataFile << "X_generator[3].err_p_t_int" << ',';
        DataFile.width(Width); DataFile << "X_generator[4].err_p_t_int" << ',';
        DataFile.width(Width); DataFile << "X_generator[1].omega" << ',';
        DataFile.width(Width); DataFile << "X_generator[2].omega" << ',';
        DataFile.width(Width); DataFile << "X_generator[3].omega" << ',';
        DataFile.width(Width); DataFile << "X_generator[4].omega" << ',';
        DataFile.width(Width); DataFile << "X_generator[1].p" << ',';
        DataFile.width(Width); DataFile << "X_generator[2].p" << ',';
        DataFile.width(Width); DataFile << "X_generator[3].p" << ',';
        DataFile.width(Width); DataFile << "X_generator[4].p" << ',';
        DataFile.width(Width); DataFile << "X_generator[1].p_t_lp" << ',';
        DataFile.width(Width); DataFile << "X_generator[2].p_t_lp" << ',';
        DataFile.width(Width); DataFile << "X_generator[3].p_t_lp" << ',';
        DataFile.width(Width); DataFile << "X_generator[4].p_t_lp" << ',';
        DataFile.width(Width); DataFile << "X_generator[1].q_ev" << ',';
        DataFile.width(Width); DataFile << "X_generator[2].q_ev" << ',';
        DataFile.width(Width); DataFile << "X_generator[3].q_ev" << ',';
        DataFile.width(Width); DataFile << "X_generator[4].q_ev" << ',';
        DataFile.width(Width); DataFile << "X_generator[1].theta" << ',';
        DataFile.width(Width); DataFile << "X_generator[2].theta" << ',';
        DataFile.width(Width); DataFile << "X_generator[3].theta" << ',';
        DataFile.width(Width); DataFile << "X_generator[4].theta" << ',';
        DataFile.width(Width); DataFile << "X_generator[1].y_t" << ',';
        DataFile.width(Width); DataFile << "X_generator[2].y_t" << ',';
        DataFile.width(Width); DataFile << "X_generator[3].y_t" << ',';
        DataFile.width(Width); DataFile << "X_generator[4].y_t" << ',';
        DataFile.width(Width); DataFile << "sPAR_P_d" << ',';
        DataFile.width(Width); DataFile << "sPAR_P_nom" << ',';
        DataFile.width(Width); DataFile << "sPAR_T_sfc" << ',';
        DataFile.width(Width); DataFile << "sPAR_droop" << ',';
        DataFile.width(Width); DataFile << "sPAR_f_ref" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].J" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].J" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].J" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].J" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].Kp_p" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].Kp_p" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].Kp_p" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].Kp_p" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].Kp_t" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].Kp_t" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].Kp_t" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].Kp_t" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].NTU" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].NTU" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].NTU" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].NTU" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].P_nom" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].P_nom" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].P_nom" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].P_nom" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].T_a" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].T_a" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].T_a" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].T_a" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].T_source" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].T_source" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].T_source" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].T_source" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].Ti_p" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].Ti_p" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].Ti_p" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].Ti_p" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].Ti_t" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].Ti_t" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].Ti_t" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].Ti_t" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].alpha" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].alpha" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].alpha" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].alpha" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].droop" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].droop" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].droop" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].droop" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].f_ref" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].f_ref" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].f_ref" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].f_ref" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].omega_ref" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].omega_ref" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].omega_ref" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].omega_ref" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].tau_b" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].tau_b" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].tau_b" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].tau_b" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].tau_q" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].tau_q" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].tau_q" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].tau_q" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].tau_sh" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].tau_sh" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].tau_sh" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].tau_sh" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].tau_t" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].tau_t" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].tau_t" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].tau_t" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].tau_y" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].tau_y" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].tau_y" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].tau_y" << ',';
        DataFile.width(Width); DataFile << "sPAR_omega_ref" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[1].pi" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[2].pi" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[3].pi" << ',';
        DataFile.width(Width); DataFile << "sPAR_generator[4].pi" << ',';
        DataFile.width(Width); DataFile << "sPAR_pi" << ',';
        DataFile.width(Width); DataFile << "sPARi_M" << ',';
        DataFile.width(Width); DataFile << "sPARi_N" << ',';
        DataFile.width(Width); DataFile << "sPARi_generator[1].N" << ',';
        DataFile.width(Width); DataFile << "sPARi_generator[2].N" << ',';
        DataFile.width(Width); DataFile << "sPARi_generator[3].N" << ',';
        DataFile.width(Width); DataFile << "sPARi_generator[4].N" << ',';
        DataFile.width(Width); DataFile << "ACC_P_a[1]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_a[2]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_a[3]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_a[4]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_diss[1,1]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_diss[1,2]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_diss[1,3]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_diss[1,4]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_diss[2,1]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_diss[2,2]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_diss[2,3]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_diss[2,4]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_diss[3,1]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_diss[3,2]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_diss[3,3]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_diss[3,4]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_diss[4,1]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_diss[4,2]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_diss[4,3]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_diss[4,4]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_ex[1,1]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_ex[1,2]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_ex[1,3]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_ex[1,4]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_ex[2,1]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_ex[2,2]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_ex[2,3]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_ex[2,4]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_ex[3,1]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_ex[3,2]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_ex[3,3]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_ex[3,4]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_ex[4,1]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_ex[4,2]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_ex[4,3]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_ex[4,4]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_f" << ',';
        DataFile.width(Width); DataFile << "ACC_P_load[1]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_load[2]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_load[3]" << ',';
        DataFile.width(Width); DataFile << "ACC_P_load[4]" << ',';
        DataFile.width(Width); DataFile << "ACC_f" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].P_a" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].P_a" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].P_a" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].P_a" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].P_sfc" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].P_sfc" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].P_sfc" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].P_sfc" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].P_t" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].P_t" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].P_t" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].P_t" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].P_t_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].P_t_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].P_t_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].P_t_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].T_s_b[1]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].T_s_b[2]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].T_s_b[3]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].T_s_b[4]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].T_s_b[5]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].T_s_b[1]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].T_s_b[2]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].T_s_b[3]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].T_s_b[4]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].T_s_b[5]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].T_s_b[1]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].T_s_b[2]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].T_s_b[3]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].T_s_b[4]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].T_s_b[5]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].T_s_b[1]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].T_s_b[2]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].T_s_b[3]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].T_s_b[4]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].T_s_b[5]" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].delta_f" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].delta_f" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].delta_f" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].delta_f" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].err_p" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].err_p" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].err_p" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].err_p" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].err_p_t" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].err_p_t" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].err_p_t" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].err_p_t" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].f" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].f" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].f" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].f" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].p_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].p_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].p_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].p_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].p_t" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].p_t" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].p_t" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].p_t" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].p_t_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].p_t_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].p_t_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].p_t_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].p_t_0_fc" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].p_t_0_fc" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].p_t_0_fc" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].p_t_0_fc" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].q_ev_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].q_ev_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].q_ev_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].q_ev_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].w_s" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].w_s" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].w_s" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].w_s" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[1].y_t_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[2].y_t_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[3].y_t_0" << ',';
        DataFile.width(Width); DataFile << "ACC_generator[4].y_t_0" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(IntegerSharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(IntegerSharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(IntegerSharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(IntegerSharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(IntegerSharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(IntegerSharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 83) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 84) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 85) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 86) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 87) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 88) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 89) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 90) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 91) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 92) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 93) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 94) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 95) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 96) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 97) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 98) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 99) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 100) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 101) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 102) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 103) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 104) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 105) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 106) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 107) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 108) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 109) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 110) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 111) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 112) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 113) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 114) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 115) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 116) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 117) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 118) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 119) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 120) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 121) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-07);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-07);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-07);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-07);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 42, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 43, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 44, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 45, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 46, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 47, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 48, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
    Solver.SolverOption(RelativeTolerance, 42, 1e-06);
    Solver.SolverOption(RelativeTolerance, 43, 1e-06);
    Solver.SolverOption(RelativeTolerance, 44, 1e-06);
    Solver.SolverOption(RelativeTolerance, 45, 1e-06);
    Solver.SolverOption(RelativeTolerance, 46, 1e-06);
    Solver.SolverOption(RelativeTolerance, 47, 1e-06);
    Solver.SolverOption(RelativeTolerance, 48, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "PowerSystemStepLoad_N_4_M_4.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

