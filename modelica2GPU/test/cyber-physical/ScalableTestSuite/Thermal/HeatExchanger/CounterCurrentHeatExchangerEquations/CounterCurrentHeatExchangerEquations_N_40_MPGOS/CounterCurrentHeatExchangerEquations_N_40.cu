#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#include "CounterCurrentHeatExchangerEquations_N_40_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 117;
const int NCP  = 1;
const int NSP  = 13;
const int NISP = 1;
const int NE   = 0;
const int NA   = 83;
const int NIA  = 0;
const int NDO  = 1000000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 20.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_TA[2]" << ',';
        DataFile.width(Width); DataFile << "X_TA[3]" << ',';
        DataFile.width(Width); DataFile << "X_TA[4]" << ',';
        DataFile.width(Width); DataFile << "X_TA[5]" << ',';
        DataFile.width(Width); DataFile << "X_TA[6]" << ',';
        DataFile.width(Width); DataFile << "X_TA[7]" << ',';
        DataFile.width(Width); DataFile << "X_TA[8]" << ',';
        DataFile.width(Width); DataFile << "X_TA[9]" << ',';
        DataFile.width(Width); DataFile << "X_TA[10]" << ',';
        DataFile.width(Width); DataFile << "X_TA[11]" << ',';
        DataFile.width(Width); DataFile << "X_TA[12]" << ',';
        DataFile.width(Width); DataFile << "X_TA[13]" << ',';
        DataFile.width(Width); DataFile << "X_TA[14]" << ',';
        DataFile.width(Width); DataFile << "X_TA[15]" << ',';
        DataFile.width(Width); DataFile << "X_TA[16]" << ',';
        DataFile.width(Width); DataFile << "X_TA[17]" << ',';
        DataFile.width(Width); DataFile << "X_TA[18]" << ',';
        DataFile.width(Width); DataFile << "X_TA[19]" << ',';
        DataFile.width(Width); DataFile << "X_TA[20]" << ',';
        DataFile.width(Width); DataFile << "X_TA[21]" << ',';
        DataFile.width(Width); DataFile << "X_TA[22]" << ',';
        DataFile.width(Width); DataFile << "X_TA[23]" << ',';
        DataFile.width(Width); DataFile << "X_TA[24]" << ',';
        DataFile.width(Width); DataFile << "X_TA[25]" << ',';
        DataFile.width(Width); DataFile << "X_TA[26]" << ',';
        DataFile.width(Width); DataFile << "X_TA[27]" << ',';
        DataFile.width(Width); DataFile << "X_TA[28]" << ',';
        DataFile.width(Width); DataFile << "X_TA[29]" << ',';
        DataFile.width(Width); DataFile << "X_TA[30]" << ',';
        DataFile.width(Width); DataFile << "X_TA[31]" << ',';
        DataFile.width(Width); DataFile << "X_TA[32]" << ',';
        DataFile.width(Width); DataFile << "X_TA[33]" << ',';
        DataFile.width(Width); DataFile << "X_TA[34]" << ',';
        DataFile.width(Width); DataFile << "X_TA[35]" << ',';
        DataFile.width(Width); DataFile << "X_TA[36]" << ',';
        DataFile.width(Width); DataFile << "X_TA[37]" << ',';
        DataFile.width(Width); DataFile << "X_TA[38]" << ',';
        DataFile.width(Width); DataFile << "X_TA[39]" << ',';
        DataFile.width(Width); DataFile << "X_TA[40]" << ',';
        DataFile.width(Width); DataFile << "X_TB[1]" << ',';
        DataFile.width(Width); DataFile << "X_TB[2]" << ',';
        DataFile.width(Width); DataFile << "X_TB[3]" << ',';
        DataFile.width(Width); DataFile << "X_TB[4]" << ',';
        DataFile.width(Width); DataFile << "X_TB[5]" << ',';
        DataFile.width(Width); DataFile << "X_TB[6]" << ',';
        DataFile.width(Width); DataFile << "X_TB[7]" << ',';
        DataFile.width(Width); DataFile << "X_TB[8]" << ',';
        DataFile.width(Width); DataFile << "X_TB[9]" << ',';
        DataFile.width(Width); DataFile << "X_TB[10]" << ',';
        DataFile.width(Width); DataFile << "X_TB[11]" << ',';
        DataFile.width(Width); DataFile << "X_TB[12]" << ',';
        DataFile.width(Width); DataFile << "X_TB[13]" << ',';
        DataFile.width(Width); DataFile << "X_TB[14]" << ',';
        DataFile.width(Width); DataFile << "X_TB[15]" << ',';
        DataFile.width(Width); DataFile << "X_TB[16]" << ',';
        DataFile.width(Width); DataFile << "X_TB[17]" << ',';
        DataFile.width(Width); DataFile << "X_TB[18]" << ',';
        DataFile.width(Width); DataFile << "X_TB[19]" << ',';
        DataFile.width(Width); DataFile << "X_TB[20]" << ',';
        DataFile.width(Width); DataFile << "X_TB[21]" << ',';
        DataFile.width(Width); DataFile << "X_TB[22]" << ',';
        DataFile.width(Width); DataFile << "X_TB[23]" << ',';
        DataFile.width(Width); DataFile << "X_TB[24]" << ',';
        DataFile.width(Width); DataFile << "X_TB[25]" << ',';
        DataFile.width(Width); DataFile << "X_TB[26]" << ',';
        DataFile.width(Width); DataFile << "X_TB[27]" << ',';
        DataFile.width(Width); DataFile << "X_TB[28]" << ',';
        DataFile.width(Width); DataFile << "X_TB[29]" << ',';
        DataFile.width(Width); DataFile << "X_TB[30]" << ',';
        DataFile.width(Width); DataFile << "X_TB[31]" << ',';
        DataFile.width(Width); DataFile << "X_TB[32]" << ',';
        DataFile.width(Width); DataFile << "X_TB[33]" << ',';
        DataFile.width(Width); DataFile << "X_TB[34]" << ',';
        DataFile.width(Width); DataFile << "X_TB[35]" << ',';
        DataFile.width(Width); DataFile << "X_TB[36]" << ',';
        DataFile.width(Width); DataFile << "X_TB[37]" << ',';
        DataFile.width(Width); DataFile << "X_TB[38]" << ',';
        DataFile.width(Width); DataFile << "X_TB[39]" << ',';
        DataFile.width(Width); DataFile << "X_TW[1]" << ',';
        DataFile.width(Width); DataFile << "X_TW[2]" << ',';
        DataFile.width(Width); DataFile << "X_TW[3]" << ',';
        DataFile.width(Width); DataFile << "X_TW[4]" << ',';
        DataFile.width(Width); DataFile << "X_TW[5]" << ',';
        DataFile.width(Width); DataFile << "X_TW[6]" << ',';
        DataFile.width(Width); DataFile << "X_TW[7]" << ',';
        DataFile.width(Width); DataFile << "X_TW[8]" << ',';
        DataFile.width(Width); DataFile << "X_TW[9]" << ',';
        DataFile.width(Width); DataFile << "X_TW[10]" << ',';
        DataFile.width(Width); DataFile << "X_TW[11]" << ',';
        DataFile.width(Width); DataFile << "X_TW[12]" << ',';
        DataFile.width(Width); DataFile << "X_TW[13]" << ',';
        DataFile.width(Width); DataFile << "X_TW[14]" << ',';
        DataFile.width(Width); DataFile << "X_TW[15]" << ',';
        DataFile.width(Width); DataFile << "X_TW[16]" << ',';
        DataFile.width(Width); DataFile << "X_TW[17]" << ',';
        DataFile.width(Width); DataFile << "X_TW[18]" << ',';
        DataFile.width(Width); DataFile << "X_TW[19]" << ',';
        DataFile.width(Width); DataFile << "X_TW[20]" << ',';
        DataFile.width(Width); DataFile << "X_TW[21]" << ',';
        DataFile.width(Width); DataFile << "X_TW[22]" << ',';
        DataFile.width(Width); DataFile << "X_TW[23]" << ',';
        DataFile.width(Width); DataFile << "X_TW[24]" << ',';
        DataFile.width(Width); DataFile << "X_TW[25]" << ',';
        DataFile.width(Width); DataFile << "X_TW[26]" << ',';
        DataFile.width(Width); DataFile << "X_TW[27]" << ',';
        DataFile.width(Width); DataFile << "X_TW[28]" << ',';
        DataFile.width(Width); DataFile << "X_TW[29]" << ',';
        DataFile.width(Width); DataFile << "X_TW[30]" << ',';
        DataFile.width(Width); DataFile << "X_TW[31]" << ',';
        DataFile.width(Width); DataFile << "X_TW[32]" << ',';
        DataFile.width(Width); DataFile << "X_TW[33]" << ',';
        DataFile.width(Width); DataFile << "X_TW[34]" << ',';
        DataFile.width(Width); DataFile << "X_TW[35]" << ',';
        DataFile.width(Width); DataFile << "X_TW[36]" << ',';
        DataFile.width(Width); DataFile << "X_TW[37]" << ',';
        DataFile.width(Width); DataFile << "X_TW[38]" << ',';
        DataFile.width(Width); DataFile << "X_TW[39]" << ',';
        DataFile.width(Width); DataFile << "sPAR_L" << ',';
        DataFile.width(Width); DataFile << "sPAR_areaA" << ',';
        DataFile.width(Width); DataFile << "sPAR_areaB" << ',';
        DataFile.width(Width); DataFile << "sPAR_cpA" << ',';
        DataFile.width(Width); DataFile << "sPAR_cpB" << ',';
        DataFile.width(Width); DataFile << "sPAR_cpW" << ',';
        DataFile.width(Width); DataFile << "sPAR_gammaA" << ',';
        DataFile.width(Width); DataFile << "sPAR_gammaB" << ',';
        DataFile.width(Width); DataFile << "sPAR_l" << ',';
        DataFile.width(Width); DataFile << "sPAR_omega" << ',';
        DataFile.width(Width); DataFile << "sPAR_rhoA" << ',';
        DataFile.width(Width); DataFile << "sPAR_rhoB" << ',';
        DataFile.width(Width); DataFile << "sPAR_wB" << ',';
        DataFile.width(Width); DataFile << "sPARi_N" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[1]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[2]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[3]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[4]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[5]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[6]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[7]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[8]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[9]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[10]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[11]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[12]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[13]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[14]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[15]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[16]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[17]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[18]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[19]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[20]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[21]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[22]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[23]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[24]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[25]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[26]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[27]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[28]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[29]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[30]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[31]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[32]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[33]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[34]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[35]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[36]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[37]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[38]" << ',';
        DataFile.width(Width); DataFile << "ACC_QA[39]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[1]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[2]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[3]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[4]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[5]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[6]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[7]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[8]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[9]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[10]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[11]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[12]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[13]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[14]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[15]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[16]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[17]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[18]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[19]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[20]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[21]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[22]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[23]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[24]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[25]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[26]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[27]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[28]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[29]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[30]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[31]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[32]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[33]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[34]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[35]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[36]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[37]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[38]" << ',';
        DataFile.width(Width); DataFile << "ACC_QB[39]" << ',';
        DataFile.width(Width); DataFile << "ACC_QtotA" << ',';
        DataFile.width(Width); DataFile << "ACC_QtotB" << ',';
        DataFile.width(Width); DataFile << "ACC_TA[1]" << ',';
        DataFile.width(Width); DataFile << "ACC_TB[40]" << ',';
        DataFile.width(Width); DataFile << "ACC_wA" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 83) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 84) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 85) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 86) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 87) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 88) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 89) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 90) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 91) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 92) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 93) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 94) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 95) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 96) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 97) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 98) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 99) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 100) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 101) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 102) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 103) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 104) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 105) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 106) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 107) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 108) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 109) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 110) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 111) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 112) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 113) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 114) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 115) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 116) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(IntegerSharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 82) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 42, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 43, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 44, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 45, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 46, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 47, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 48, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 49, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 50, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 51, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 52, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 53, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 54, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 55, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 56, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 57, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 58, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 59, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 60, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 61, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 62, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 63, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 64, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 65, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 66, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 67, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 68, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 69, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 70, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 71, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 72, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 73, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 74, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 75, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 76, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 77, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 78, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 79, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 80, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 81, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 82, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 83, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 84, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 85, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 86, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 87, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 88, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 89, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 90, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 91, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 92, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 93, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 94, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 95, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 96, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 97, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 98, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 99, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 100, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 101, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 102, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 103, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 104, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 105, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 106, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 107, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 108, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 109, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 110, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 111, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 112, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 113, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 114, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 115, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 116, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
    Solver.SolverOption(RelativeTolerance, 42, 1e-06);
    Solver.SolverOption(RelativeTolerance, 43, 1e-06);
    Solver.SolverOption(RelativeTolerance, 44, 1e-06);
    Solver.SolverOption(RelativeTolerance, 45, 1e-06);
    Solver.SolverOption(RelativeTolerance, 46, 1e-06);
    Solver.SolverOption(RelativeTolerance, 47, 1e-06);
    Solver.SolverOption(RelativeTolerance, 48, 1e-06);
    Solver.SolverOption(RelativeTolerance, 49, 1e-06);
    Solver.SolverOption(RelativeTolerance, 50, 1e-06);
    Solver.SolverOption(RelativeTolerance, 51, 1e-06);
    Solver.SolverOption(RelativeTolerance, 52, 1e-06);
    Solver.SolverOption(RelativeTolerance, 53, 1e-06);
    Solver.SolverOption(RelativeTolerance, 54, 1e-06);
    Solver.SolverOption(RelativeTolerance, 55, 1e-06);
    Solver.SolverOption(RelativeTolerance, 56, 1e-06);
    Solver.SolverOption(RelativeTolerance, 57, 1e-06);
    Solver.SolverOption(RelativeTolerance, 58, 1e-06);
    Solver.SolverOption(RelativeTolerance, 59, 1e-06);
    Solver.SolverOption(RelativeTolerance, 60, 1e-06);
    Solver.SolverOption(RelativeTolerance, 61, 1e-06);
    Solver.SolverOption(RelativeTolerance, 62, 1e-06);
    Solver.SolverOption(RelativeTolerance, 63, 1e-06);
    Solver.SolverOption(RelativeTolerance, 64, 1e-06);
    Solver.SolverOption(RelativeTolerance, 65, 1e-06);
    Solver.SolverOption(RelativeTolerance, 66, 1e-06);
    Solver.SolverOption(RelativeTolerance, 67, 1e-06);
    Solver.SolverOption(RelativeTolerance, 68, 1e-06);
    Solver.SolverOption(RelativeTolerance, 69, 1e-06);
    Solver.SolverOption(RelativeTolerance, 70, 1e-06);
    Solver.SolverOption(RelativeTolerance, 71, 1e-06);
    Solver.SolverOption(RelativeTolerance, 72, 1e-06);
    Solver.SolverOption(RelativeTolerance, 73, 1e-06);
    Solver.SolverOption(RelativeTolerance, 74, 1e-06);
    Solver.SolverOption(RelativeTolerance, 75, 1e-06);
    Solver.SolverOption(RelativeTolerance, 76, 1e-06);
    Solver.SolverOption(RelativeTolerance, 77, 1e-06);
    Solver.SolverOption(RelativeTolerance, 78, 1e-06);
    Solver.SolverOption(RelativeTolerance, 79, 1e-06);
    Solver.SolverOption(RelativeTolerance, 80, 1e-06);
    Solver.SolverOption(RelativeTolerance, 81, 1e-06);
    Solver.SolverOption(RelativeTolerance, 82, 1e-06);
    Solver.SolverOption(RelativeTolerance, 83, 1e-06);
    Solver.SolverOption(RelativeTolerance, 84, 1e-06);
    Solver.SolverOption(RelativeTolerance, 85, 1e-06);
    Solver.SolverOption(RelativeTolerance, 86, 1e-06);
    Solver.SolverOption(RelativeTolerance, 87, 1e-06);
    Solver.SolverOption(RelativeTolerance, 88, 1e-06);
    Solver.SolverOption(RelativeTolerance, 89, 1e-06);
    Solver.SolverOption(RelativeTolerance, 90, 1e-06);
    Solver.SolverOption(RelativeTolerance, 91, 1e-06);
    Solver.SolverOption(RelativeTolerance, 92, 1e-06);
    Solver.SolverOption(RelativeTolerance, 93, 1e-06);
    Solver.SolverOption(RelativeTolerance, 94, 1e-06);
    Solver.SolverOption(RelativeTolerance, 95, 1e-06);
    Solver.SolverOption(RelativeTolerance, 96, 1e-06);
    Solver.SolverOption(RelativeTolerance, 97, 1e-06);
    Solver.SolverOption(RelativeTolerance, 98, 1e-06);
    Solver.SolverOption(RelativeTolerance, 99, 1e-06);
    Solver.SolverOption(RelativeTolerance, 100, 1e-06);
    Solver.SolverOption(RelativeTolerance, 101, 1e-06);
    Solver.SolverOption(RelativeTolerance, 102, 1e-06);
    Solver.SolverOption(RelativeTolerance, 103, 1e-06);
    Solver.SolverOption(RelativeTolerance, 104, 1e-06);
    Solver.SolverOption(RelativeTolerance, 105, 1e-06);
    Solver.SolverOption(RelativeTolerance, 106, 1e-06);
    Solver.SolverOption(RelativeTolerance, 107, 1e-06);
    Solver.SolverOption(RelativeTolerance, 108, 1e-06);
    Solver.SolverOption(RelativeTolerance, 109, 1e-06);
    Solver.SolverOption(RelativeTolerance, 110, 1e-06);
    Solver.SolverOption(RelativeTolerance, 111, 1e-06);
    Solver.SolverOption(RelativeTolerance, 112, 1e-06);
    Solver.SolverOption(RelativeTolerance, 113, 1e-06);
    Solver.SolverOption(RelativeTolerance, 114, 1e-06);
    Solver.SolverOption(RelativeTolerance, 115, 1e-06);
    Solver.SolverOption(RelativeTolerance, 116, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "CounterCurrentHeatExchangerEquations_N_40.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

