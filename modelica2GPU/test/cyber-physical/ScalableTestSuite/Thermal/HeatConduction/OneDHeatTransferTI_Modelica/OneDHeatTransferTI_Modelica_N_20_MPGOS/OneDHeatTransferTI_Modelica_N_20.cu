#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#include "OneDHeatTransferTI_Modelica_N_20_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 19;
const int NCP  = 1;
const int NSP  = 48;
const int NISP = 1;
const int NE   = 0;
const int NA   = 173;
const int NIA  = 0;
const int NDO  = 15000000.0;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 1500.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_heatcapacitor[1].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[2].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[3].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[4].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[5].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[6].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[7].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[8].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[9].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[10].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[11].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[12].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[13].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[14].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[15].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[16].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[17].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[18].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[19].T" << ',';
        DataFile.width(Width); DataFile << "sPAR_A" << ',';
        DataFile.width(Width); DataFile << "sPAR_L" << ',';
        DataFile.width(Width); DataFile << "sPAR_T0" << ',';
        DataFile.width(Width); DataFile << "sPAR_TN" << ',';
        DataFile.width(Width); DataFile << "sPAR_cp" << ',';
        DataFile.width(Width); DataFile << "sPAR_fixedtemperature.T" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[1].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[2].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[3].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[4].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[5].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[6].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[7].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[8].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[9].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[10].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[11].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[12].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[13].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[14].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[15].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[16].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[17].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[18].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[19].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_lambda" << ',';
        DataFile.width(Width); DataFile << "sPAR_m" << ',';
        DataFile.width(Width); DataFile << "sPAR_rho" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[1].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[2].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[3].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[4].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[5].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[6].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[7].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[8].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[9].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[10].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[11].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[12].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[13].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[14].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[15].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[16].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[17].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[18].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[19].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_v" << ',';
        DataFile.width(Width); DataFile << "sPARi_N" << ',';
        DataFile.width(Width); DataFile << "ACC_fixedtemperature.port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_fixedtemperature.port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[1].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[2].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[3].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[4].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[5].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[6].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[7].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[8].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[9].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[10].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[11].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[12].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[13].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[14].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[15].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[16].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[17].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[18].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[19].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[1].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[2].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[3].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[4].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[5].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[6].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[7].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[8].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[9].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[10].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[11].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[12].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[13].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[14].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[15].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[16].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[17].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[18].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[19].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[1].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[2].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[3].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[4].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[5].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[6].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[7].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[8].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[9].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[10].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[11].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[12].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[13].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[14].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[15].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[16].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[17].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[18].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[19].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[1].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[2].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[3].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[4].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[5].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[6].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[7].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[8].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[9].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[10].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[11].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[12].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[13].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[14].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[15].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[16].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[17].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[18].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[19].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[1].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[2].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[3].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[4].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[5].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[6].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[7].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[8].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[9].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[10].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[11].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[12].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[13].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[14].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[15].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[16].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[17].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[18].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[19].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[1].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[2].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[3].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[4].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[5].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[6].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[7].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[8].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[9].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[10].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[11].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[12].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[13].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[14].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[15].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[16].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[17].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[18].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[19].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[1].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[2].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[3].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[4].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[5].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[6].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[7].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[8].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[9].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[10].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[11].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[12].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[13].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[14].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[15].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[16].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[17].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[18].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[19].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[1].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[2].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[3].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[4].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[5].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[6].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[7].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[8].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[9].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[10].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[11].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[12].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[13].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[14].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[15].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[16].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[17].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[18].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[19].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[1].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[2].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[3].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[4].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[5].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[6].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[7].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[8].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[9].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[10].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[11].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[12].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[13].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[14].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[15].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[16].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[17].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[18].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[19].port_b.T" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(IntegerSharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 83) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 84) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 85) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 86) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 87) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 88) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 89) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 90) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 91) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 92) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 93) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 94) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 95) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 96) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 97) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 98) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 99) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 100) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 101) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 102) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 103) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 104) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 105) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 106) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 107) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 108) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 109) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 110) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 111) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 112) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 113) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 114) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 115) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 116) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 117) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 118) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 119) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 120) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 121) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 122) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 123) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 124) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 125) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 126) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 127) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 128) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 129) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 130) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 131) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 132) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 133) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 134) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 135) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 136) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 137) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 138) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 139) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 140) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 141) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 142) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 143) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 144) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 145) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 146) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 147) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 148) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 149) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 150) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 151) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 152) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 153) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 154) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 155) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 156) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 157) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 158) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 159) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 160) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 161) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 162) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 163) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 164) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 165) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 166) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 167) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 168) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 169) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 170) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 171) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 172) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "OneDHeatTransferTI_Modelica_N_20.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

