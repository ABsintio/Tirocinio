#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#include "OneDHeatTransferTI_Modelica_N_10_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 9;
const int NCP  = 1;
const int NSP  = 28;
const int NISP = 1;
const int NE   = 0;
const int NA   = 83;
const int NIA  = 0;
const int NDO  = 15000000.0;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 1500.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_heatcapacitor[1].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[2].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[3].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[4].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[5].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[6].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[7].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[8].T" << ',';
        DataFile.width(Width); DataFile << "X_heatcapacitor[9].T" << ',';
        DataFile.width(Width); DataFile << "sPAR_A" << ',';
        DataFile.width(Width); DataFile << "sPAR_L" << ',';
        DataFile.width(Width); DataFile << "sPAR_T0" << ',';
        DataFile.width(Width); DataFile << "sPAR_TN" << ',';
        DataFile.width(Width); DataFile << "sPAR_cp" << ',';
        DataFile.width(Width); DataFile << "sPAR_fixedtemperature.T" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[1].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[2].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[3].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[4].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[5].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[6].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[7].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[8].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_heatcapacitor[9].C" << ',';
        DataFile.width(Width); DataFile << "sPAR_lambda" << ',';
        DataFile.width(Width); DataFile << "sPAR_m" << ',';
        DataFile.width(Width); DataFile << "sPAR_rho" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[1].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[2].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[3].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[4].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[5].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[6].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[7].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[8].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_thermalconductor[9].G" << ',';
        DataFile.width(Width); DataFile << "sPAR_v" << ',';
        DataFile.width(Width); DataFile << "sPARi_N" << ',';
        DataFile.width(Width); DataFile << "ACC_fixedtemperature.port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_fixedtemperature.port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[1].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[2].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[3].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[4].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[5].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[6].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[7].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[8].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[9].der_T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[1].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[2].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[3].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[4].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[5].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[6].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[7].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[8].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[9].port.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[1].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[2].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[3].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[4].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[5].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[6].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[7].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[8].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_heatcapacitor[9].port.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[1].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[2].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[3].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[4].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[5].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[6].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[7].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[8].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[9].Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[1].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[2].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[3].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[4].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[5].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[6].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[7].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[8].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[9].dT" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[1].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[2].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[3].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[4].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[5].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[6].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[7].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[8].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[9].port_a.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[1].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[2].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[3].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[4].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[5].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[6].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[7].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[8].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[9].port_a.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[1].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[2].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[3].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[4].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[5].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[6].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[7].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[8].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[9].port_b.Q_flow" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[1].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[2].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[3].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[4].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[5].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[6].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[7].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[8].port_b.T" << ',';
        DataFile.width(Width); DataFile << "ACC_thermalconductor[9].port_b.T" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(IntegerSharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 82) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 512; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "OneDHeatTransferTI_Modelica_N_10.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

