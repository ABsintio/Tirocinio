#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "TysonModel_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45
#define PRECISION double
const int NT   = 1;
const int SD   = 6;     // [C2], [CP], [M], [pM], [Y], [YP]
const int NCP  = 1;
const int NSP  = 10;
const int NISP = 0;
const int NE   = 2;
const int NA   = 4;
const int NIA  = 0;
const int NDO  = 100;

void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>&, 
    vector<PRECISION>&, vector<PRECISION>&, int, int
);

void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
	int Width = 18;
	DataFile.precision(10);
	DataFile.flags(ios::scientific);
	
	for (int tid=0; tid<NumberOfThreads; tid++)
	{
        // GetHost -> valore del parametro.
        DataFile.width(Width); DataFile << "[C2]" << ',';
        DataFile.width(Width); DataFile << "[CP]" << ',';
        DataFile.width(Width); DataFile << "[M]" << ',';
        DataFile.width(Width); DataFile << "[pM]" << ',';
        DataFile.width(Width); DataFile << "[Y]" << ',';
        DataFile.width(Width); DataFile << "[YP]" << ',';
		DataFile.width(Width); DataFile << "[CT]" << ',';
		DataFile.width(Width); DataFile << "[YT]" << ',';
        DataFile.width(Width); DataFile << "[M]/[CT]" << ',';
        DataFile.width(Width); DataFile << "[YT]/[CT]" << endl;
		DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
		DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
		DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
		DataFile << '\n';
	}
}

int main() {

    int NumberOfProblems = 1;
    int BlockSize        = 128;

    ListCUDADevices();

    int MajorVersion = 7;
    int MinorVersion = 5;

    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);
    
    PRECISION InitialCondition_C2 = 0;
    PRECISION InitialCondition_CP = 0.75;
    PRECISION InitialCondition_M  = 0;
    PRECISION InitialCondition_pM = 0.25;
    PRECISION InitialCondition_Y  = 0;
    PRECISION InitialCondition_YP = 0;
    vector<PRECISION> Parameter_X = {
        InitialCondition_C2, InitialCondition_CP, InitialCondition_M,
        InitialCondition_pM, InitialCondition_Y, InitialCondition_YP
    };

    PRECISION Parameter_k1aaCT    = 0.015;
    PRECISION Parameter_k2        = 0;
    PRECISION Parameter_k3CT      = 200;
    PRECISION Parameter_k4        = 180;
    PRECISION Parameter_k4prime   = 0.018;
    PRECISION Parameter_k5tildeP  = 0;
    PRECISION Parameter_k6        = 1;
    PRECISION Parameter_K7        = 0.6;
    PRECISION Parameter_k8tildeP  = 1e+06;
    PRECISION Parameter_k9        = 1000;
    vector<PRECISION> Parameter_shared = {
        Parameter_k1aaCT, Parameter_k2, Parameter_k3CT, Parameter_k4, Parameter_k4prime,
        Parameter_k5tildeP, Parameter_k6, Parameter_K7, Parameter_k8tildeP, Parameter_k9
    };

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, BlockSize);
    Solver.SolverOption(PreferSharedMemory, 0);
    Solver.SolverOption(InitialTimeStep, 1e-2);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1e+6);
    Solver.SolverOption(MinimumTimeStep, 1e-12);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);
    Solver.SolverOption(TimeStepShrinkLimit, 0.1);
    Solver.SolverOption(EventDirection, 0, 0);
    Solver.SolverOption(EventDirection, 1, 0);
    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);

    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "TysonModel.csv" );
    clock_t SimulationStart = clock();
	clock_t TransientStart;
    clock_t TransientEnd;

    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, Parameter_shared, Parameter_X, i * NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            cout << "Ciao" << endl;
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
	cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
	cout << "Test finished!" << endl;

    return 0;
}

void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    vector<PRECISION>& Parameters_Shared, vector<PRECISION>& Variable_X, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
	int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 6);

        int i = 0;
        for (PRECISION x : Variable_X) {
            Solver.SetHost(ProblemNumber, ActualState, i++, x);
        }

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);

        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);

        Solver.SetHost(ProblemNumber, Accessories, 0, 1.0);
        Solver.SetHost(ProblemNumber, Accessories, 1, 0.25);
        Solver.SetHost(ProblemNumber, Accessories, 2, 0.0);
        Solver.SetHost(ProblemNumber, Accessories, 3, 0.25);

        Solver.SetHost(ProblemNumber, DenseIndex, 0 );
		
		ProblemNumber++;
        k_begin++;
    }

    int s{0};
    for (PRECISION shared : Parameters_Shared) {
        Solver.SetHost(SharedParameters, s++, shared);
    }
}