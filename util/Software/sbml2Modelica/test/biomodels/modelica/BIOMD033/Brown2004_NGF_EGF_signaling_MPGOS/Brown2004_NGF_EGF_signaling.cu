#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "Brown2004_NGF_EGF_signaling_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45 // Runge-Kutta Order 4th
#define PRECISION double
const int NT   = 1;
const int SD   = 137;
const int NCP  = 1;
const int NSP  = 0;
const int NISP = 0;
const int NE   = 0;
const int NA   = 58;
const int NIA  = 0;
const int NDO  = 10000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 120.0);  

        // Settaggio dei valori iniziali degli ActualState
 

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        // Settaggio dei valori iniziali per ACC (se presenti)


        // Settaggio dei valori iniziali per ACCi (se presenti)

		
        ProblemNumber++;
        k_begin++;
    }

    // Settaggio dei valori iniziali per sPAR (se presenti)


    // Settaggio dei valori iniziali per sPARi (se presenti)

}


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_Params.elmt_KmAkt" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmC3G" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmC3GNGF" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmEGF" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmNGF" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmPI3K" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmPI3KRas" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmRaf1ByAkt" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmRap1ToBRaf" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmRapGap" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmRasGap" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmRasToRaf1" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmSos" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmdBRaf" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmdErk" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmdMek" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmdRaf1" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmdSos" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmpBRaf" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmpMekCytoplasmic" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmpP90Rsk" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_KmpRaf1" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kAkt" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kC3G" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kC3GNGF" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kEGF" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kNGF" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kPI3K" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kPI3KRas" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kRap1ToBRaf" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kRapGap" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kRasGap" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kRasToRaf1" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kSos" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kdBRaf" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kdErk" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kdMek" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kdRaf1" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kdRaf1ByAkt" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kdSos" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kpBRaf" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kpMekCytoplasmic" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kpP90Rsk" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kpRaf1" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_krbEGF" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_krbNGF" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kruEGF" << ',';
        DataFile.width(Width); DataFile << "X_Params.elmt_kruNGF" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product10" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product11" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product13" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product15" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product17" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product19" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product2" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product21" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product23" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product25" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product27" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product29" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product31" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product33" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product35" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product37" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product39" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product4" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product41" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product43" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product45" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product47" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product49" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product5" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product51" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product53" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product55" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product8" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant0" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant1" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant12" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant14" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant16" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant18" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant20" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant22" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant24" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant26" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant28" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant3" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant30" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant32" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant34" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant36" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant38" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant40" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant42" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant44" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant46" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant48" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant50" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant52" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant54" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant6" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant7" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant9" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_AktActive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_AktInactive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_BRafActive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_BRafInactive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_C3GActive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_C3GInactive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_EGF_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_ErkActive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_ErkInactive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_MekActive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_MekInactive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_NGF_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_P90RskActive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_P90RskInactive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_PI3KActive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_PI3KInactive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_PP2AActive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_Raf1Active_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_Raf1Inactive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_Raf1PPtase_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_Rap1Active_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_Rap1Inactive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_RapGapActive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_RasActive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_RasGapActive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_RasInactive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_SosActive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_SosInactive_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_boundEGFReceptor_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_boundNGFReceptor_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_cell" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_freeEGFReceptor_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_freeNGFReceptor_amount" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_AktActivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_AktActive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_AktInactive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_BRafActive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_BRafByRap1ActivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_BRafInactive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_BRafbyPPtaseDeactivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_C3GActivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_C3GActive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_C3GInactive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_EGFBindingReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_EGFUnbindingReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_EGF_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_ErkActivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_ErkActive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_ErkDeactivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_ErkInactive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_MekActive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_MekDeactivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_MekInactive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_MekbyBRafActivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_MekbyRaf1ActivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_NGFBindingReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_NGFUnbindingReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_NGF_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_P90RskActivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_P90RskActive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_P90RskInactive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_PI3KActive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_PI3KInactive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_PI3KbyEGFRActivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_PI3KbyRasActivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_PP2AActive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Raf1Active_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Raf1ByAktDeactivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Raf1ByRasActivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Raf1Inactive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Raf1PPtase_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Raf1byPPtaseDeactivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Rap1ActivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Rap1Active_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Rap1DeactivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Rap1Inactive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_RapGapActive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_RasActivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_RasActive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_RasDeactivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_RasGapActive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_RasInactive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_SosActivationByEGFReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_SosActivationByNGFReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_SosActive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_SosDeactivationReaction" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_SosInactive_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_boundEGFReceptor_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_boundNGFReceptor_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_freeEGFReceptor_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_freeNGFReceptor_conc" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 57) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 58) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 59) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 60) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 61) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 62) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 63) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 64) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 65) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 66) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 67) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 68) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 69) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 70) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 71) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 72) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 73) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 74) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 75) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 76) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 77) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 78) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 79) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 80) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 81) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 82) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 83) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 84) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 85) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 86) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 87) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 88) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 89) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 90) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 91) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 92) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 93) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 94) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 95) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 96) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 97) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 98) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 99) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 100) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 101) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 102) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 103) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 104) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 105) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 106) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 107) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 108) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 109) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 110) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 111) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 112) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 113) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 114) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 115) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 116) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 117) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 118) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 119) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 120) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 121) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 122) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 123) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 124) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 125) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 126) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 127) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 128) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 129) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 130) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 131) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 132) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 133) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 134) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 135) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 136) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 29) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 30) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 31) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 32) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 33) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 34) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 35) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 36) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 37) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 38) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 39) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 40) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 41) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 42) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 43) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 44) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 45) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 46) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 47) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 48) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 49) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 50) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 51) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 52) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 53) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 54) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 55) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 56) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 57) << ',';

        DataFile << '\n';
    }
}


int main() {
    int NumberOfProblems = NT; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 64; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 1.0);

    Solver.SolverOption(TimeStepShrinkLimit, 0.2);



    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 29, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 30, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 31, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 32, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 33, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 34, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 35, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 36, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 37, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 38, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 39, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 40, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 41, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 42, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 43, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 44, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 45, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 46, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 47, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 48, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 49, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 50, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 51, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 52, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 53, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 54, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 55, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 56, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 57, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 58, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 59, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 60, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 61, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 62, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 63, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 64, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 65, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 66, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 67, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 68, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 69, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 70, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 71, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 72, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 73, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 74, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 75, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 76, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 77, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 78, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 79, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 80, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 81, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 82, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 83, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 84, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 85, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 86, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 87, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 88, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 89, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 90, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 91, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 92, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 93, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 94, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 95, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 96, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 97, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 98, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 99, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 100, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 101, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 102, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 103, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 104, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 105, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 106, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 107, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 108, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 109, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 110, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 111, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 112, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 113, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 114, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 115, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 116, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 117, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 118, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 119, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 120, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 121, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 122, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 123, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 124, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 125, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 126, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 127, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 128, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 129, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 130, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 131, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 132, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 133, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 134, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 135, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 136, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
    Solver.SolverOption(RelativeTolerance, 29, 1e-06);
    Solver.SolverOption(RelativeTolerance, 30, 1e-06);
    Solver.SolverOption(RelativeTolerance, 31, 1e-06);
    Solver.SolverOption(RelativeTolerance, 32, 1e-06);
    Solver.SolverOption(RelativeTolerance, 33, 1e-06);
    Solver.SolverOption(RelativeTolerance, 34, 1e-06);
    Solver.SolverOption(RelativeTolerance, 35, 1e-06);
    Solver.SolverOption(RelativeTolerance, 36, 1e-06);
    Solver.SolverOption(RelativeTolerance, 37, 1e-06);
    Solver.SolverOption(RelativeTolerance, 38, 1e-06);
    Solver.SolverOption(RelativeTolerance, 39, 1e-06);
    Solver.SolverOption(RelativeTolerance, 40, 1e-06);
    Solver.SolverOption(RelativeTolerance, 41, 1e-06);
    Solver.SolverOption(RelativeTolerance, 42, 1e-06);
    Solver.SolverOption(RelativeTolerance, 43, 1e-06);
    Solver.SolverOption(RelativeTolerance, 44, 1e-06);
    Solver.SolverOption(RelativeTolerance, 45, 1e-06);
    Solver.SolverOption(RelativeTolerance, 46, 1e-06);
    Solver.SolverOption(RelativeTolerance, 47, 1e-06);
    Solver.SolverOption(RelativeTolerance, 48, 1e-06);
    Solver.SolverOption(RelativeTolerance, 49, 1e-06);
    Solver.SolverOption(RelativeTolerance, 50, 1e-06);
    Solver.SolverOption(RelativeTolerance, 51, 1e-06);
    Solver.SolverOption(RelativeTolerance, 52, 1e-06);
    Solver.SolverOption(RelativeTolerance, 53, 1e-06);
    Solver.SolverOption(RelativeTolerance, 54, 1e-06);
    Solver.SolverOption(RelativeTolerance, 55, 1e-06);
    Solver.SolverOption(RelativeTolerance, 56, 1e-06);
    Solver.SolverOption(RelativeTolerance, 57, 1e-06);
    Solver.SolverOption(RelativeTolerance, 58, 1e-06);
    Solver.SolverOption(RelativeTolerance, 59, 1e-06);
    Solver.SolverOption(RelativeTolerance, 60, 1e-06);
    Solver.SolverOption(RelativeTolerance, 61, 1e-06);
    Solver.SolverOption(RelativeTolerance, 62, 1e-06);
    Solver.SolverOption(RelativeTolerance, 63, 1e-06);
    Solver.SolverOption(RelativeTolerance, 64, 1e-06);
    Solver.SolverOption(RelativeTolerance, 65, 1e-06);
    Solver.SolverOption(RelativeTolerance, 66, 1e-06);
    Solver.SolverOption(RelativeTolerance, 67, 1e-06);
    Solver.SolverOption(RelativeTolerance, 68, 1e-06);
    Solver.SolverOption(RelativeTolerance, 69, 1e-06);
    Solver.SolverOption(RelativeTolerance, 70, 1e-06);
    Solver.SolverOption(RelativeTolerance, 71, 1e-06);
    Solver.SolverOption(RelativeTolerance, 72, 1e-06);
    Solver.SolverOption(RelativeTolerance, 73, 1e-06);
    Solver.SolverOption(RelativeTolerance, 74, 1e-06);
    Solver.SolverOption(RelativeTolerance, 75, 1e-06);
    Solver.SolverOption(RelativeTolerance, 76, 1e-06);
    Solver.SolverOption(RelativeTolerance, 77, 1e-06);
    Solver.SolverOption(RelativeTolerance, 78, 1e-06);
    Solver.SolverOption(RelativeTolerance, 79, 1e-06);
    Solver.SolverOption(RelativeTolerance, 80, 1e-06);
    Solver.SolverOption(RelativeTolerance, 81, 1e-06);
    Solver.SolverOption(RelativeTolerance, 82, 1e-06);
    Solver.SolverOption(RelativeTolerance, 83, 1e-06);
    Solver.SolverOption(RelativeTolerance, 84, 1e-06);
    Solver.SolverOption(RelativeTolerance, 85, 1e-06);
    Solver.SolverOption(RelativeTolerance, 86, 1e-06);
    Solver.SolverOption(RelativeTolerance, 87, 1e-06);
    Solver.SolverOption(RelativeTolerance, 88, 1e-06);
    Solver.SolverOption(RelativeTolerance, 89, 1e-06);
    Solver.SolverOption(RelativeTolerance, 90, 1e-06);
    Solver.SolverOption(RelativeTolerance, 91, 1e-06);
    Solver.SolverOption(RelativeTolerance, 92, 1e-06);
    Solver.SolverOption(RelativeTolerance, 93, 1e-06);
    Solver.SolverOption(RelativeTolerance, 94, 1e-06);
    Solver.SolverOption(RelativeTolerance, 95, 1e-06);
    Solver.SolverOption(RelativeTolerance, 96, 1e-06);
    Solver.SolverOption(RelativeTolerance, 97, 1e-06);
    Solver.SolverOption(RelativeTolerance, 98, 1e-06);
    Solver.SolverOption(RelativeTolerance, 99, 1e-06);
    Solver.SolverOption(RelativeTolerance, 100, 1e-06);
    Solver.SolverOption(RelativeTolerance, 101, 1e-06);
    Solver.SolverOption(RelativeTolerance, 102, 1e-06);
    Solver.SolverOption(RelativeTolerance, 103, 1e-06);
    Solver.SolverOption(RelativeTolerance, 104, 1e-06);
    Solver.SolverOption(RelativeTolerance, 105, 1e-06);
    Solver.SolverOption(RelativeTolerance, 106, 1e-06);
    Solver.SolverOption(RelativeTolerance, 107, 1e-06);
    Solver.SolverOption(RelativeTolerance, 108, 1e-06);
    Solver.SolverOption(RelativeTolerance, 109, 1e-06);
    Solver.SolverOption(RelativeTolerance, 110, 1e-06);
    Solver.SolverOption(RelativeTolerance, 111, 1e-06);
    Solver.SolverOption(RelativeTolerance, 112, 1e-06);
    Solver.SolverOption(RelativeTolerance, 113, 1e-06);
    Solver.SolverOption(RelativeTolerance, 114, 1e-06);
    Solver.SolverOption(RelativeTolerance, 115, 1e-06);
    Solver.SolverOption(RelativeTolerance, 116, 1e-06);
    Solver.SolverOption(RelativeTolerance, 117, 1e-06);
    Solver.SolverOption(RelativeTolerance, 118, 1e-06);
    Solver.SolverOption(RelativeTolerance, 119, 1e-06);
    Solver.SolverOption(RelativeTolerance, 120, 1e-06);
    Solver.SolverOption(RelativeTolerance, 121, 1e-06);
    Solver.SolverOption(RelativeTolerance, 122, 1e-06);
    Solver.SolverOption(RelativeTolerance, 123, 1e-06);
    Solver.SolverOption(RelativeTolerance, 124, 1e-06);
    Solver.SolverOption(RelativeTolerance, 125, 1e-06);
    Solver.SolverOption(RelativeTolerance, 126, 1e-06);
    Solver.SolverOption(RelativeTolerance, 127, 1e-06);
    Solver.SolverOption(RelativeTolerance, 128, 1e-06);
    Solver.SolverOption(RelativeTolerance, 129, 1e-06);
    Solver.SolverOption(RelativeTolerance, 130, 1e-06);
    Solver.SolverOption(RelativeTolerance, 131, 1e-06);
    Solver.SolverOption(RelativeTolerance, 132, 1e-06);
    Solver.SolverOption(RelativeTolerance, 133, 1e-06);
    Solver.SolverOption(RelativeTolerance, 134, 1e-06);
    Solver.SolverOption(RelativeTolerance, 135, 1e-06);
    Solver.SolverOption(RelativeTolerance, 136, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "Brown2004_NGF_EGF_signaling.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}

