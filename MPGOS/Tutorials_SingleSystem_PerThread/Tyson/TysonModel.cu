#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "TysonModel_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45
#define PRECISION double
const int NT   = 10000;
const int SD   = 4;
const int NCP  = 1;
const int NSP  = 10;
const int NISP = 0;
const int NE   = 2;
const int NA   = 0;
const int NIA  = 0;
const int NDO  = 100;

void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>&, 
    vector<PRECISION>&, vector<PRECISION>&, int, int
);

void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
	int Width = 18;
	DataFile.precision(10);
	DataFile.flags(ios::scientific);
	
	for (int tid=0; tid<NumberOfThreads; tid++)
	{
        DataFile.width(Width); DataFile << "u" << ',';
        DataFile.width(Width); DataFile << "v" << ',';
        DataFile.width(Width); DataFile << "w" << ',';
        DataFile.width(Width); DataFile << "y" << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3);
		DataFile << '\n';
	}
}

int main() {

    int NumberOfProblems = NT;
    int BlockSize        = 32;

    ListCUDADevices();

    int MajorVersion = 6;
    int MinorVersion = 1;

    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);
    
    // Concentrazioni iniziali
    PRECISION C2 = 0;
    PRECISION CP = 0.75;
    PRECISION M  = 0;
    PRECISION pM = 0.25;
    PRECISION Y  = 0;
    PRECISION YP = 0;
    PRECISION CT = 1.0;
    PRECISION YT = 0.25;
    
    // Valore iniziali delle variabili da integrare
    PRECISION InitialCondition_u = M/CT;
    PRECISION InitialCondition_v = (Y + pM + M)/CT;
    PRECISION InitialCondition_w = (pM + M)/CT;
    PRECISION InitialCondition_y = YT/CT;
    vector<PRECISION> Parameter_X = {
        InitialCondition_u, InitialCondition_v,
        InitialCondition_w, InitialCondition_y
    };

    PRECISION Parameter_k1aaCT    = 0.015;
    PRECISION Parameter_k2        = 0;
    PRECISION Parameter_k3CT      = 200;
    PRECISION Parameter_k4        = 180;
    PRECISION Parameter_k4prime   = 0.018;
    PRECISION Parameter_k5tildeP  = 0;
    PRECISION Parameter_k6        = 1;
    PRECISION Parameter_K7        = 0.6;
    PRECISION Parameter_k8tildeP  = 1e+06;
    PRECISION Parameter_k9        = 1000;
    vector<PRECISION> Parameter_shared = {
        Parameter_k1aaCT, Parameter_k2, Parameter_k3CT, Parameter_k4, Parameter_k4prime,
        Parameter_k5tildeP, Parameter_k6, Parameter_K7, Parameter_k8tildeP, Parameter_k9
    };

    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, BlockSize);
    Solver.SolverOption(PreferSharedMemory, 0);
    Solver.SolverOption(InitialTimeStep, 1e-2);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1e+6);
    Solver.SolverOption(MinimumTimeStep, 1e-12);
    Solver.SolverOption(TimeStepGrowLimit, 5.0);
    Solver.SolverOption(TimeStepShrinkLimit, 0.1);
    Solver.SolverOption(EventDirection, 0, 0);
    Solver.SolverOption(EventDirection, 1, 0);
    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);

    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "TysonModel.csv" );
    clock_t SimulationStart = clock();
	clock_t TransientStart;
    clock_t TransientEnd;

    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        FillSolverObject(Solver, Parameter_shared, Parameter_X, i * NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            cout << "Ciao" << endl;
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
	cout << "Test finished!" << endl;

    return 0;
}

void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    vector<PRECISION>& Parameters_Shared, vector<PRECISION>& Variable_X, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
	int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 100);  

        int i = 0;
        for (PRECISION x : Variable_X) {
            Solver.SetHost(ProblemNumber, ActualState, i++, x);
        }

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );
		
		ProblemNumber++;
        k_begin++;
    }

    int s{0};
    for (PRECISION shared : Parameters_Shared) {
        Solver.SetHost(SharedParameters, s++, shared);
    }
}
