#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "BIOMD0000000005_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45
#define PRECISION double
const int NT   = 1000;
const int SD   = 28;
const int NCP  = 0;
const int NSP  = 10;
const int NISP = 0;
const int NE   = 0;
const int NA   = 20;
const int NIA  = 0;
const int NDO  = 1000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>&, 
    vector<PRECISION>&, vector<PRECISION>&, int, int
);


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product1" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product11" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product13" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product15" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product17" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product19" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product2" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product4" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product6" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product9" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant0" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant10" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant12" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant14" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant16" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant18" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant3" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant5" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant7" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant8" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_C2_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_CP_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_EmptySet_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_M_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_YP_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_Y_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_cell" << ',';
        DataFile.width(Width); DataFile << "X_elmt_cell.elmt_pM_amount" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction1_elmt_k6" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction2_elmt_k8notP" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction3_elmt_k9" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction4_elmt_k3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction5_elmt_k5notP" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction6_elmt_k1aa" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction7_elmt_k2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction8_elmt_k7" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction9_elmt_k4" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_Reaction9_elmt_k4prime" << ',';
        DataFile.width(Width); DataFile << "ACC_$cse1" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_C2_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_CP_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_CT_amount" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_EmptySet_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_M_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction1" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction2" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction3" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction4" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction5" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction6" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction7" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction8" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Reaction9" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_YP_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_YT_amount" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_YT_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_Y_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_cell.elmt_pM_conc" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 19) << ',';

        DataFile << '\n';
    }
}


int main() {
    int numberOfProblems = 2000; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 1024; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    PRECISION InitialCondtition_Reacs_elmt_product1=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product11=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product13=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product15=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product17=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product19=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product2=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product4=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product6=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product9=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant0=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant10=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant12=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant14=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant16=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant18=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant3=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant5=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant7=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant8=1.0;
    PRECISION InitialCondtition_elmt_cell_elmt_cell=1.0;
    vector<PRECISION> Parameter_X = {InitialCondition_Reacs_elmt_product1,InitialCondition_Reacs_elmt_product11,InitialCondition_Reacs_elmt_product13,InitialCondition_Reacs_elmt_product15,InitialCondition_Reacs_elmt_product17,InitialCondition_Reacs_elmt_product19,InitialCondition_Reacs_elmt_product2,InitialCondition_Reacs_elmt_product4,InitialCondition_Reacs_elmt_product6,InitialCondition_Reacs_elmt_product9,InitialCondition_Reacs_elmt_reactant0,InitialCondition_Reacs_elmt_reactant10,InitialCondition_Reacs_elmt_reactant12,InitialCondition_Reacs_elmt_reactant14,InitialCondition_Reacs_elmt_reactant16,InitialCondition_Reacs_elmt_reactant18,InitialCondition_Reacs_elmt_reactant3,InitialCondition_Reacs_elmt_reactant5,InitialCondition_Reacs_elmt_reactant7,InitialCondition_Reacs_elmt_reactant8,InitialCondition_elmt_cell_elmt_cell};
    PRECISION InitialCondtition_Reacs_elmt_Reaction1_elmt_k6=1.0;
    PRECISION InitialCondtition_Reacs_elmt_Reaction2_elmt_k8notP=1000000.0;
    PRECISION InitialCondtition_Reacs_elmt_Reaction3_elmt_k9=1000.0;
    PRECISION InitialCondtition_Reacs_elmt_Reaction4_elmt_k3=200.0;
    PRECISION InitialCondtition_Reacs_elmt_Reaction5_elmt_k5notP=0.0;
    PRECISION InitialCondtition_Reacs_elmt_Reaction6_elmt_k1aa=0.015;
    PRECISION InitialCondtition_Reacs_elmt_Reaction7_elmt_k2=0.0;
    PRECISION InitialCondtition_Reacs_elmt_Reaction8_elmt_k7=0.6;
    PRECISION InitialCondtition_Reacs_elmt_Reaction9_elmt_k4=180.0;
    PRECISION InitialCondtition_Reacs_elmt_Reaction9_elmt_k4prime=0.018;
    vector<PRECISION> Parameter_sPAR = {InitialCondition_Reacs_elmt_Reaction1_elmt_k6,InitialCondition_Reacs_elmt_Reaction2_elmt_k8notP,InitialCondition_Reacs_elmt_Reaction3_elmt_k9,InitialCondition_Reacs_elmt_Reaction4_elmt_k3,InitialCondition_Reacs_elmt_Reaction5_elmt_k5notP,InitialCondition_Reacs_elmt_Reaction6_elmt_k1aa,InitialCondition_Reacs_elmt_Reaction7_elmt_k2,InitialCondition_Reacs_elmt_Reaction8_elmt_k7,InitialCondition_Reacs_elmt_Reaction9_elmt_k4,InitialCondition_Reacs_elmt_Reaction9_elmt_k4prime};


    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 10.0);
    Solver.SolverOption(TimeStepShrinkLimit, 0.2);

    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "BIOMD0000000005.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        fillSolverObject(Solver, Parameter_sPAR, Parameter_X, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    vector<PRECISION>& Parameter_sPAR,  vector<PRECISION>& Variable_X, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 10.0);  

        int i = 0;
        for (PRECISION x : Variable_X) {
            Solver.SetHost(ProblemNumber, ActualState, i++, x);
        }

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        Solver.SetHost(ProblemNumber, Accessories, 0, 0.0);
        Solver.SetHost(ProblemNumber, Accessories, 1, 0.0);
        Solver.SetHost(ProblemNumber, Accessories, 2, 0.0);
        Solver.SetHost(ProblemNumber, Accessories, 3, 0.0);
        Solver.SetHost(ProblemNumber, Accessories, 4, 0.0);

		
		ProblemNumber++;
        k_begin++;
    }

    int spar_i{0};
    for (PRECISION spar: Parameter_sPAR){
        Solver.SetHost(SharedParameters, spar_i++, spar);
    }


}

