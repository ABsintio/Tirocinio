#include "hip/hip_runtime.h"


#include <iostream>
#include <vector>
#include <iomanip>
#include <string>
#include <fstream>

#define PI 3.14159265358979323846

#include "Goldbeter1995_SystemDefinition.cuh"
#include "SingleSystem_PerThread_Interface.cuh"

using namespace std;

#define SOLVER RKCK45
#define PRECISION double
const int NT   = 1000;
const int SD   = 29;
const int NCP  = 0;
const int NSP  = 18;
const int NISP = 0;
const int NE   = 0;
const int NA   = 20;
const int NIA  = 0;
const int NDO  = 1000;


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>&, 
    vector<PRECISION>&, vector<PRECISION>&, int, int
);


void SaveData(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    ofstream& DataFile, int NumberOfThreads
) {
    int Width = 18;
    DataFile.precision(10);
    DataFile.flags(ios::scientific);
	
    for (int tid=0; tid<NumberOfThreads; tid++)
    {
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product1" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product11" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product13" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product15" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product17" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product19" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product3" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product5" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product7" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_product9" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant0" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant10" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant12" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant14" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant16" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant18" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant2" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant4" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant6" << ',';
        DataFile.width(Width); DataFile << "X_Reacs.elmt_reactant8" << ',';
        DataFile.width(Width); DataFile << "X_elmt_CYTOPLASM.elmt_CYTOPLASM" << ',';
        DataFile.width(Width); DataFile << "X_elmt_CYTOPLASM.elmt_M_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_CYTOPLASM.elmt_P0_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_CYTOPLASM.elmt_P1_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_CYTOPLASM.elmt_P2_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_compartment_0000004.elmt_Pn_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_compartment_0000004.elmt_compartment_0000004" << ',';
        DataFile.width(Width); DataFile << "X_elmt_default.elmt_EmptySet_amount" << ',';
        DataFile.width(Width); DataFile << "X_elmt_default.elmt_default" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rM_elmt_KI" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rM_elmt_Vs" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rM_elmt_n" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rP01_elmt_K1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rP01_elmt_V1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rP10_elmt_K2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rP10_elmt_V2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rP12_elmt_K3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rP12_elmt_V3" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rP21_elmt_K4" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rP21_elmt_V4" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rP2n_elmt_k1" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rPn2_elmt_k2" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rTL_elmt_ks" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rVd_elmt_Kd" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rVd_elmt_Vd" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rmRNAd_elmt_Km" << ',';
        DataFile.width(Width); DataFile << "sPAR_Reacs.elmt_rmRNAd_elmt_Vm" << ',';
        DataFile.width(Width); DataFile << "ACC_$cse1" << ',';
        DataFile.width(Width); DataFile << "ACC_$cse2" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_CYTOPLASM.elmt_M_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_CYTOPLASM.elmt_P0_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_CYTOPLASM.elmt_P1_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_CYTOPLASM.elmt_P2_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_CYTOPLASM.elmt_Pt_amount" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_CYTOPLASM.elmt_Pt_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_CYTOPLASM.elmt_rM" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_CYTOPLASM.elmt_rP01" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_CYTOPLASM.elmt_rP10" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_CYTOPLASM.elmt_rP12" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_CYTOPLASM.elmt_rP21" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_CYTOPLASM.elmt_rTL" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_CYTOPLASM.elmt_rVd" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_CYTOPLASM.elmt_rmRNAd" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_compartment_0000004.elmt_Pn_conc" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_compartment_0000004.elmt_rP2n" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_compartment_0000004.elmt_rPn2" << ',';
        DataFile.width(Width); DataFile << "ACC_elmt_default.elmt_EmptySet_conc" << ',';
        DataFile.width(Width); DataFile << endl;
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 19) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 20) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 21) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 22) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 23) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 24) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 25) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 26) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 27) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, ActualState, 28) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(SharedParameters, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 0) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 1) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 2) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 3) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 4) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 5) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 6) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 7) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 8) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 9) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 10) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 11) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 12) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 13) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 14) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 15) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 16) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 17) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 18) << ',';
        DataFile.width(Width); DataFile << Solver.GetHost<PRECISION>(tid, Accessories, 19) << ',';

        DataFile << '\n';
    }
}


int main() {
    int numberOfProblems = 2000; // Numero di problemi da risolvere, uno per thread
    int blockSize        = 1024; // Numero di Thread per blocchi
    
    // Listing dei Device CUDA
    ListCUDADevices();

    int MajorVersion = 7; // Major version della CUDA compute capability
    int MinorVersion = 5; // Minor version della CUDA compute capability

    // Seleziona il Device da utilizzare dando in input la CUDA compute capability e ne stampa le caratteristiche
    int SelectedDevice = SelectDeviceByClosestRevision(MajorVersion, MinorVersion);
    PrintPropertiesOfSpecificDevice(SelectedDevice);

    PRECISION InitialCondtition_Reacs_elmt_product1=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product11=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product13=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product15=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product17=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product19=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product3=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product5=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product7=1.0;
    PRECISION InitialCondtition_Reacs_elmt_product9=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant0=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant10=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant12=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant14=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant16=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant18=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant2=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant4=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant6=1.0;
    PRECISION InitialCondtition_Reacs_elmt_reactant8=1.0;
    PRECISION InitialCondtition_elmt_CYTOPLASM_elmt_CYTOPLASM=1e-15;
    PRECISION InitialCondtition_elmt_compartment_0000004_elmt_compartment_0000004=1e-15;
    PRECISION InitialCondtition_elmt_default_elmt_default=1e-15;
    vector<PRECISION> Parameter_X = {InitialCondition_Reacs_elmt_product1,InitialCondition_Reacs_elmt_product11,InitialCondition_Reacs_elmt_product13,InitialCondition_Reacs_elmt_product15,InitialCondition_Reacs_elmt_product17,InitialCondition_Reacs_elmt_product19,InitialCondition_Reacs_elmt_product3,InitialCondition_Reacs_elmt_product5,InitialCondition_Reacs_elmt_product7,InitialCondition_Reacs_elmt_product9,InitialCondition_Reacs_elmt_reactant0,InitialCondition_Reacs_elmt_reactant10,InitialCondition_Reacs_elmt_reactant12,InitialCondition_Reacs_elmt_reactant14,InitialCondition_Reacs_elmt_reactant16,InitialCondition_Reacs_elmt_reactant18,InitialCondition_Reacs_elmt_reactant2,InitialCondition_Reacs_elmt_reactant4,InitialCondition_Reacs_elmt_reactant6,InitialCondition_Reacs_elmt_reactant8,InitialCondition_elmt_CYTOPLASM_elmt_CYTOPLASM,InitialCondition_elmt_compartment_0000004_elmt_compartment_0000004,InitialCondition_elmt_default_elmt_default};
    PRECISION InitialCondtition_Reacs_elmt_rM_elmt_KI=1.0;
    PRECISION InitialCondtition_Reacs_elmt_rM_elmt_Vs=0.76;
    PRECISION InitialCondtition_Reacs_elmt_rM_elmt_n=4.0;
    PRECISION InitialCondtition_Reacs_elmt_rP01_elmt_K1=2.0;
    PRECISION InitialCondtition_Reacs_elmt_rP01_elmt_V1=3.2;
    PRECISION InitialCondtition_Reacs_elmt_rP10_elmt_K2=2.0;
    PRECISION InitialCondtition_Reacs_elmt_rP10_elmt_V2=1.58;
    PRECISION InitialCondtition_Reacs_elmt_rP12_elmt_K3=2.0;
    PRECISION InitialCondtition_Reacs_elmt_rP12_elmt_V3=5.0;
    PRECISION InitialCondtition_Reacs_elmt_rP21_elmt_K4=2.0;
    PRECISION InitialCondtition_Reacs_elmt_rP21_elmt_V4=2.5;
    PRECISION InitialCondtition_Reacs_elmt_rP2n_elmt_k1=1.9;
    PRECISION InitialCondtition_Reacs_elmt_rPn2_elmt_k2=1.3;
    PRECISION InitialCondtition_Reacs_elmt_rTL_elmt_ks=0.38;
    PRECISION InitialCondtition_Reacs_elmt_rVd_elmt_Kd=0.2;
    PRECISION InitialCondtition_Reacs_elmt_rVd_elmt_Vd=0.95;
    PRECISION InitialCondtition_Reacs_elmt_rmRNAd_elmt_Km=0.5;
    PRECISION InitialCondtition_Reacs_elmt_rmRNAd_elmt_Vm=0.65;
    vector<PRECISION> Parameter_sPAR = {InitialCondition_Reacs_elmt_rM_elmt_KI,InitialCondition_Reacs_elmt_rM_elmt_Vs,InitialCondition_Reacs_elmt_rM_elmt_n,InitialCondition_Reacs_elmt_rP01_elmt_K1,InitialCondition_Reacs_elmt_rP01_elmt_V1,InitialCondition_Reacs_elmt_rP10_elmt_K2,InitialCondition_Reacs_elmt_rP10_elmt_V2,InitialCondition_Reacs_elmt_rP12_elmt_K3,InitialCondition_Reacs_elmt_rP12_elmt_V3,InitialCondition_Reacs_elmt_rP21_elmt_K4,InitialCondition_Reacs_elmt_rP21_elmt_V4,InitialCondition_Reacs_elmt_rP2n_elmt_k1,InitialCondition_Reacs_elmt_rPn2_elmt_k2,InitialCondition_Reacs_elmt_rTL_elmt_ks,InitialCondition_Reacs_elmt_rVd_elmt_Kd,InitialCondition_Reacs_elmt_rVd_elmt_Vd,InitialCondition_Reacs_elmt_rmRNAd_elmt_Km,InitialCondition_Reacs_elmt_rmRNAd_elmt_Vm};


    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION> Solver(SelectedDevice);
    Solver.SolverOption(ThreadsPerBlock, blockSize);
    Solver.SolverOption(PreferSharedMemory, 1);
    Solver.SolverOption(InitialTimeStep, 0.01);
    Solver.SolverOption(ActiveNumberOfThreads, NT);
    Solver.SolverOption(MaximumTimeStep, 1000000.0);
    Solver.SolverOption(MinimumTimeStep, 1e-14);
    Solver.SolverOption(TimeStepGrowLimit, 10.0);
    Solver.SolverOption(TimeStepShrinkLimit, 0.2);

    Solver.SolverOption(DenseOutputMinimumTimeStep, 0.0);
    Solver.SolverOption(DenseOutputSaveFrequency, 1);
    Solver.SolverOption(AbsoluteTolerance, 0, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 1, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 2, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 3, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 4, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 5, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 6, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 7, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 8, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 9, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 10, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 11, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 12, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 13, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 14, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 15, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 16, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 17, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 18, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 19, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 20, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 21, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 22, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 23, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 24, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 25, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 26, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 27, 1e-06);
    Solver.SolverOption(AbsoluteTolerance, 28, 1e-06);

    Solver.SolverOption(RelativeTolerance, 0, 1e-06);
    Solver.SolverOption(RelativeTolerance, 1, 1e-06);
    Solver.SolverOption(RelativeTolerance, 2, 1e-06);
    Solver.SolverOption(RelativeTolerance, 3, 1e-06);
    Solver.SolverOption(RelativeTolerance, 4, 1e-06);
    Solver.SolverOption(RelativeTolerance, 5, 1e-06);
    Solver.SolverOption(RelativeTolerance, 6, 1e-06);
    Solver.SolverOption(RelativeTolerance, 7, 1e-06);
    Solver.SolverOption(RelativeTolerance, 8, 1e-06);
    Solver.SolverOption(RelativeTolerance, 9, 1e-06);
    Solver.SolverOption(RelativeTolerance, 10, 1e-06);
    Solver.SolverOption(RelativeTolerance, 11, 1e-06);
    Solver.SolverOption(RelativeTolerance, 12, 1e-06);
    Solver.SolverOption(RelativeTolerance, 13, 1e-06);
    Solver.SolverOption(RelativeTolerance, 14, 1e-06);
    Solver.SolverOption(RelativeTolerance, 15, 1e-06);
    Solver.SolverOption(RelativeTolerance, 16, 1e-06);
    Solver.SolverOption(RelativeTolerance, 17, 1e-06);
    Solver.SolverOption(RelativeTolerance, 18, 1e-06);
    Solver.SolverOption(RelativeTolerance, 19, 1e-06);
    Solver.SolverOption(RelativeTolerance, 20, 1e-06);
    Solver.SolverOption(RelativeTolerance, 21, 1e-06);
    Solver.SolverOption(RelativeTolerance, 22, 1e-06);
    Solver.SolverOption(RelativeTolerance, 23, 1e-06);
    Solver.SolverOption(RelativeTolerance, 24, 1e-06);
    Solver.SolverOption(RelativeTolerance, 25, 1e-06);
    Solver.SolverOption(RelativeTolerance, 26, 1e-06);
    Solver.SolverOption(RelativeTolerance, 27, 1e-06);
    Solver.SolverOption(RelativeTolerance, 28, 1e-06);
   
    
    int NumberOfSimulationLaunches = NumberOfProblems / NT + (NumberOfProblems % NT == 0 ? 0:1);
    ofstream DataFile;
    DataFile.open ( "Goldbeter1995.csv" );
    clock_t SimulationStart = clock();
    clock_t TransientStart;
    clock_t TransientEnd;    
    for (int i=0; i < NumberOfSimulationLaunches; i++) {
        fillSolverObject(Solver, Parameter_sPAR, Parameter_X, i*NT, NT);
        Solver.SynchroniseFromHostToDevice(All);
        Solver.InsertSynchronisationPoint();
        Solver.SynchroniseSolver();

        TransientStart = clock();
        for (int j=0; j < 1; j++) {
            Solver.Solve();
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
        }
        TransientEnd = clock();
        cout << "Launches: " << i << "  Simulation time: ";
        cout << 1000.0*(TransientEnd-TransientStart) / CLOCKS_PER_SEC << "ms" << endl << endl;
        for (int j=0; j<1; j++)
        {
            Solver.Solve();
            Solver.SynchroniseFromDeviceToHost(All);
            Solver.InsertSynchronisationPoint();
            Solver.SynchroniseSolver();
            SaveData(Solver, DataFile, NT);
        }
    }

    clock_t SimulationEnd = clock();
    cout << "Total simulation time: " << 1000.0*(SimulationEnd-SimulationStart) / CLOCKS_PER_SEC << "ms";
    cout << endl;
	
    DataFile.close();
    
    Solver.Print(DenseOutput, 0);
    Solver.Print(ActualState);
    Solver.Print(ActualTime);
	
    cout << "Test finished!" << endl;

    return 0;
}


void FillSolverObject(
    ProblemSolver<NT,SD,NCP,NSP,NISP,NE,NA,NIA,NDO,SOLVER,PRECISION>& Solver, 
    vector<PRECISION>& Parameter_sPAR,  vector<PRECISION>& Variable_X, 
    int FirstProblemNumber, int NumberOfThreads
) {
    int k_begin = FirstProblemNumber;
    int k_end   = FirstProblemNumber + NumberOfThreads;
	
    int ProblemNumber = 0;
    while (k_begin < k_end) {
        Solver.SetHost(ProblemNumber, TimeDomain, 0, 0.0);
        Solver.SetHost(ProblemNumber, TimeDomain, 1, 10.0);  

        int i = 0;
        for (PRECISION x : Variable_X) {
            Solver.SetHost(ProblemNumber, ActualState, i++, x);
        }

        Solver.SetHost(ProblemNumber, ActualTime, 0.0);
        Solver.SetHost(ProblemNumber, ControlParameters, 0, 0.0);
        Solver.SetHost(ProblemNumber, DenseIndex, 0 );

        Solver.SetHost(ProblemNumber, Accessories, 0, 0.1);
        Solver.SetHost(ProblemNumber, Accessories, 1, 0.25);
        Solver.SetHost(ProblemNumber, Accessories, 2, 0.25);
        Solver.SetHost(ProblemNumber, Accessories, 3, 0.25);
        Solver.SetHost(ProblemNumber, Accessories, 4, 0.25);
        Solver.SetHost(ProblemNumber, Accessories, 5, 0.0);

		
		ProblemNumber++;
        k_begin++;
    }

    int spar_i{0};
    for (PRECISION spar: Parameter_sPAR){
        Solver.SetHost(SharedParameters, spar_i++, spar);
    }


}

